#include "hip/hip_runtime.h"


#include <vtkm/cont/Initialize.h>
#include <vtkm/cont/DataSetBuilderUniform.h>
#include <vtkm/cont/DataSetFieldAdd.h>
#include <vtkm/filter/MapFieldPermutation.h>
#include <vtkm/filter/contour/worklet/Contour.h>
#include <vtkm/filter/contour/Contour.h>

#include <vtkm/rendering/Actor.h>
#include <vtkm/rendering/MapperWireframer.h>
#include <vtkm/rendering/CanvasRayTracer.h>
#include <vtkm/rendering/MapperRayTracer.h>
#include <vtkm/rendering/Scene.h>
#include <vtkm/rendering/View3D.h>

#include <vtkm/io/VTKDataSetReader.h>
#include <vtkm/io/VTKDataSetWriter.h>

#include "mgard/mgard-x/DataRefactoring/MultiDimension/DataRefactoring.hpp"
#include "mgard/mgard-x/DataRefactoring/MultiDimension/DataRefactoringAdaptiveResolution.hpp"
#include "mgard/mgard-x/Utilities/ErrorCalculator.h"

#include <iostream>
#include <vector>

// using namespace mgard_x;

namespace mgard_x {

template <DIM D, typename T, typename DeviceType>
struct SurfaceDetect {
  T iso_value;

  SurfaceDetect(T iso_value): iso_value(iso_value) {}
  bool operator()(AdaptiveResolutionTreeNode<D, T, DeviceType> * node, 
                  typename AdaptiveResolutionTreeNode<D, T, DeviceType>::T_error error, 
                  SubArray<D, T, DeviceType> v){
    SIZE data_index[D];
    T max_data = std::numeric_limits<T>::min();
    T min_data = std::numeric_limits<T>::max();
    for (int i = 0; i < std::pow(2, D); i++) {
      for (int d = 0; d < D; d++) {
        int linearized_index = i;
        if (linearized_index % 2 == 0) {
          data_index[d] = node->index_start_reordered[d];
        } else {
          data_index[d] = node->index_end_reordered[d];
        }
        linearized_index /= 2;
      }
      T data = 0;
      MemoryManager<DeviceType>::Copy1D(&data, v(data_index), 1, 0);
      DeviceRuntime<DeviceType>::SyncQueue(0);
      max_data = std::max(max_data, data);
      min_data = std::min(min_data, data);
    }

    std::cout << "max_data: " << max_data << " "
              << "min_data: " << min_data << " "
              << "error: " << error << "\n";
    if (max_data + error >= iso_value &&
        min_data - error <= iso_value) {
      return true;
    } else {
      return false;
    }

  }
};

template <DIM D, typename T>
void test(T * data, std::vector<SIZE> shape, T tol, T iso_value) {
  std::cout << "Preparing data...";
  //... load data into in_array_cpu
  Hierarchy<D, T, CUDA> hierarchy(shape);
  Array<D, T, CUDA> in_array(shape);
  in_array.load(data);
  SubArray in_subarray(in_array);

  Array<D, T, CUDA> org_array = in_array;
  std::cout << "Done\n";

  // PrintSubarray("Input data", SubArray(org_array));

  std::cout << "Decomposing with MGARD-X CUDA backend...\n";
  decompose(hierarchy, in_subarray, hierarchy.l_target, 0);

  // PrintSubarray("Decomposed data", in_subarray);

  std::cout << "Done\n";

  std::cout << "Recomposing with MGARD-X CUDA backend...\n";
  bool interpolate_full_resolution = true;
  SurfaceDetect<D, T, CUDA> surface_detector(iso_value);
  Array<D, T, CUDA> out_array = recompose_adaptive_resolution(hierarchy, in_subarray, tol, interpolate_full_resolution, surface_detector, 0); 
  std::cout << "Done\n";

  DeviceRuntime<CUDA>::SyncQueue(0);

  size_t n = 1;
  for (int i = 0; i < shape.size(); i++) n *= shape[i];
  enum error_bound_type mode = error_bound_type::ABS;
  std::cout << "L_inf_error: " << L_inf_error(n, org_array.hostCopy(), out_array.hostCopy(), mode) << "\n";

  


}

}


VTKM_CONT bool DoMapField(vtkm::cont::DataSet& result,
                          const vtkm::cont::Field& field,
                          vtkm::worklet::Contour& worklet)
{
  if (field.IsFieldPoint())
  {
    auto functor = [&](const auto& concrete) {
      auto fieldArray = worklet.ProcessPointField(concrete);
      result.AddPointField(field.GetName(), fieldArray);
    };
    field.GetData()
      .CastAndCallForTypesWithFloatFallback<vtkm::TypeListField, VTKM_DEFAULT_STORAGE_LIST>(
        functor);
    return true;
  }
  else if (field.IsFieldCell())
  {
    // Use the precompiled field permutation function.
    vtkm::cont::ArrayHandle<vtkm::Id> permutation = worklet.GetCellIdMap();
    return vtkm::filter::MapFieldPermutation(field, permutation, result);
  }
  else if (field.IsFieldGlobal())
  {
    result.AddField(field);
    return true;
  }
  else
  {
    return false;
  }
}

template <mgard_x::DIM D, typename T>
void test_vtkm(int argc, char *argv[], T * data, std::vector<mgard_x::SIZE> shape, T tol, T iso_value) {
  vtkm::cont::Initialize(argc, argv);
  vtkm::cont::ScopedRuntimeDeviceTracker(vtkm::cont::DeviceAdapterTagCuda{});
  vtkm::cont::DataSet dataSet;
  vtkm::cont::DataSetBuilderUniform dataSetBuilder;
  vtkm::cont::DataSetFieldAdd dsf;
  if (D == 2) shape.push_back(1);
  vtkm::Id3 dims(shape[2], shape[1], shape[0]);
  vtkm::Id3 org(0,0,0);
  vtkm::Id3 spc(1,1,1);
  dataSet = dataSetBuilder.Create(dims, org, spc);
  std::vector<T> data_vec(shape[0]*shape[1]*shape[2]);
  for (int i = 0; i < shape[0]*shape[1]*shape[2]; i++) {
    data_vec[i] = data[i];
  }
  std::string field_name = "test_field";
  dsf.AddPointField(dataSet, field_name, data_vec);
  vtkm::filter::Contour contour_filter;
  contour_filter.SetGenerateNormals(true);
  contour_filter.SetMergeDuplicatePoints(false);
  contour_filter.SetIsoValue(0, iso_value);
  // contour_filter.SetIsoValue(1, iso_value+10);
  contour_filter.SetActiveField(field_name);
  contour_filter.SetFieldsToPass({ field_name });
  vtkm::cont::DataSet outputData = contour_filter.Execute(dataSet);

  

  vtkm::cont::DataSet outputData2;
  vtkm::cont::Field field = dataSet.GetField(0);
  // outputData2.AddField(field);
  std::vector<T> iso_data_vec(shape[0]*shape[1]*shape[2], iso_value);
  outputData2.AddPointField(field_name, iso_data_vec);

  vtkm::cont::CellSetSingleType<> cellset = outputData.GetCellSet().AsCellSet<vtkm::cont::CellSetSingleType<>>();

  vtkm::cont::CellSetSingleType<> cellset2;
  vtkm::cont::ArrayHandle<vtkm::Id, VTKM_DEFAULT_CONNECTIVITY_STORAGE_TAG> connectivity;
  connectivity.Allocate(cellset.GetNumberOfCells()*3);
  vtkm::cont::ArrayHandle<vtkm::Id, VTKM_DEFAULT_CONNECTIVITY_STORAGE_TAG>::WritePortalType writePortal = connectivity.WritePortal();
  for (vtkm::Id i = 0; i < cellset.GetNumberOfCells(); i++) {
    vtkm::Vec<vtkm::Id, 3> ids;
    cellset.GetIndices(i, ids);
    writePortal.Set(i*3, ids[0]);
    writePortal.Set(i*3+1, ids[1]);
    writePortal.Set(i*3+2, ids[2]);
  }

  cellset2.Fill(cellset.GetNumberOfPoints(),
                vtkm::CELL_SHAPE_TRIANGLE, 3,
                connectivity);

  outputData2.SetCellSet(cellset2);
  outputData2.AddCoordinateSystem(outputData.GetCoordinateSystem());

  vtkm::io::VTKDataSetWriter writer_output("contour_output.vtk");
  writer_output.WriteDataSet(outputData);

  vtkm::io::VTKDataSetWriter writer_input("input.vtk");
  writer_input.WriteDataSet(dataSet);

  std::cout << "outputData.GetNumberOfCells() = " << outputData.GetNumberOfCells() << "\n";
  std::cout << "outputData.GetNumberOfPoints() = " << outputData.GetNumberOfPoints() << "\n";
 
  // using Mapper = vtkm::rendering::MapperWireframer;
  using Mapper = vtkm::rendering::MapperRayTracer;
  using Canvas = vtkm::rendering::CanvasRayTracer;

  vtkm::rendering::Scene scene;
  vtkm::cont::ColorTable colorTable("inferno");
  vtkm::cont::ColorTable colorTable2("inferno");
  scene.AddActor(vtkm::rendering::Actor(outputData2.GetCellSet(),
                                      outputData2.GetCoordinateSystem(),
                                      outputData2.GetField(field_name),
                                      colorTable));

  // scene.AddActor(vtkm::rendering::Actor(outputData.GetCellSet(),
  //                                     outputData.GetCoordinateSystem(),
  //                                     outputData.GetField(field_name),
  //                                     colorTable));

  Mapper mapper;
  Canvas canvas(1024, 1024);

  vtkm::rendering::Color bg(0.2f, 0.2f, 0.2f, 1.0f);

  const vtkm::cont::CoordinateSystem coords = outputData2.GetCoordinateSystem();
  vtkm::Bounds coordsBounds = coords.GetBounds();
  vtkm::rendering::Camera camera = vtkm::rendering::Camera();
  // camera.SetViewUp(vtkm::make_Vec(0.f, 0.f, 1.f));
  camera.ResetToBounds(coordsBounds);

  vtkm::Vec<vtkm::Float32, 3> totalExtent;
  totalExtent[0] = vtkm::Float32(shape[2]);
  totalExtent[1] = vtkm::Float32(shape[1]);
  totalExtent[2] = vtkm::Float32(shape[0]);
  vtkm::Float32 mag = vtkm::Magnitude(totalExtent);
  vtkm::Normalize(totalExtent);
  camera.SetLookAt(totalExtent * (mag * .5f));
  camera.SetViewUp(vtkm::make_Vec(0.f, 0.f, 1.f));
  // camera.SetClippingRange(1.f, 1000.f);
  camera.SetFieldOfView(60.f);
  camera.SetPosition(totalExtent * (mag * 2.f));
  vtkm::rendering::View3D view(scene, mapper, canvas, camera, bg);
  view.Initialize();
  view.Paint();
  string outputfile = "output";
  view.SaveAs(outputfile + " .pnm"); 
}





bool require_arg(int argc, char *argv[], std::string option) {
  for (int i = 0; i < argc; i++) {
    if (option.compare(std::string(argv[i])) == 0) {
      return true;
    }
  }
  std::cout << "missing option: " + option + "\n";
  return false;
}

std::string get_arg(int argc, char *argv[], std::string option) {
  if (require_arg(argc, argv, option)) {
    for (int i = 0; i < argc; i++) {
      if (option.compare(std::string(argv[i])) == 0) {
        return std::string(argv[i + 1]);
      }
    }
  }
  return std::string("");
}

std::vector<mgard_x::SIZE> get_arg_dims(int argc, char *argv[],
                                        std::string option) {
  std::vector<mgard_x::SIZE> shape;
  if (require_arg(argc, argv, option)) {
    std::string arg;
    int arg_idx = 0, i;
    for (i = 0; i < argc; i++) {
      if (option.compare(std::string(argv[i])) == 0) {
        arg = std::string(argv[i + 1]);
        arg_idx = i + 1;
      }
    }
    try {
      int d = std::stoi(arg);
      for (int i = 0; i < d; i++) {
        shape.push_back(std::stoi(argv[arg_idx + 1 + i]));
      }
      return shape;
    } catch (std::invalid_argument const &e) {
      std::cout << "illegal argument for option " + option + "\n";
      return shape;
    }
  }
  return shape;
}

double get_arg_double(int argc, char *argv[], std::string option) {
  if (require_arg(argc, argv, option)) {
    std::string arg;
    int i;
    for (i = 0; i < argc; i++) {
      if (option.compare(std::string(argv[i])) == 0) {
        arg = std::string(argv[i + 1]);
      }
    }
    try {
      double d = std::stod(arg);
      return d;
    } catch (std::invalid_argument const &e) {
      std::cout << "illegal argument for option " + option + "\n";
    }
  }
  return 0;
}

template <typename T> size_t readfile(const char *input_file, T *&in_buff) {
  std::cout << "Loading file: " << input_file << "\n";

  FILE *pFile;
  pFile = fopen(input_file, "rb");
  if (pFile == NULL) {
    std::cout << "file open error!\n";
    exit(1);
  }
  fseek(pFile, 0, SEEK_END);
  size_t lSize = ftell(pFile);
  rewind(pFile);
  in_buff = (T *)malloc(lSize);
  lSize = fread(in_buff, 1, lSize, pFile);
  fclose(pFile);
  // min_max(lSize/sizeof(T), in_buff);
  return lSize;
}

int main(int argc, char *argv[]) {
  std::string input_file = get_arg(argc, argv, "-i");
  std::string dt = get_arg(argc, argv, "-t");
  std::vector<mgard_x::SIZE> shape = get_arg_dims(argc, argv, "-n");
  double tol = get_arg_double(argc, argv, "-e");
  double iso_value = get_arg_double(argc, argv, "-v");

  if (dt.compare("s") == 0) {
    float * data = NULL;
    readfile(input_file.c_str(), data);
    if (shape.size() == 2) {
      test_vtkm<2, float>(argc, argv, data, shape, (float)tol, (float)iso_value);
      // mgard_x::test<2, float>(data, shape, (float)tol, (float)iso_value);
    } else if (shape.size() == 3) {
      test_vtkm<3, float>(argc, argv, data, shape, (float)tol, (float)iso_value);
      // mgard_x::test<3, float>(data, shape, (float)tol, (float)iso_value);
    } else {
      std::cout << "wrong num of dim.\n";
    }
  } else if (dt.compare("d") == 0) {
    double * data = NULL;
    readfile(input_file.c_str(), data);
    if (shape.size() == 2) {
      test_vtkm<2, double>(argc, argv, data, shape, (float)tol, (float)iso_value);
      // mgard_x::test<2, double>(data, shape, (double)tol, (float)iso_value);
    } else if (shape.size() == 3) {
      test_vtkm<3, double>(argc, argv, data, shape, (float)tol, (float)iso_value);
      // mgard_x::test<3, double>(data, shape, (double)tol, (float)iso_value);
    } else {
      std::cout << "wrong num of dim.\n";
    }
  } else {
    std::cout << "wrong data type.\n";
  }
  
  
  // prepare
  // std::cout << "Preparing data...";
  // double *in_array_cpu = new double[n1 * n2 * n3];
  // //... load data into in_array_cpu
  // std::vector<SIZE> shape{n1, n2, n3};
  // Hierarchy<3, double, CUDA> hierarchy(shape);
  // Array<3, double, CUDA> in_array(shape);
  // in_array.load(in_array_cpu);
  // SubArray in_subarray(in_array);

  // std::cout << "Done\n";

  // std::cout << "Decomposing with MGARD-X CUDA backend...";
  // decompose(hierarchy, in_subarray, hierarchy.l_target, 0);
  // Get compressed size in number of bytes.
  // size_t compressed_size = compressed_array.shape()[0];
  // unsigned char *compressed_array_cpu = compressed_array.hostCopy();
  // std::cout << "Done\n";

  // std::cout << "Decompressing with MGARD-X CUDA backend...";
  // // decompression
  // Array<3, double, CUDA> decompressed_array =
  //     decompress(hierarchy, compressed_array,
  //                         error_bound_type::REL, tol, s, norm, config);
  // delete[] in_array_cpu;
  // double *decompressed_array_cpu = decompressed_array.hostCopy();
  // std::cout << "Done\n";
}