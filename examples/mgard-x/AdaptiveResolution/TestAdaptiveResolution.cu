#include "hip/hip_runtime.h"


#include <vtkm/cont/Initialize.h>
#include <vtkm/cont/DataSetBuilderUniform.h>
#include <vtkm/cont/DataSetFieldAdd.h>
#include <vtkm/filter/MapFieldPermutation.h>
#include <vtkm/filter/contour/worklet/Contour.h>
#include <vtkm/filter/contour/Contour.h>

#include <vtkm/rendering/Actor.h>
#include <vtkm/rendering/MapperWireframer.h>
#include <vtkm/rendering/CanvasRayTracer.h>
#include <vtkm/rendering/MapperRayTracer.h>
#include <vtkm/rendering/Scene.h>
#include <vtkm/rendering/View3D.h>

#include <vtkm/io/VTKDataSetReader.h>
#include <vtkm/io/VTKDataSetWriter.h>

#include "mgard/mgard-x/DataRefactoring/MultiDimension/DataRefactoring.hpp"
#include "mgard/mgard-x/DataRefactoring/MultiDimension/DataRefactoringAdaptiveResolution.hpp"
#include "mgard/mgard-x/Utilities/ErrorCalculator.h"

#include "SparseFlyingEdges.hpp"

#include <iostream>
#include <vector>

// using namespace mgard_x;


void vtkm_render(vtkm::cont::DataSet dataSet, std::vector<mgard_x::SIZE> shape, std::string field_name, std::string output) {
  using Mapper = vtkm::rendering::MapperRayTracer;
  using Canvas = vtkm::rendering::CanvasRayTracer;

  vtkm::rendering::Scene scene;
  vtkm::cont::ColorTable colorTable("inferno");
  scene.AddActor(vtkm::rendering::Actor(dataSet.GetCellSet(),
                                      dataSet.GetCoordinateSystem(),
                                      dataSet.GetField(field_name),
                                      colorTable));

  Mapper mapper;
  Canvas canvas(1024, 1024);

  vtkm::rendering::Color bg(0.2f, 0.2f, 0.2f, 1.0f);

  const vtkm::cont::CoordinateSystem coords = dataSet.GetCoordinateSystem();
  vtkm::Bounds coordsBounds = coords.GetBounds();
  vtkm::rendering::Camera camera = vtkm::rendering::Camera();
  // camera.SetViewUp(vtkm::make_Vec(0.f, 0.f, 1.f));
  camera.ResetToBounds(coordsBounds);

  vtkm::Vec<vtkm::Float32, 3> totalExtent;
  totalExtent[0] = vtkm::Float32(shape[2]);
  totalExtent[1] = vtkm::Float32(shape[1]);
  totalExtent[2] = vtkm::Float32(shape[0]);
  vtkm::Float32 mag = vtkm::Magnitude(totalExtent);
  vtkm::Normalize(totalExtent);
  camera.SetLookAt(totalExtent * (mag * .5f));
  camera.SetViewUp(vtkm::make_Vec(0.f, 0.f, 1.f));
  // camera.SetClippingRange(1.f, 1000.f);
  camera.SetFieldOfView(60.f);
  camera.SetPosition(totalExtent * (mag * 2.f));
  vtkm::rendering::View3D view(scene, mapper, canvas, camera, bg);
  view.Initialize();
  view.Paint();
  view.SaveAs(output + " .pnm"); 
}

template <typename T>
vtkm::cont::DataSet ArrayToDataset(std::vector<mgard_x::SIZE> shape, T iso_value,
                                  mgard_x::Array<1, mgard_x::SIZE, mgard_x::CUDA> TrianglesArray,
                                  mgard_x::Array<1, T, mgard_x::CUDA> PointsArray,
                                  std::string field_name) {
  mgard_x::SIZE * Triangles = new mgard_x::SIZE[TrianglesArray.shape()[0]];
  T * Points = new T[PointsArray.shape()[0]];

  mgard_x::SIZE numTriangles = TrianglesArray.shape()[0] / 3;
  mgard_x::SIZE numPoints = PointsArray.shape()[0] / 3;

  memcpy(Triangles, TrianglesArray.hostCopy(),
         numTriangles * 3 * sizeof(mgard_x::SIZE));
  memcpy(Points, PointsArray.hostCopy(), numPoints * 3 * sizeof(T));

  // mgard_x::PrintSubarray("Triangles", mgard_x::SubArray(TrianglesArray));
  // mgard_x::PrintSubarray("Points", mgard_x::SubArray(PointsArray));

  vtkm::cont::DataSet ds_from_mc;
  std::vector<T> iso_data_vec(shape[0]*shape[1]*shape[2], iso_value);
  ds_from_mc.AddPointField(field_name, iso_data_vec);
  vtkm::cont::CellSetSingleType<> cellset;
  vtkm::cont::ArrayHandle<vtkm::Id, VTKM_DEFAULT_CONNECTIVITY_STORAGE_TAG> connectivity;
  connectivity.Allocate(TrianglesArray.shape()[0]);

  // std::cout << "connectivity.GetNumberOfValues() = " << connectivity.GetNumberOfValues() << "\n";

  vtkm::cont::ArrayHandle<vtkm::Id, VTKM_DEFAULT_CONNECTIVITY_STORAGE_TAG>::WritePortalType writePortal = connectivity.WritePortal();
  for (vtkm::Id i = 0; i < numTriangles; i++) {
    writePortal.Set(i*3, Triangles[i*3]);
    writePortal.Set(i*3+1, Triangles[i*3+1]);
    writePortal.Set(i*3+2, Triangles[i*3+2]);
  }

  cellset.Fill(numPoints,
                vtkm::CELL_SHAPE_TRIANGLE, 3,
                connectivity);
  ds_from_mc.SetCellSet(cellset);

  vtkm::cont::ArrayHandle<vtkm::Vec3f> coordinate_points;
  coordinate_points.Allocate(numPoints);
  for (vtkm::Id pointId = 0; pointId < numPoints; pointId++) {
    vtkm::Vec3f point;
    point[0] = Points[pointId*3];
    point[1] = Points[pointId*3+1];
    point[2] = Points[pointId*3+2];
    coordinate_points.WritePortal().Set(pointId, point);
  }
  vtkm::cont::CoordinateSystem coordinate_system("cs", coordinate_points);
  ds_from_mc.AddCoordinateSystem(coordinate_system);

  return ds_from_mc;
}

template <mgard_x::DIM D, typename T>
void test_vtkm(int argc, char *argv[], T * data, std::vector<mgard_x::SIZE> shape, T tol, T iso_value) {
  vtkm::cont::Initialize(argc, argv);
  vtkm::cont::ScopedRuntimeDeviceTracker(vtkm::cont::DeviceAdapterTagCuda{});
  vtkm::cont::DataSet dataSet;
  vtkm::cont::DataSetBuilderUniform dataSetBuilder;
  vtkm::cont::DataSetFieldAdd dsf;
  if (D == 2) shape.push_back(1);
  vtkm::Id3 dims(shape[2], shape[1], shape[0]);
  vtkm::Id3 org(0,0,0);
  vtkm::Id3 spc(1,1,1);
  dataSet = dataSetBuilder.Create(dims, org, spc);
  std::vector<T> data_vec(shape[0]*shape[1]*shape[2]);
  for (int i = 0; i < shape[0]*shape[1]*shape[2]; i++) {
    data_vec[i] = data[i];
  }
  std::string field_name = "test_field";
  dsf.AddPointField(dataSet, field_name, data_vec);
  vtkm::filter::Contour contour_filter;
  contour_filter.SetGenerateNormals(true);
  contour_filter.SetMergeDuplicatePoints(false);
  contour_filter.SetIsoValue(0, iso_value);
  // contour_filter.SetIsoValue(1, iso_value+10);
  contour_filter.SetActiveField(field_name);
  contour_filter.SetFieldsToPass({ field_name });
  vtkm::cont::DataSet outputData = contour_filter.Execute(dataSet);

  // std::cout << "outputData.GetNumberOfCells() = " << outputData.GetNumberOfCells() << "\n";
  // std::cout << "outputData.GetNumberOfPoints() = " << outputData.GetNumberOfPoints() << "\n";

  std::cout << "vtkm::FlyingEdges::numPoints: " << outputData.GetNumberOfPoints() << "\n";
  std::cout << "vtkm::FlyingEdges::numTris: " << outputData.GetNumberOfCells() << "\n";
 
  vtkm_render(outputData, shape, field_name, "vtkm_render_output");
}

template <mgard_x::DIM D, typename T>
void test_mine(T *original_data, std::vector<mgard_x::SIZE> shape, T iso_value) {

  mgard_x::SIZE numTriangles;
  mgard_x::SIZE *Triangles;
  mgard_x::SIZE numPoints;
  T *Points;

  mgard_x::Array<3, T, mgard_x::CUDA> v(shape);
  v.load(original_data);

  mgard_x::PrintSubarray("input", mgard_x::SubArray<3, T, mgard_x::CUDA>(v));

  mgard_x::Array<1, mgard_x::SIZE, mgard_x::CUDA> TrianglesArray;
  mgard_x::Array<1, T, mgard_x::CUDA> PointsArray;

  mgard_x::FlyingEdges<T, mgard_x::CUDA>().Execute(
      shape[0], shape[1], shape[2], mgard_x::SubArray<3, T, mgard_x::CUDA>(v),
      iso_value, TrianglesArray, PointsArray, 0);

  // numTriangles = TrianglesArray.shape()[0] / 3;
  // numPoints = PointsArray.shape()[0] / 3;

  // if (numTriangles == 0 || numPoints == 0) {
  //   printf("returing %u %u from test_mine\n", numTriangles, numPoints);
  //   return;
  // }

  // Triangles = new mgard_x::SIZE[TrianglesArray.shape()[0]];
  // Points = new T[PointsArray.shape()[0]];

  // memcpy(Triangles, TrianglesArray.hostCopy(),
  //        numTriangles * 3 * sizeof(mgard_x::SIZE));
  // memcpy(Points, PointsArray.hostCopy(), numPoints * 3 * sizeof(T));

  // // mgard_x::PrintSubarray("Triangles", mgard_x::SubArray(TrianglesArray));
  // // mgard_x::PrintSubarray("Points", mgard_x::SubArray(PointsArray));

  // std::string field_name = "test_field";
  // vtkm::cont::DataSet ds_from_mc;
  // std::vector<T> iso_data_vec(shape[0]*shape[1]*shape[2], iso_value);
  // ds_from_mc.AddPointField(field_name, iso_data_vec);
  // vtkm::cont::CellSetSingleType<> cellset;
  // vtkm::cont::ArrayHandle<vtkm::Id, VTKM_DEFAULT_CONNECTIVITY_STORAGE_TAG> connectivity;
  // connectivity.Allocate(TrianglesArray.shape()[0]);

  // // std::cout << "connectivity.GetNumberOfValues() = " << connectivity.GetNumberOfValues() << "\n";

  // vtkm::cont::ArrayHandle<vtkm::Id, VTKM_DEFAULT_CONNECTIVITY_STORAGE_TAG>::WritePortalType writePortal = connectivity.WritePortal();
  // for (vtkm::Id i = 0; i < numTriangles; i++) {
  //   writePortal.Set(i*3, Triangles[i*3]);
  //   writePortal.Set(i*3+1, Triangles[i*3+1]);
  //   writePortal.Set(i*3+2, Triangles[i*3+2]);
  // }

  // cellset.Fill(numPoints,
  //               vtkm::CELL_SHAPE_TRIANGLE, 3,
  //               connectivity);
  // ds_from_mc.SetCellSet(cellset);

  // vtkm::cont::ArrayHandle<vtkm::Vec3f> coordinate_points;
  // coordinate_points.Allocate(numPoints);
  // for (vtkm::Id pointId = 0; pointId < numPoints; pointId++) {
  //   vtkm::Vec3f point;
  //   point[0] = Points[pointId*3];
  //   point[1] = Points[pointId*3+1];
  //   point[2] = Points[pointId*3+2];
  //   coordinate_points.WritePortal().Set(pointId, point);
  // }
  // vtkm::cont::CoordinateSystem coordinate_system("cs", coordinate_points);
  // ds_from_mc.AddCoordinateSystem(coordinate_system);
  std::string field_name = "test_field";
  vtkm::cont::DataSet dataset = ArrayToDataset(shape, iso_value, TrianglesArray, PointsArray, field_name);
  vtkm_render(dataset, shape, field_name, "my_flying_edges");

}




namespace mgard_x {

template <DIM D, typename T, typename DeviceType>
struct SurfaceDetect {
  T iso_value;

  SurfaceDetect(T iso_value): iso_value(iso_value) {}
  bool operator()(AdaptiveResolutionTreeNode<D, T, DeviceType> * node, 
                  typename AdaptiveResolutionTreeNode<D, T, DeviceType>::T_error error, 
                  SubArray<D, T, DeviceType> v){
    SIZE data_index[D];
    T max_data = std::numeric_limits<T>::min();
    T min_data = std::numeric_limits<T>::max();
    // std::cout << "cell: ";
    for (int i = 0; i < std::pow(2, D); i++) {
      int linearized_index = i;
      for (int d = 0; d < D; d++) {
        if (linearized_index % 2 == 0) {
          data_index[d] = node->index_start_reordered[d];
        } else {
          data_index[d] = node->index_end_reordered[d];
        }
        linearized_index /= 2;
      }
      T data = 0;
      MemoryManager<DeviceType>::Copy1D(&data, v(data_index), 1, 0);
      DeviceRuntime<DeviceType>::SyncQueue(0);
      max_data = std::max(max_data, data);
      min_data = std::min(min_data, data);
      // std::cout << data << ", ";
    }

    // std::cout << "("<<node->index_start[2] << ", " << node->index_start[1] << ", " << node->index_start[0] << "), ";
    // std::cout << "("<<node->index_end[2] << ", " << node->index_end[1] << ", " << node->index_end[0] << ")";
   
    // return true;
    if (max_data + error >= iso_value &&
        min_data - error <= iso_value) {
       // std::cout << "max_data: " << max_data << " "
       //        << "min_data: " << min_data << " "
       //        << "error: " << error << "\n";
      // std::cout << "Keep\n";
      return true;
    } else {
      // std::cout << "Discard\n";
      return false;
    }

  }
};

template <DIM D, typename T>
void test(T * data, std::vector<SIZE> shape, T tol, T iso_value) {
  std::cout << "Preparing data...";
  //... load data into in_array_cpu
  Hierarchy<D, T, CUDA> hierarchy(shape);
  Array<D, T, CUDA> in_array(shape);
  in_array.load(data);
  SubArray in_subarray(in_array);

  Array<D, T, CUDA> org_array = in_array;
  std::cout << "Done\n";

  // PrintSubarray("Input data", SubArray(org_array));

  std::cout << "Decomposing with MGARD-X CUDA backend...\n";
  decompose(hierarchy, in_subarray, hierarchy.l_target, 0);

  // PrintSubarray("Decomposed data", in_subarray);

  std::cout << "Done\n";

  std::cout << "Recomposing with MGARD-X CUDA backend...\n";
  bool interpolate_full_resolution = true;
  SurfaceDetect<D, T, CUDA> surface_detector(iso_value);
  std::vector<CompressedSparseEdge<D, T, CUDA>> cse_list;
  Array<D, T, CUDA> out_array = recompose_adaptive_resolution(hierarchy, in_subarray, tol, interpolate_full_resolution, surface_detector, cse_list, 0); 
  std::cout << "Done\n";

  DeviceRuntime<CUDA>::SyncQueue(0);
  size_t n = 1;
  for (int i = 0; i < shape.size(); i++) n *= shape[i];
  enum error_bound_type mode = error_bound_type::ABS;
  std::cout << "L_inf_error: " << L_inf_error(n, org_array.hostCopy(), out_array.hostCopy(), mode) << "\n";

  for (int i = 0; i < cse_list.size(); i++) {
    if(!cse_list[i].empty) {
      mgard_x::Array<1, mgard_x::SIZE, mgard_x::CUDA> TrianglesArray;
      mgard_x::Array<1, T, mgard_x::CUDA> PointsArray;
      SparseFlyingEdges<D, T, CUDA>().Execute(cse_list[i], iso_value, TrianglesArray, PointsArray, 0);
      std::string field_name = "test_field";
      vtkm::cont::DataSet dataset = ArrayToDataset(shape, iso_value, TrianglesArray, PointsArray, field_name);
      vtkm_render(dataset, shape, field_name, "my_render_output");
    }
  }
}

}


bool require_arg(int argc, char *argv[], std::string option) {
  for (int i = 0; i < argc; i++) {
    if (option.compare(std::string(argv[i])) == 0) {
      return true;
    }
  }
  std::cout << "missing option: " + option + "\n";
  return false;
}

std::string get_arg(int argc, char *argv[], std::string option) {
  if (require_arg(argc, argv, option)) {
    for (int i = 0; i < argc; i++) {
      if (option.compare(std::string(argv[i])) == 0) {
        return std::string(argv[i + 1]);
      }
    }
  }
  return std::string("");
}

std::vector<mgard_x::SIZE> get_arg_dims(int argc, char *argv[],
                                        std::string option) {
  std::vector<mgard_x::SIZE> shape;
  if (require_arg(argc, argv, option)) {
    std::string arg;
    int arg_idx = 0, i;
    for (i = 0; i < argc; i++) {
      if (option.compare(std::string(argv[i])) == 0) {
        arg = std::string(argv[i + 1]);
        arg_idx = i + 1;
      }
    }
    try {
      int d = std::stoi(arg);
      for (int i = 0; i < d; i++) {
        shape.push_back(std::stoi(argv[arg_idx + 1 + i]));
      }
      return shape;
    } catch (std::invalid_argument const &e) {
      std::cout << "illegal argument for option " + option + "\n";
      return shape;
    }
  }
  return shape;
}

double get_arg_double(int argc, char *argv[], std::string option) {
  if (require_arg(argc, argv, option)) {
    std::string arg;
    int i;
    for (i = 0; i < argc; i++) {
      if (option.compare(std::string(argv[i])) == 0) {
        arg = std::string(argv[i + 1]);
      }
    }
    try {
      double d = std::stod(arg);
      return d;
    } catch (std::invalid_argument const &e) {
      std::cout << "illegal argument for option " + option + "\n";
    }
  }
  return 0;
}

template <typename T> size_t readfile(const char *input_file, T *&in_buff) {
  std::cout << "Loading file: " << input_file << "\n";

  FILE *pFile;
  pFile = fopen(input_file, "rb");
  if (pFile == NULL) {
    std::cout << "file open error!\n";
    exit(1);
  }
  fseek(pFile, 0, SEEK_END);
  size_t lSize = ftell(pFile);
  rewind(pFile);
  in_buff = (T *)malloc(lSize);
  lSize = fread(in_buff, 1, lSize, pFile);
  fclose(pFile);
  // min_max(lSize/sizeof(T), in_buff);
  return lSize;
}

int main(int argc, char *argv[]) {
  std::string input_file = get_arg(argc, argv, "-i");
  std::string dt = get_arg(argc, argv, "-t");
  std::vector<mgard_x::SIZE> shape = get_arg_dims(argc, argv, "-n");
  double tol = get_arg_double(argc, argv, "-e");
  double iso_value = get_arg_double(argc, argv, "-v");

  if (dt.compare("s") == 0) {
    float * data = NULL;
    readfile(input_file.c_str(), data);
    if (shape.size() == 2) {
      test_vtkm<2, float>(argc, argv, data, shape, (float)tol, (float)iso_value);
      test_mine<2, float>(data, shape, (float)iso_value);
      mgard_x::test<2, float>(data, shape, (float)tol, (float)iso_value);
    } else if (shape.size() == 3) {
      test_vtkm<3, float>(argc, argv, data, shape, (float)tol, (float)iso_value);
      test_mine<3, float>(data, shape, (float)iso_value);
      mgard_x::test<3, float>(data, shape, (float)tol, (float)iso_value);
    } else {
      std::cout << "wrong num of dim.\n";
    }
  } else if (dt.compare("d") == 0) {
    double * data = NULL;
    readfile(input_file.c_str(), data);
    if (shape.size() == 2) {
      test_vtkm<2, double>(argc, argv, data, shape, (float)tol, (float)iso_value);
      test_mine<2, double>(data, shape, (float)iso_value);
      mgard_x::test<2, double>(data, shape, (double)tol, (float)iso_value);
    } else if (shape.size() == 3) {
      test_vtkm<3, double>(argc, argv, data, shape, (float)tol, (float)iso_value);
      test_mine<3, double>(data, shape, (float)iso_value);
      mgard_x::test<3, double>(data, shape, (double)tol, (float)iso_value);
    } else {
      std::cout << "wrong num of dim.\n";
    }
  } else {
    std::cout << "wrong data type.\n";
  }
}