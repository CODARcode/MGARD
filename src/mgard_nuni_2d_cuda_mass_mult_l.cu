#include "hip/hip_runtime.h"
#include "mgard_nuni.h"
#include "mgard.h"
#include "mgard_nuni_2d_cuda.h"
#include "mgard_cuda_helper.h"
#include "mgard_cuda_compact_helper.h"
#include "mgard_cuda_helper_internal.h"
#include <fstream>
#include <cmath>

namespace mgard_2d {
namespace mgard_gen {  

__device__ double
_dist_mass_mult_l(double * dcoord, int x, int y) {
  return dcoord[y] - dcoord[x];
}
__global__ void
_mass_mult_l_row_cuda_sm(int nrow,       int ncol,
                         int nr,         int nc,
                         int row_stride, int col_stride,
                         double * __restrict__ dv,    int lddv,
                         // double * __restrict__ dcoords_x,
                         double * ddist_x,
                         int ghost_col) {

  //int ghost_col = 2;
  register int total_row = ceil((double)nr/(row_stride));
  register int total_col = ceil((double)nc/(col_stride));


  // index on dirow and dicol
  register int r0 = (blockIdx.y * blockDim.y + threadIdx.y) * row_stride;
  register int c0 = threadIdx.x;
  register int c0_stride = threadIdx.x * col_stride;

  // index on sm
  register int r0_sm = threadIdx.y;
  register int c0_sm = threadIdx.x;

  extern __shared__ double sm[]; // row = blockDim.y; col = blockDim.x + ghost_col;
  register int ldsm = blockDim.x + ghost_col;
  
  double * vec_sm = sm + r0_sm * ldsm;
  double * dist_x_sm = sm + (blockDim.y) * ldsm;
  
  register double result = 1;
  register double h1 = 1;
  register double h2 = 1;
  
  register int rest_col;
  register int real_ghost_col;
  register int real_main_col;

  register double prev_vec_sm;
  register double prev_dist_x;
  
  for (int r = r0; r < nr; r += gridDim.y * blockDim.y * row_stride) {
    
    double * vec = dv + r * lddv;

    prev_vec_sm = 0.0;
    prev_dist_x = 0.0;
    
    rest_col = total_col;    
    real_ghost_col = min(ghost_col, rest_col);

    // load first ghost
    if (c0_sm < real_ghost_col) {
      vec_sm[c0_sm] = vec[c0_stride];
      if (r0_sm == 0) {
        dist_x_sm[c0_sm] = ddist_x[c0];
      }
    }
    rest_col -= real_ghost_col;
    __syncthreads();

    while (rest_col > blockDim.x - real_ghost_col) {
      //load main column
      real_main_col = min(blockDim.x, rest_col);
      if (c0_sm < real_main_col) {
        vec_sm[c0_sm + real_ghost_col] = vec[c0_stride + real_ghost_col * col_stride];
        if (r0_sm == 0) {
          dist_x_sm[c0_sm + real_ghost_col] = ddist_x[c0 + real_ghost_col];
        }
      }
      __syncthreads();

      //computation
      if (c0_sm == 0) {
        h1 = prev_dist_x;
        h2 = dist_x_sm[c0_sm];
        result = h1 * prev_vec_sm + 2 * (h1 + h2) * vec_sm[c0_sm] + h2 * vec_sm[c0_sm + 1];
      } else {
        h1 = dist_x_sm[c0_sm - 1];
        h2 = dist_x_sm[c0_sm];
        result = h1 * vec_sm[c0_sm - 1] + 2 * (h1 + h2) * vec_sm[c0_sm] + h2 * vec_sm[c0_sm + 1];
      }
      vec[c0] = result;
      __syncthreads();
      
      // store last column
      if (c0_sm == 0) {
        prev_vec_sm = vec_sm[blockDim.x - 1];
        prev_dist_x = dist_x_sm[blockDim.x - 1];
      }

      // advance c0
      c0_stride += blockDim.x * col_stride;
      c0 += blockDim.x;

      // copy ghost to main
      real_ghost_col = min(ghost_col, real_main_col - (blockDim.x - ghost_col));
      if (c0_sm < real_ghost_col) {
        vec_sm[c0_sm] = vec_sm[c0_sm + blockDim.x];
        if (r0_sm == 0) {
          dist_x_sm[c0_sm] = dist_x_sm[c0_sm + blockDim.x];
        }
      }
      __syncthreads();
      rest_col -= real_main_col;
    } //end while

    if (c0_sm < rest_col) {
      vec_sm[c0_sm + real_ghost_col] = vec[c0_stride + real_ghost_col * col_stride];
      if (r0_sm == 0) {
        dist_x_sm[c0_sm + real_ghost_col] = ddist_x[c0 + real_ghost_col];
      }
    }
    __syncthreads();

    if (real_ghost_col + rest_col == 1) {
      if (c0_sm == 0) {
        h1 = prev_dist_x;
        result = h1 * prev_vec_sm + 2 * h1 * vec_sm[c0_sm];
        vec[c0_stride] = result;
      }
    } else {
      if (c0_sm < real_ghost_col + rest_col) {     
        if (c0_sm == 0) {
          h1 = prev_dist_x;
          h2 = dist_x_sm[c0_sm];
          result = h1 * prev_vec_sm + 2 * (h1 + h2) * vec_sm[c0_sm] + h2 * vec_sm[c0_sm + 1];
        } else if (c0_sm == real_ghost_col + rest_col - 1) {
          h1 = dist_x_sm[c0_sm - 1];
          result = h1 * vec_sm[c0_sm - 1] + 2 * h1 * vec_sm[c0_sm];
        } else {
          h1 = dist_x_sm[c0_sm - 1];
          h2 = dist_x_sm[c0_sm];
          result = h1 * vec_sm[c0_sm - 1] + 2 * (h1 + h2) * vec_sm[c0_sm] + h2 * vec_sm[c0_sm + 1];
        }
        __syncthreads();
        vec[c0_stride] = result;
      }
    }
  }
}


__global__ void
_mass_mult_l_row_cuda_sm(int nrow,       int ncol,
                         int nr,         int nc,
                         int row_stride, int col_stride,
                         int * __restrict__ dirow,    int * __restrict__ dicol,
                         double * __restrict__ dv,    int lddv,
                         double * __restrict__ dcoords_x,
                         int ghost_col) {

  //int ghost_col = 2;
  register int total_row = ceil((double)nr/(row_stride));
  register int total_col = ceil((double)nc/(col_stride));


  // index on dirow and dicol
  register int r0 = (blockIdx.y * blockDim.y + threadIdx.y) * row_stride;
  register int c0 = threadIdx.x * col_stride;

  // index on sm
  register int r0_sm = threadIdx.y;
  register int c0_sm = threadIdx.x;

  extern __shared__ double sm[]; // row = blockDim.y; col = blockDim.x + ghost_col;
  register int ldsm = blockDim.x + ghost_col;
  // printf("ldsm = %d\n", ldsm);
  
  double * vec_sm = sm + r0_sm * ldsm;
  double * dcoords_x_sm = sm + blockDim.y * ldsm;
  
  register double result = 1;
  register double h1 = 1;
  register double h2 = 1;
  
  register int rest_col;
  register int real_ghost_col;
  register int real_main_col;

  register double prev_vec_sm;
  register double prev_dicol;
  register double prev_dcoord_x;
  
  for (int r = r0; r < nr; r += gridDim.y * blockDim.y * row_stride) {
    
    double * vec = dv + dirow[r] * lddv;

    prev_vec_sm = 0.0;
    prev_dicol = dicol[c0];
    prev_dcoord_x = dcoords_x[dicol[c0]];
    
    rest_col = total_col;    
    real_ghost_col = min(ghost_col, rest_col);

    // load first ghost
    if (c0_sm < real_ghost_col) {
      vec_sm[c0_sm] = vec[dicol[c0]];
      if (r0_sm == 0) {
        dcoords_x_sm[c0_sm] = dcoords_x[dicol[c0]];
      }
    }
    rest_col -= real_ghost_col;
    __syncthreads();

    while (rest_col > blockDim.x - real_ghost_col) {
      //load main column
      real_main_col = min(blockDim.x, rest_col);
      if (c0_sm < real_main_col) {
        vec_sm[c0_sm + real_ghost_col] = vec[dicol[c0 + real_ghost_col * col_stride]];
        if (r0_sm == 0) {
          dcoords_x_sm[c0_sm + real_ghost_col] = dcoords_x[dicol[c0 + real_ghost_col * col_stride]];
        }
      }
      __syncthreads();

      //computation
      if (c0_sm == 0) {
        //h1 = _dist_mass_mult_l(dcoords_x,  prev_dicol, dicol[c0]);
        h1 = dcoords_x_sm[c0_sm] - prev_dcoord_x;
        //h2 = _dist_mass_mult_l(dcoords_x,  dicol[c0], dicol[c0 + col_stride]);
        h2 = _dist_mass_mult_l(dcoords_x_sm,  c0_sm, c0_sm + 1);
        // register double tmp1 = h1 * prev_vec_sm;
        // register double tmp2 = 2 * (h1 + h2) * vec_sm[c0_sm];
        // register double tmp3 = h2 * vec_sm[c0_sm + 1];
        // tmp1 += tmp2;
        // result += tmp3;
        // result += tmp1;
        //result = h1 + h2;
        result = h1 * prev_vec_sm + 2 * (h1 + h2) * vec_sm[c0_sm] + h2 * vec_sm[c0_sm + 1];
      } else {
        //h1 = _dist_mass_mult_l(dcoords_x, dicol[c0 - col_stride], dicol[c0]);
        h1 = _dist_mass_mult_l(dcoords_x_sm, c0_sm - 1, c0_sm);
        //h2 = _dist_mass_mult_l(dcoords_x, dicol[c0], dicol[c0 + col_stride]);
        h2 = _dist_mass_mult_l(dcoords_x_sm, c0_sm, c0_sm + 1);
        // register double tmp1 = h1 * vec_sm[c0_sm - 1];
        // register double tmp2 = 2 * (h1 + h2) * vec_sm[c0_sm];
        // register double tmp3 = h2 * vec_sm[c0_sm + 1];
        // tmp1 += tmp2;
        // result += tmp3;
        // result += tmp1;
        //result = h1 + h2;
        result = h1 * vec_sm[c0_sm - 1] + 2 * (h1 + h2) * vec_sm[c0_sm] + h2 * vec_sm[c0_sm + 1];
      }
      vec[dicol[c0]] = result;
      __syncthreads();
      

      // store last column
      if (c0_sm == 0) {
        prev_vec_sm = vec_sm[blockDim.x - 1];
        prev_dicol = dicol[c0 + (blockDim.x - 1) * col_stride];
        prev_dcoord_x = dcoords_x[dicol[c0 + (blockDim.x - 1) * col_stride]];//dcoords_x_sm[blockDim.x - 1];
      }

      // advance c0
      c0 += blockDim.x * col_stride;

      // copy ghost to main
      real_ghost_col = min(ghost_col, real_main_col - (blockDim.x - ghost_col));
      if (c0_sm < real_ghost_col) {
        vec_sm[c0_sm] = vec_sm[c0_sm + blockDim.x];
        if (r0_sm == 0) {
          dcoords_x_sm[c0_sm] = dcoords_x_sm[c0_sm + blockDim.x];
        }
      }
      __syncthreads();
      rest_col -= real_main_col;
    } //end while

    if (c0_sm < rest_col) {
       vec_sm[c0_sm + real_ghost_col] = vec[dicol[c0 + real_ghost_col * col_stride]];
       dcoords_x_sm[c0_sm + real_ghost_col] = dcoords_x[dicol[c0 + real_ghost_col * col_stride]];
    }
    __syncthreads();

    if (real_ghost_col + rest_col == 1) {
      if (c0_sm == 0) {
        //h1 = _dist_mass_mult_l(dcoords_x,  prev_dicol, dicol[c0]);
        h1 = dcoords_x_sm[c0_sm] - prev_dcoord_x;
        result = h1 * prev_vec_sm + 2 * h1 * vec_sm[c0_sm];
        vec[dicol[c0]] = result;
      }

    } else {

    if (c0_sm < real_ghost_col + rest_col) {
        
      if (c0_sm == 0) {
        //h1 = _dist_mass_mult_l(dcoords_x,  prev_dicol, dicol[c0]);
        //h2 = _dist_mass_mult_l(dcoords_x,  dicol[c0], dicol[c0 + col_stride]);

        h1 = dcoords_x_sm[c0_sm] - prev_dcoord_x;
        h2 = _dist_mass_mult_l(dcoords_x_sm,  c0_sm, c0_sm + 1);

        result = h1 * prev_vec_sm + 2 * (h1 + h2) * vec_sm[c0_sm] + h2 * vec_sm[c0_sm + 1];
      } else if (c0_sm == real_ghost_col + rest_col - 1) {
        //h1 = _dist_mass_mult_l(dcoords_x, dicol[c0 - col_stride], dicol[c0]);
        h1 = _dist_mass_mult_l(dcoords_x_sm, c0_sm - 1, c0_sm);
        result = h1 * vec_sm[c0_sm - 1] + 2 * h1 * vec_sm[c0_sm];
      } else {
        // h1 = _dist_mass_mult_l(dcoords_x, dicol[c0 - col_stride], dicol[c0]);
        // h2 = _dist_mass_mult_l(dcoords_x, dicol[c0], dicol[c0 + col_stride]);

        h1 = _dist_mass_mult_l(dcoords_x_sm, c0_sm - 1, c0_sm);
        h2 = _dist_mass_mult_l(dcoords_x_sm, c0_sm, c0_sm + 1);
        result = h1 * vec_sm[c0_sm - 1] + 2 * (h1 + h2) * vec_sm[c0_sm] + h2 * vec_sm[c0_sm + 1];
      }
      __syncthreads();
      vec[dicol[c0]] = result;
    }
  }
    

  }
}

mgard_cuda_ret 
mass_mult_l_row_cuda_sm(int nrow,       int ncol,
                     int nr,         int nc,
                     int row_stride, int col_stride,
                     int * dirow,    int * dicol,
                     double * dv,    int lddv,
                     double * dcoords_x,
                     int B, int ghost_col) {
 

  //hipMemcpyToSymbol(HIP_SYMBOL(dcoords_x_const), dcoords_x, sizeof(double)*nc );
  double * ddist_x;
  //int len_ddist_x = ceil((float)nc/col_stride)-1;
  int len_ddist_x = ceil((float)nc/col_stride); // add one for better consistance for backward
  cudaMallocHelper((void**)&ddist_x, len_ddist_x*sizeof(double));
  calc_cpt_dist(nc, col_stride, dcoords_x, ddist_x);
  // printf("dcoords_x %d:\n", nc);
  // print_matrix_cuda(1, nc, dcoords_x, nc);
  // printf("ddist_x:\n");
  // print_matrix_cuda(1, len_ddist_x, ddist_x, len_ddist_x);


  // int B = 4;
  // int ghost_col = 2;
  int total_row = ceil((double)nr/(row_stride));
  int total_col = ceil((double)nc/(col_stride));
  int total_thread_y = ceil((double)nr/(row_stride));
  int total_thread_x = min(B, total_col);

  int tby = min(B, total_thread_y);
  int tbx = min(B, total_thread_x);


  size_t sm_size = ((tbx + ghost_col) * (tby + 1 + 1)) * sizeof(double);

  int gridy = ceil((float)total_thread_y/tby);
  int gridx = 1; //ceil((float)total_thread_x/tbx);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);

  // std::cout << "thread block: " << tby << ", " << tbx << std::endl;
  // std::cout << "grid: " << gridy << ", " << gridx<< std::endl;



  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  _mass_mult_l_row_cuda_sm<<<blockPerGrid, threadsPerBlock, sm_size>>>(nrow,       ncol,
                                                                       nr,         nc,
                                                                       row_stride, col_stride,
                                                                       dv,         lddv,
                                                                       // dcoords_x,
                                                                       ddist_x,
                                                                       ghost_col);

  // _mass_mult_l_row_cuda_sm<<<blockPerGrid, threadsPerBlock, sm_size>>>(nrow,       ncol,
  //                                                                      nr,         nc,
  //                                                                      row_stride, col_stride,
  //                                                                      dirow,      dicol,
  //                                                                      dv,         lddv,
  //                                                                      dcoords_x,
  //                                                                      // ddist_x,
  //                                                                      ghost_col);


  gpuErrchk(hipGetLastError ());

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return mgard_cuda_ret(0, milliseconds/1000.0);
}



__global__ void
_mass_mult_l_col_cuda_sm(int nrow,       int ncol,
                         int nr,         int nc,
                         int row_stride, int col_stride,
                         double * __restrict__ dv,    int lddv,
                         // double * __restrict__ dcoords_x,
                         double * ddist_y,
                         int ghost_row) {

  register int total_row = ceil((double)nr/(row_stride));
  register int total_col = ceil((double)nc/(col_stride));


  // index on dirow and dicol
  register int c0 = (blockIdx.x * blockDim.x + threadIdx.x) * col_stride;
  register int r0 = threadIdx.y;
  register int r0_stride = threadIdx.y * row_stride;
  register int c_dist = threadIdx.x;

  // index on sm
  register int r0_sm = threadIdx.y;
  register int c0_sm = threadIdx.x;

  extern __shared__ double sm[]; // row = blockDim.y; col = blockDim.x + ghost_col;
  register int ldsm = blockDim.x;
  
  double * vec_sm = sm + c0_sm;
  double * dist_y_sm = sm + (blockDim.y + ghost_row) * ldsm;
  
  register double result = 1;
  register double h1 = 1;
  register double h2 = 1;
  
  register int rest_row;
  register int real_ghost_row;
  register int real_main_row;

  register double prev_vec_sm;
  register double prev_dist_y;
  
  for (int c = c0; c < nc; c += gridDim.x * blockDim.x * col_stride) {
    
    double * vec = dv + c;

    prev_vec_sm = 0.0;
    prev_dist_y = 0.0;
    
    rest_row = total_row;    
    real_ghost_row = min(ghost_row, rest_row);

    // load first ghost
    if (r0_sm < real_ghost_row) {
      vec_sm[r0_sm * ldsm] = vec[r0_stride * lddv];
    }
    if (c0_sm == 0 && r0_sm < real_ghost_row) {
        dist_y_sm[r0_sm] = ddist_y[r0];
        // printf("load dist[%d] = %f\n", c0_sm, dist_y_sm[c0_sm]);
    }

    rest_row -= real_ghost_row;
    __syncthreads();

    // if (c0 == nc-1 && r0_sm == 0) {
    //     printf("vec_sm: ");
    //     for (int j = 0; j < blockDim.y + ghost_row; j++) {
    //       printf("%f ", vec_sm[j * ldsm]);
    //     }
    //     printf("\n");
    //     printf("dist_y_sm: ");
    //     for (int j = 0; j < blockDim.y + ghost_row; j++) {
    //       printf("%f ", dist_y_sm[j]);
    //     }
    //     printf("\n");

    //   }
    // __syncthreads();

    while (rest_row > blockDim.y - real_ghost_row) {
      //load main column
      real_main_row = min(blockDim.y, rest_row);
      if (r0_sm < real_main_row) {
        vec_sm[(r0_sm + real_ghost_row) * ldsm] = vec[(r0_stride + real_ghost_row * row_stride) * lddv];
      }
      if (c0_sm == 0 && r0_sm < real_main_row) {
          dist_y_sm[r0_sm + real_ghost_row] = ddist_y[r0 + real_ghost_row];
      }
      __syncthreads();

      // if (c0 == nc-1 && r0_sm == 0) {
      //   printf("vec_sm: ");
      //   for (int j = 0; j < blockDim.y + ghost_row; j++) {
      //     printf("%f ", vec_sm[j * ldsm]);
      //   }
      //   printf("\n");
      //   printf("dist_y_sm: ");
      //   for (int j = 0; j < blockDim.y + ghost_row; j++) {
      //     printf("%f ", dist_y_sm[j]);
      //   }
      //   printf("\n");

      // }
      // __syncthreads();
      //computation
      if (r0_sm == 0) {
        h1 = prev_dist_y;
        h2 = dist_y_sm[r0_sm]; // broadcast from sm
        result = h1 * prev_vec_sm + 2 * (h1 + h2) * vec_sm[r0_sm * ldsm] + h2 * vec_sm[(r0_sm + 1)*ldsm];
      } else {
        h1 = dist_y_sm[r0_sm - 1];
        h2 = dist_y_sm[r0_sm];
        result = h1 * vec_sm[(r0_sm - 1) * ldsm] + 2 * (h1 + h2) * vec_sm[r0_sm * ldsm] + h2 * vec_sm[(r0_sm + 1) * ldsm];
      }
      vec[r0 * lddv] = result;
      __syncthreads();
      
      // store last column
      if (r0_sm == 0) {
        prev_vec_sm = vec_sm[(blockDim.y - 1) * ldsm];
        prev_dist_y = dist_y_sm[blockDim.y - 1];
      }

      // advance c0
      r0_stride += blockDim.y * row_stride;
      r0 += blockDim.y;
      c_dist += blockDim.y;

      // copy ghost to main
      real_ghost_row = min(ghost_row, real_main_row - (blockDim.y - ghost_row));
      if (r0_sm < real_ghost_row) {
        vec_sm[r0_sm * ldsm] = vec_sm[(r0_sm + blockDim.y) * ldsm];
      }
      if (c0_sm == 0 && r0_sm < real_ghost_row) {
        dist_y_sm[r0_sm] = dist_y_sm[r0_sm + blockDim.y];
      }
      __syncthreads();
      rest_row -= real_main_row;
      // if (c0 == nc-1 && r0_sm == 0) {
      //   printf("vec_sm: ");
      //   for (int j = 0; j < blockDim.y + ghost_row; j++) {
      //     printf("%f ", vec_sm[j * ldsm]);
      //   }
      //   printf("\n");
      //   printf("dist_y_sm: ");
      //   for (int j = 0; j < blockDim.y + ghost_row; j++) {
      //     printf("%f ", dist_y_sm[j]);
      //   }
      //   printf("\n");

      // }
      // __syncthreads();


    } //end while

    if (r0_sm < rest_row) {
      vec_sm[(r0_sm + real_ghost_row) * ldsm] = vec[(r0_stride + real_ghost_row * row_stride) * lddv]; 
    }
    if (c0_sm == 0 && r0_sm < rest_row) {
      dist_y_sm[r0_sm + real_ghost_row] = ddist_y[r0 + real_ghost_row];
    }
    __syncthreads();

    if (real_ghost_row + rest_row == 1) {
      if (r0_sm == 0) {
        h1 = prev_dist_y;
        result = h1 * prev_vec_sm + 2 * h1 * vec_sm[r0_sm * ldsm];
        vec[r0_stride * lddv] = result;
      }
    } else {
      if (r0_sm < real_ghost_row + rest_row) {     
        if (r0_sm == 0) {
          h1 = prev_dist_y;
          h2 = dist_y_sm[r0_sm];
          result = h1 * prev_vec_sm + 2 * (h1 + h2) * vec_sm[r0_sm * ldsm] + h2 * vec_sm[(r0_sm + 1) * ldsm];
        } else if (r0_sm == real_ghost_row + rest_row - 1) {
          h1 = dist_y_sm[r0_sm - 1];
          result = h1 * vec_sm[(r0_sm - 1) * ldsm] + 2 * h1 * vec_sm[r0_sm * ldsm];
        } else {
          h1 = dist_y_sm[r0_sm - 1];
          h2 = dist_y_sm[r0_sm];
          result = h1 * vec_sm[(r0_sm - 1) * ldsm] + 2 * (h1 + h2) * vec_sm[r0_sm * ldsm] + h2 * vec_sm[(r0_sm + 1) * ldsm];
        }
        __syncthreads();
        vec[r0_stride * lddv] = result;
      }
    }
  }
}


mgard_cuda_ret 
mass_mult_l_col_cuda_sm(int nrow,       int ncol,
                     int nr,         int nc,
                     int row_stride, int col_stride,
                     int * dirow,    int * dicol,
                     double * dv,    int lddv,
                     double * dcoords_y,
                     int B, int ghost_row) {
 

  //hipMemcpyToSymbol(HIP_SYMBOL(dcoords_x_const), dcoords_x, sizeof(double)*nc );
  double * ddist_y;
  //int len_ddist_x = ceil((float)nc/col_stride)-1;
  int len_ddist_y = ceil((float)nr/row_stride); // add one for better consistance for backward
  cudaMallocHelper((void**)&ddist_y, len_ddist_y*sizeof(double));
  calc_cpt_dist(nr, row_stride, dcoords_y, ddist_y);
  // printf("dcoords_y %d:\n", nc);
  // print_matrix_cuda(1, nr, dcoords_y, nr);
  // printf("ddist_y:\n");
  // print_matrix_cuda(1, len_ddist_y, ddist_y, len_ddist_y);


  // int B = 4;
  // int ghost_row = 2;
  int total_row = ceil((double)nr/(row_stride));
  int total_col = ceil((double)nc/(col_stride));
  int total_thread_y = min(B, total_row);
  int total_thread_x = ceil((double)nc/(col_stride));

  int tby = min(B, total_thread_y);
  int tbx = min(B, total_thread_x);


  size_t sm_size = ((tby + ghost_row) * (tbx + 1)) * sizeof(double);

  int gridy = 1;
  int gridx = ceil((float)total_thread_x/tbx); //ceil((float)total_thread_x/tbx);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);

  // std::cout << "thread block: " << tby << ", " << tbx << std::endl;
  // std::cout << "grid: " << gridy << ", " << gridx<< std::endl;



  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  _mass_mult_l_col_cuda_sm<<<blockPerGrid, threadsPerBlock, sm_size>>>(nrow,       ncol,
                                                                       nr,         nc,
                                                                       row_stride, col_stride,
                                                                       dv,         lddv,
                                                                       // dcoords_x,
                                                                       ddist_y,
                                                                       ghost_row);

  // _mass_mult_l_row_cuda_sm<<<blockPerGrid, threadsPerBlock, sm_size>>>(nrow,       ncol,
  //                                                                      nr,         nc,
  //                                                                      row_stride, col_stride,
  //                                                                      dirow,      dicol,
  //                                                                      dv,         lddv,
  //                                                                      dcoords_x,
  //                                                                      // ddist_x,
  //                                                                      ghost_col);


  gpuErrchk(hipGetLastError ());

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return mgard_cuda_ret(0, milliseconds/1000.0);
}



__global__ void
_mass_mult_l_row_cuda_sm_pf(int nrow,       int ncol,
                         int nr,         int nc,
                         int row_stride, int col_stride,
                         int * __restrict__ dirow,    int * __restrict__ dicol,
                         double * __restrict__ dv,    int lddv,
                         double * __restrict__ dcoords_x,
                         int ghost_col) {

  //int ghost_col = 2;
  register int total_row = ceil((double)nr/(row_stride));
  register int total_col = ceil((double)nc/(col_stride));


  // index on dirow and dicol
  register int r0 = (blockIdx.y * blockDim.y + threadIdx.y) * row_stride;
  register int c0 = threadIdx.x * col_stride;

  // index on sm
  register int r0_sm = threadIdx.y;
  register int c0_sm = threadIdx.x;

  extern __shared__ double sm[]; // row = blockDim.y; col = blockDim.x + ghost_col;
  register int ldsm = blockDim.x + ghost_col;
  // printf("ldsm = %d\n", ldsm);
  
  double * vec_sm = sm + r0_sm * ldsm;
  double * dcoords_x_sm = sm + blockDim.y * ldsm;
  
  register double result = 1;
  register double h1 = 1;
  register double h2 = 1;
  
  register int main_col = blockDim.x;
  register int rest_load_col;
  register int rest_comp_col;
  register int curr_ghost_col;
  register int curr_main_col;
  register int next_ghost_col;
  register int next_main_col;

  register double prev_vec_sm;
  register double prev_dicol;
  register double prev_dcoord_x;

  register double next_dv;
  register double next_dcoords_x;
  
  for (int r = r0; r < nr; r += gridDim.y * blockDim.y * row_stride) {
    
    double * vec = dv + dirow[r] * lddv;

    prev_vec_sm = 0.0;
    prev_dicol = dicol[c0];
    prev_dcoord_x = dcoords_x[dicol[c0]];
    
    rest_load_col = total_col;
    rest_comp_col = total_col;
    curr_ghost_col = min(ghost_col, rest_load_col);

    // load first ghost
    if (c0_sm < curr_ghost_col) {
      vec_sm[c0_sm] = vec[dicol[c0]];
      if (r0_sm == 0) {
        dcoords_x_sm[c0_sm] = dcoords_x[dicol[c0]];
      }
    }
    rest_load_col -= curr_ghost_col;
    //load main column
    curr_main_col = min(blockDim.x, rest_load_col);
    if (c0_sm < curr_main_col) {
      vec_sm[c0_sm + curr_ghost_col] = vec[dicol[c0 + curr_ghost_col * col_stride]];
      if (r0_sm == 0) {
        dcoords_x_sm[c0_sm + curr_ghost_col] = dcoords_x[dicol[c0 + curr_ghost_col * col_stride]];
      }
    }
    rest_load_col -= curr_main_col;
    __syncthreads();



    while (rest_comp_col > main_col) {
      //load next main column
      next_main_col = min(blockDim.x, rest_load_col);
      int next_c0 = c0 + (curr_main_col + curr_ghost_col) * col_stride;
      if (c0_sm < next_main_col) {
        next_dv = vec[dicol[next_c0]];
        if (r0_sm == 0) {
          next_dcoords_x = dcoords_x[dicol[next_c0]];
        }
      }
      __syncthreads();

      //computation
      if (c0_sm == 0) {
        //h1 = _dist_mass_mult_l(dcoords_x,  prev_dicol, dicol[c0]);
        h1 = dcoords_x_sm[c0_sm] - prev_dcoord_x;
        //h2 = _dist_mass_mult_l(dcoords_x,  dicol[c0], dicol[c0 + col_stride]);
        h2 = _dist_mass_mult_l(dcoords_x_sm,  c0_sm, c0_sm + 1);

        result = h1 * prev_vec_sm + 2 * (h1 + h2) * vec_sm[c0_sm] + h2 * vec_sm[c0_sm + 1];
      } else {
        //h1 = _dist_mass_mult_l(dcoords_x, dicol[c0 - col_stride], dicol[c0]);
        h1 = _dist_mass_mult_l(dcoords_x_sm, c0_sm - 1, c0_sm);
        //h2 = _dist_mass_mult_l(dcoords_x, dicol[c0], dicol[c0 + col_stride]);
        h2 = _dist_mass_mult_l(dcoords_x_sm, c0_sm, c0_sm + 1);

        result = h1 * vec_sm[c0_sm - 1] + 2 * (h1 + h2) * vec_sm[c0_sm] + h2 * vec_sm[c0_sm + 1];
      }
      vec[dicol[c0]] = result;
      
      rest_comp_col -= main_col;

      // store last column
      if (c0_sm == 0) {
        prev_vec_sm = vec_sm[blockDim.x - 1];
        prev_dicol = dicol[c0 + (blockDim.x - 1) * col_stride];
        prev_dcoord_x = dcoords_x[dicol[c0 + (blockDim.x - 1) * col_stride]];//dcoords_x_sm[blockDim.x - 1];
      }

      __syncthreads();
      


      // advance c0
      c0 += blockDim.x * col_stride;

      // copy ghost to main
      next_ghost_col = curr_main_col + curr_ghost_col - main_col;
      if (c0_sm < next_ghost_col) {
        vec_sm[c0_sm] = vec_sm[c0_sm + main_col];
        if (r0_sm == 0) {
          dcoords_x_sm[c0_sm] = dcoords_x_sm[c0_sm + main_col];
        }
      }
      __syncthreads();
      // copy next main to main
      if (c0_sm < next_main_col) {
        vec_sm[c0_sm + next_ghost_col] = next_dv;
        if (r0_sm == 0) {
          dcoords_x_sm[c0_sm + next_ghost_col] = next_dcoords_x;
        }
      }
      rest_load_col -= next_main_col;

      curr_ghost_col = next_ghost_col;
      curr_main_col = next_main_col;
      //rest_col -= real_main_col;
    } //end while

    // if (c0_sm < col) {
    //    vec_sm[c0_sm + real_ghost_col] = vec[dicol[c0 + real_ghost_col * col_stride]];
    //    dcoords_x_sm[c0_sm + real_ghost_col] = dcoords_x[dicol[c0 + real_ghost_col * col_stride]];
    // }
    // __syncthreads();

    if (rest_comp_col == 1) {
      if (c0_sm == 0) {
        //h1 = _dist_mass_mult_l(dcoords_x,  prev_dicol, dicol[c0]);
        h1 = dcoords_x_sm[c0_sm] - prev_dcoord_x;
        result = h1 * prev_vec_sm + 2 * h1 * vec_sm[c0_sm];
        vec[dicol[c0]] = result;
      }

    } else {

    if (c0_sm < rest_comp_col) {
        
      if (c0_sm == 0) {
        //h1 = _dist_mass_mult_l(dcoords_x,  prev_dicol, dicol[c0]);
        //h2 = _dist_mass_mult_l(dcoords_x,  dicol[c0], dicol[c0 + col_stride]);

        h1 = dcoords_x_sm[c0_sm] - prev_dcoord_x;
        h2 = _dist_mass_mult_l(dcoords_x_sm,  c0_sm, c0_sm + 1);

        result = h1 * prev_vec_sm + 2 * (h1 + h2) * vec_sm[c0_sm] + h2 * vec_sm[c0_sm + 1];
      } else if (c0_sm == rest_comp_col - 1) {
        //h1 = _dist_mass_mult_l(dcoords_x, dicol[c0 - col_stride], dicol[c0]);
        h1 = _dist_mass_mult_l(dcoords_x_sm, c0_sm - 1, c0_sm);
        result = h1 * vec_sm[c0_sm - 1] + 2 * h1 * vec_sm[c0_sm];
      } else {
        // h1 = _dist_mass_mult_l(dcoords_x, dicol[c0 - col_stride], dicol[c0]);
        // h2 = _dist_mass_mult_l(dcoords_x, dicol[c0], dicol[c0 + col_stride]);

        h1 = _dist_mass_mult_l(dcoords_x_sm, c0_sm - 1, c0_sm);
        h2 = _dist_mass_mult_l(dcoords_x_sm, c0_sm, c0_sm + 1);
        result = h1 * vec_sm[c0_sm - 1] + 2 * (h1 + h2) * vec_sm[c0_sm] + h2 * vec_sm[c0_sm + 1];
      }
      __syncthreads();
      vec[dicol[c0]] = result;
    }
  }
    

  }
}


mgard_cuda_ret 
mass_mult_l_row_cuda_sm_pf(int nrow,       int ncol,
                     int nr,         int nc,
                     int row_stride, int col_stride,
                     int * dirow,    int * dicol,
                     double * dv,    int lddv,
                     double * dcoords_x,
                     int B, int ghost_col) {
 

  //hipMemcpyToSymbol(HIP_SYMBOL(dcoords_x_const), dcoords_x, sizeof(double)*nc );

  // int B = 4;
  // int ghost_col = 2;
  int total_row = ceil((double)nr/(row_stride));
  int total_col = ceil((double)nc/(col_stride));
  int total_thread_y = ceil((double)nr/(row_stride));
  int total_thread_x = min(B, total_col);

  int tby = min(B, total_thread_y);
  int tbx = min(B, total_thread_x);


  size_t sm_size = ((tbx + ghost_col) * (tby + 1)) * sizeof(double);

  int gridy = ceil((float)total_thread_y/tby);
  int gridx = 1; //ceil((float)total_thread_x/tbx);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);

  // std::cout << "thread block: " << tby << ", " << tbx << std::endl;
  // std::cout << "grid: " << gridy << ", " << gridx<< std::endl;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  _mass_mult_l_row_cuda_sm_pf<<<blockPerGrid, threadsPerBlock, sm_size>>>(nrow,       ncol,
                                                                       nr,         nc,
                                                                       row_stride, col_stride,
                                                                       dirow,      dicol,
                                                                       dv,         lddv,
                                                                       dcoords_x,
                                                                       ghost_col);
  gpuErrchk(hipGetLastError ());

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return mgard_cuda_ret(0, milliseconds/1000.0);
}
} // mgard_gen
} // mgard_2d
