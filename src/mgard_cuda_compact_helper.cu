#include "hip/hip_runtime.h"
#include <iomanip> 
#include <iostream>
#include "mgard_cuda_helper.h"
#include "mgard_cuda_helper_internal.h"
#include "mgard_cuda.h"
#include "mgard_cuda_compacted.h"

/* 3D Original to (2^k)+1 */
__global__ void
_org_to_pow2p1(int nfib, int nrow,     int ncol,   
               int nf,   int nr,       int nc,     
               int ifib, int * irow,   int * icol, 
               double * dv,  int lddv1, int lddv2,
               double * dcv, int lddcv1, int lddcv2) {
  
  int z0 = blockIdx.z * blockDim.z + threadIdx.z;
  int y0 = blockIdx.y * blockDim.y + threadIdx.y;
  int x0 = blockIdx.x * blockDim.x + threadIdx.x;

  for (int z = z0; y < nf; z += blockDim.z * gridDim.z) {
    for (int y = y0; y < nr; y += blockDim.y * gridDim.y) {
      for (int x = x0; x < nc; x += blockDim.x * gridDim.x) {
        dcv[get_idx(lddv1, lddv2, z, y, x)] = dv[get_idx(lddcv1, lddcv2, ifib[z], irow[y], icol[x])];
      }
    }
  }
}

mgard_cuda_ret 
org_to_pow2p1(int nfib, int nrow, int ncol,
              int nf,   int nr,   int nc,
              int * dirow,  int * dicol,
              double * dv,  int lddv1, int lddv2,
              double * dcv, int lddcv1, int lddcv2) {
  int B = 16;
  int total_thread_z = nf;
  int total_thread_y = nr;
  int total_thread_x = nc;
  int tbz = min(B, total_thread_z);
  int tby = min(B, total_thread_y);
  int tbx = min(B, total_thread_x);
  int gridz = ceil((float)total_thread_z/tbz);
  int gridy = ceil((float)total_thread_y/tby);
  int gridx = ceil((float)total_thread_x/tbx);
  dim3 threadsPerBlock(tbx, tby, tbz);
  dim3 blockPerGrid(gridx, gridy, gridz);

  // std::cout << "_copy_level_l_cuda" << std::endl;
  // std::cout << "thread block: " << tby << ", " << tbx << std::endl;
  // std::cout << "grid: " << gridy << ", " << gridx << std::endl;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  _org_to_pow2p1<<<blockPerGrid, threadsPerBlock>>>(nfib,  nrow,  ncol,
                                                    nf,    nr,    nc,
                                                    difib, dirow, dicol,
                                                    dv,    lddv1, lddv2,
                                                    dcv,   lddcv1, lddcv2);
  gpuErrchk(hipGetLastError ());

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  return mgard_cuda_ret(0, milliseconds/1000.0);
}


/* 2D Original to (2^k)+1 */
__global__ void
_org_to_pow2p1(int nrow,     int ncol,
               int nr,       int nc,
               int * irow,   int * icol,
               double * dv,  int lddv,
               double * dcv, int lddcv) {
  
  int y0 = blockIdx.y * blockDim.y + threadIdx.y;
  int x0 = blockIdx.x * blockDim.x + threadIdx.x;

  for (int y = y0; y < nr; y += blockDim.y * gridDim.y) {
    for (int x = x0; x < nc; x += blockDim.x * gridDim.x) {
      dcv[get_idx(lddcv, y, x)] = dv[get_idx(lddv, irow[y], icol[x])];
    }
  }
}


mgard_cuda_ret 
org_to_pow2p1(int nrow,     int ncol,
             int nr,       int nc,
             int * dirow,  int * dicol,
             double * dv,  int lddv,
             double * dcv, int lddcv) {

  int B = 16;
  int total_thread_y = nr;
  int total_thread_x = nc;
  int tby = min(B, total_thread_y);
  int tbx = min(B, total_thread_x);
  int gridy = ceil((float)total_thread_y/tby);
  int gridx = ceil((float)total_thread_x/tbx);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);

  // std::cout << "_copy_level_l_cuda" << std::endl;
  // std::cout << "thread block: " << tby << ", " << tbx << std::endl;
  // std::cout << "grid: " << gridy << ", " << gridx << std::endl;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  _org_to_pow2p1<<<blockPerGrid, threadsPerBlock>>>(nrow,  ncol,
                                                           nr,    nc,
                                                           dirow, dicol,
                                                           dv,    lddv,
                                                           dcv,   lddcv);

  gpuErrchk(hipGetLastError ());

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return mgard_cuda_ret(0, milliseconds/1000.0);
}


/* 1D Original to (2^k)+1 */
__global__ void
_org_to_pow2p1(int nrow,    int nr,       
               int * irow, 
               double * dv, double * dcv) {
  
  int x0 = blockIdx.x * blockDim.x + threadIdx.x;
  for (int x = x0; x < nr; x += blockDim.x * gridDim.x) {
    dcv[x] = dv[irow[x]];
  }
}


mgard_cuda_ret 
org_to_pow2p1(int nrow,    int nr,
              int * dirow, 
              double * dv, double * dcv) {

  int B = 16;
  int total_thread_y = 1;
  int total_thread_x = nr;
  int tby = 1;
  int tbx = min(B, total_thread_x);
  int gridy = ceil((float)total_thread_y/tby);
  int gridx = ceil((float)total_thread_x/tbx);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);

  // std::cout << "_copy_level_l_cuda" << std::endl;
  // std::cout << "thread block: " << tby << ", " << tbx << std::endl;
  // std::cout << "grid: " << gridy << ", " << gridx << std::endl;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  _org_to_pow2p1<<<blockPerGrid, threadsPerBlock>>>(nrow,  nr,    
                                                            dirow,
                                                            dv,    dcv);

  gpuErrchk(hipGetLastError ());

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return mgard_cuda_ret(0, milliseconds/1000.0);
}


/* 3D (2^k)+1 to original*/
__global__ void
_pow2p1_to_org(int nfib, int nrow,     int ncol,   
               int nf,   int nr,       int nc,     
               int ifib, int * irow,   int * icol, 
               double * dv,  int lddv1, int lddv2,
               double * dcv, int lddcv1, int lddcv2) {
  
  int z0 = blockIdx.z * blockDim.z + threadIdx.z;
  int y0 = blockIdx.y * blockDim.y + threadIdx.y;
  int x0 = blockIdx.x * blockDim.x + threadIdx.x;

  for (int z = z0; y < nf; z += blockDim.z * gridDim.z) {
    for (int y = y0; y < nr; y += blockDim.y * gridDim.y) {
      for (int x = x0; x < nc; x += blockDim.x * gridDim.x) {
        dcv[get_idx(lddv1, lddv2, z, y, x)] = dv[get_idx(lddcv1, lddcv2, ifib[z], irow[y], icol[x])];
      }
    }
  }
}

mgard_cuda_ret 
pow2p1_to_org(int nfib, int nrow,     int ncol,   
              int nf,   int nr,       int nc,     
              int ifib, int * irow,   int * icol, 
              double * dv,  int lddv1, int lddv2,
               double * dcv, int lddcv1, int lddcv2) {

  int B = 16;
  int total_thread_z = nf;
  int total_thread_y = nr;
  int total_thread_x = nc;
  int tbz = min(B, total_thread_z);
  int tby = min(B, total_thread_y);
  int tbx = min(B, total_thread_x);
  int gridz = ceil((float)total_thread_z/tbz);
  int gridy = ceil((float)total_thread_y/tby);
  int gridx = ceil((float)total_thread_x/tbx);
  dim3 threadsPerBlock(tbx, tby, tbz);
  dim3 blockPerGrid(gridx, gridy, gridz);

  // std::cout << "_copy_level_l_cuda" << std::endl;
  // std::cout << "thread block: " << tby << ", " << tbx << std::endl;
  // std::cout << "grid: " << gridy << ", " << gridx << std::endl;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  _pow2p1_to_org<<<blockPerGrid, threadsPerBlock>>>(nfib,  nrow,  ncol,
                                                    nf,    nr,    nc,
                                                    difib, dirow, dicol,
                                                    dv,    lddv1, lddv2,
                                                    dcv,   lddcv1, lddcv2);
  gpuErrchk(hipGetLastError ());

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  return mgard_cuda_ret(0, milliseconds/1000.0);
}




/* 2D (2^k)+1 to original*/
__global__ void
_pow2p1_to_org(int nrow,     int ncol,
               int nr,       int nc,
               int * irow,   int * icol,
               double * dcv,  int lddcv,
               double * dv, int lddv) {
  
  int y0 = blockIdx.y * blockDim.y + threadIdx.y;
  int x0 = blockIdx.x * blockDim.x + threadIdx.x;
  for (int y = y0; y < nr; y += blockDim.y * gridDim.y) {
    for (int x = x0; x < nc; x += blockDim.x * gridDim.x) {
       dv[get_idx(lddv, irow[y], icol[x])] = dcv[get_idx(lddcv, y, x)];
    }
  }
}

mgard_cuda_ret 
pow2p1_to_org(int nrow,     int ncol,
                       int nr,       int nc,
                       int * dirow,  int * dicol,
                       double * dcv,  int lddcv,
                       double * dv, int lddv) {

  int B = 16;
  int total_thread_y = nr;
  int total_thread_x = nc;
  int tby = min(B, total_thread_y);
  int tbx = min(B, total_thread_x);
  int gridy = ceil((float)total_thread_y/tby);
  int gridx = ceil((float)total_thread_x/tbx);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);

  // std::cout << "_copy_level_l_cuda" << std::endl;
  // std::cout << "thread block: " << tby << ", " << tbx << std::endl;
  // std::cout << "grid: " << gridy << ", " << gridx << std::endl;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  _pow2p1_to_org<<<blockPerGrid, threadsPerBlock>>>(nrow,  ncol,
                                                             nr,    nc,
                                                             dirow, dicol,
                                                             dcv,   lddcv,
                                                             dv,    lddv);

  gpuErrchk(hipGetLastError ());

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return mgard_cuda_ret(0, milliseconds/1000.0);
}


/* 1D (2^k)+1 to original */
__global__ void
_pow2p1_to_org(int nrow, int nr, 
               int * irow,   
               double * dcv, double * dv) {
  int x0 = blockIdx.x * blockDim.x + threadIdx.x;
  for (int x = x0; x < nr; x += blockDim.x * gridDim.x) {
     dv[irow[x]] = dcv[x];
  }
}

mgard_cuda_ret 
pow2p1_to_org(int nrow, int nr,      
              int * dirow,  
              double * dcv, double * dv) {

  int B = 16;
  int total_thread_y = 1;
  int total_thread_x = nr;
  int tby = 1;
  int tbx = min(B, total_thread_x);
  int gridy = ceil((float)total_thread_y/tby);
  int gridx = ceil((float)total_thread_x/tbx);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);

  // std::cout << "_copy_level_l_cuda" << std::endl;
  // std::cout << "thread block: " << tby << ", " << tbx << std::endl;
  // std::cout << "grid: " << gridy << ", " << gridx << std::endl;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  _pow2p1_to_org<<<blockPerGrid, threadsPerBlock>>>(nrow, nr,   
                                                    dirow, 
                                                    dcv, dv);

  gpuErrchk(hipGetLastError ());

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return mgard_cuda_ret(0, milliseconds/1000.0);
}


/* 3D (2^k)+1 to compact */
__global__ void 
_pow2p1_to_cpt(int nfib, int nrow, int ncol, 
               int fib_stride, int row_stride, int col_stride,
               double * dv, int lddv1, int lddv2,
               double * dcv, int lddcv1, int lddcv2) {
  int z0 = blockIdx.z * blockDim.z + threadIdx.z;
  int y0 = blockIdx.y * blockDim.y + threadIdx.y;
  int x0 = blockIdx.x * blockDim.x + threadIdx.x;
  for (int z = z0; z * fib_stride < nfib; z += blockDim.z * gridDim.z) {
    for (int y = y0; y * row_stride < nrow; y += blockDim.y * gridDim.y) {
      for (int x = x0; x * col_stride < ncol; x += blockDim.x * gridDim.x) {
        int z_strided = z * fib_stride;
        int y_strided = y * row_stride;
        int x_strided = x * col_stride;
        
        dcv[get_idx(lddcv1, lddcv2, z, y, x)] = dv[get_idx(lddv1, lddv2, z_strided, y_strided, x_strided)];

      }
    }
  }
}

mgard_cuda_ret 
pow2p1_to_cpt(int nfib, int nrow,      int ncol, 
              int fib_stride, int row_stride, int col_stride,
              double * dv,    int lddv1, int lddv2,
              double * dcv,   int lddcv1, int lddcv2) {
  int B = 16;
  int total_thread_z = ceil((float)nfib/fib_stride);
  int total_thread_y = ceil((float)nrow/row_stride);
  int total_thread_x = ceil((float)ncol/col_stride);
  int tbz = min(B, total_thread_z);
  int tby = min(B, total_thread_y);
  int tbx = min(B, total_thread_x);
  int gridz = ceil((float)total_thread_z/tbz);
  int gridy = ceil((float)total_thread_y/tby);
  int gridx = ceil((float)total_thread_x/tbx);
  dim3 threadsPerBlock(tbx, tby, tbz);
  dim3 blockPerGrid(gridx, gridy, gridz);

  //std::cout << "thread block: " << tbx << ", " << tby <<std::endl;
  //std::cout << "grid: " << gridx << ", " << gridy <<std::endl;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  _pow2p1_to_cpt<<<blockPerGrid, threadsPerBlock>>>(nfib, nrow, ncol,
                                                    fib_stride, row_stride, col_stride,
                                                    dv, lddv1, lddv2, 
                                                    dcv, lddcv1, lddcv2);
  gpuErrchk(hipGetLastError ());

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return mgard_cuda_ret(0, milliseconds/1000.0);
}



/* 2D (2^k)+1 to compact */
__global__ void 
_pow2p1_to_cpt(int nrow,           int ncol, 
               int row_stride,      int col_stride,
               double * dv,         int lddv, 
               double * dcv,        int lddcv) {
  int x0 = blockIdx.x * blockDim.x + threadIdx.x;
  int y0 = blockIdx.y * blockDim.y + threadIdx.y;
  for (int y = y0; y * row_stride < nrow; y += blockDim.y * gridDim.y) {
    for (int x = x0; x * col_stride < ncol; x += blockDim.x * gridDim.x) {
      int x_strided = x * col_stride;
      int y_strided = y * row_stride;
      dcv[get_idx(lddcv, y, x)] = dv[get_idx(lddv, y_strided, x_strided)];

    }
  }
}

mgard_cuda_ret 
pow2p1_to_cpt(int nrow,      int ncol, 
              int row_stride, int col_stride,
              double * dv,    int lddv, 
              double * dcv,   int lddcv) {
  int B = 16;
  int total_thread_y = ceil((float)nrow/row_stride);
  int total_thread_x = ceil((float)ncol/col_stride);
  int tby = min(B, total_thread_y);
  int tbx = min(B, total_thread_x);
  int gridy = ceil((float)total_thread_y/tby);
  int gridx = ceil((float)total_thread_x/tbx);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);

  //std::cout << "thread block: " << tbx << ", " << tby <<std::endl;
  //std::cout << "grid: " << gridx << ", " << gridy <<std::endl;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  _pow2p1_to_cpt<<<blockPerGrid, threadsPerBlock>>>(nrow,      ncol,
                                                    row_stride, col_stride,
                                                    dv,         lddv, 
                                                    dcv,        lddcv);
  gpuErrchk(hipGetLastError ());

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return mgard_cuda_ret(0, milliseconds/1000.0);
}


/* 1D (2^k)+1 to compact */
__global__ void 
_pow2p1_to_cpt(int nrow, int row_stride,     
               double * dv,double * dcv) {
  int x0 = blockIdx.x * blockDim.x + threadIdx.x;
  for (int x = x0; x * row_stride < nrow; x += blockDim.x * gridDim.x) {
    int x_strided = x * row_stride;
    dcv[x] = dv[x_strided];
  }
}

mgard_cuda_ret 
pow2p1_to_cpt(int nrow,  int row_stride, 
              double * dv, double * dcv) {
  int B = 16;
  int total_thread_y = 1;
  int total_thread_x = ceil((float)nrow/row_stride);
  int tby = 1;
  int tbx = min(B, total_thread_x);
  int gridy = ceil((float)total_thread_y/tby);
  int gridx = ceil((float)total_thread_x/tbx);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);

  //std::cout << "thread block: " << tbx << ", " << tby <<std::endl;
  //std::cout << "grid: " << gridx << ", " << gridy <<std::endl;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  _pow2p1_to_cpt<<<blockPerGrid, threadsPerBlock>>>(nrow, row_stride, 
                                                    dv, dcv);
  gpuErrchk(hipGetLastError ());

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return mgard_cuda_ret(0, milliseconds/1000.0);
}


/* 3D compact to (2^k)+1*/
__global__ void 
_cpt_to_pow2p1(int nfib, int nrow, int ncol,
              int fib_stride, int row_stride, int col_stride,
              double * dcv, int lddcv1, int lddcv2,
              double * dv, int lddv1, int lddv2) {
  int z0 = blockIdx.z * blockDim.z + threadIdx.z;
  int y0 = blockIdx.y * blockDim.y + threadIdx.y;
  int x0 = blockIdx.x * blockDim.x + threadIdx.x;
  for (int z = z0; z * fib_stride < nfib; z += blockDim.z * gridDim.z) {
    for (int y = y0; y * row_stride < nrow; y += blockDim.y * gridDim.y) {
      for (int x = x0; x * col_stride < ncol; x += blockDim.x * gridDim.x) {
        int z_strided = z * fib_stride;
        int y_strided = y * row_stride;
        int x_strided = x * col_stride;
        dv[get_idx(lddv1, lddv2, z_strided, y_strided, x_strided)] = dcv[get_idx(lddcv1, lddcv2, z, y, x)];
      }
    }
  }
}


mgard_cuda_ret
cpt_to_pow2p1(int nfib, int nrow, int ncol,
              int fib_stride, int row_stride, int col_stride,
              double * dcv, int lddcv1, int lddcv2,
              double * dv, int lddv1, int lddv2) {
  int B = 16;
  int total_thread_z = ceil((float)nfib/fib_stride);
  int total_thread_y = ceil((float)ncol/col_stride);
  int total_thread_x = ceil((float)nrow/row_stride);
  int tbz = min(B, total_thread_z);
  int tby = min(B, total_thread_y);
  int tbx = min(B, total_thread_x);
  int gridz = ceil((float)total_thread_z/tbz);
  int gridy = ceil((float)total_thread_y/tby);
  int gridx = ceil((float)total_thread_x/tbx);
  dim3 threadsPerBlock(tbx, tby, tbz);
  dim3 blockPerGrid(gridx, gridy, gridz);

  //std::cout << "thread block: " << tbx << ", " << tby <<std::endl;
  //std::cout << "grid: " << gridx << ", " << gridy <<std::endl;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  _cpt_to_pow2p1<<<blockPerGrid, threadsPerBlock>>>(nfib, nrow, ncol,
                                                    fib_stride, row_stride, col_stride, 
                                                    dcv, lddcv1, lddcv2,
                                                    dv, lddv1, lddv2);
  
  gpuErrchk(hipGetLastError ());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  return mgard_cuda_ret(0, milliseconds/1000.0);

}




/* 2D compact to (2^k)+1*/
__global__ void 
_cpt_to_pow2p1(int nrow,     int ncol,
              int row_stride, int col_stride,
              double * dcv,   int lddcv,
              double * dv,    int lddv) {
  int x0 = blockIdx.x * blockDim.x + threadIdx.x;
  int y0 = blockIdx.y * blockDim.y + threadIdx.y;
  for (int y = y0; y * row_stride < nrow; y += blockDim.y * gridDim.y) {
    for (int x = x0; x * col_stride < ncol; x += blockDim.x * gridDim.x) {
      int x_strided = x * col_stride;
      int y_strided = y * row_stride;
      dv[get_idx(lddv, y_strided, x_strided)] = dcv[get_idx(lddcv, y, x)];
    }
  }
}


mgard_cuda_ret
cpt_to_pow2p1(int nrow, int ncol, 
              int row_stride, int col_stride, 
              double * dcv, int lddcv,
              double * dv, int lddv) {
  int B = 16;
  int total_thread_x = ceil((float)nrow/row_stride);
  int total_thread_y = ceil((float)ncol/col_stride);
  int tbx = min(B, total_thread_x);
  int tby = min(B, total_thread_y);
  int gridx = ceil((float)total_thread_x/tbx);
  int gridy = ceil((float)total_thread_y/tby);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);

  //std::cout << "thread block: " << tbx << ", " << tby <<std::endl;
  //std::cout << "grid: " << gridx << ", " << gridy <<std::endl;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  _cpt_to_pow2p1<<<blockPerGrid, threadsPerBlock>>>(nrow, ncol,
                                                    row_stride, col_stride, 
                                                    dcv, lddcv,
                                                    dv, lddv);
  
  gpuErrchk(hipGetLastError ());
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  return mgard_cuda_ret(0, milliseconds/1000.0);

}


/* 1D compact to (2^k)+1*/
__global__ void 
_cpt_to_pow2p1(int nrow, int row_stride, 
              double * dcv, double * dv) {
  int x0 = blockIdx.x * blockDim.x + threadIdx.x;
  for (int x = x0; x * row_stride < nrow; x += blockDim.x * gridDim.x) {
    int x_strided = x * row_stride;
    dv[x_strided] = dcv[x];
  }
}



mgard_cuda_ret
cpt_to_pow2p1(int nrow, int row_stride, 
              double * dcv, double * dv) {
    int B = 16;
    int total_thread_x = ceil((float)nrow/row_stride);
    int total_thread_y = 1;
    int tbx = min(B, total_thread_x);
    int tby = 1;
    int gridx = ceil((float)total_thread_x/tbx);
    int gridy = ceil((float)total_thread_y/tby);
    dim3 threadsPerBlock(tbx, tby);
    dim3 blockPerGrid(gridx, gridy);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    _cpt_to_pow2p1<<<blockPerGrid, threadsPerBlock>>>(nrow, row_stride, 
                                                      dcv,  dv);
    
    gpuErrchk(hipGetLastError ());
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return mgard_cuda_ret(0, milliseconds/1000.0);
}

__device__ double
_dist(double * dcoord, int x, int y) {
  return dcoord[y] - dcoord[x];
}


__global__ void
_calc_cpt_dist(int n, int stride,
               double * dcoord, double * ddist) {
  extern __shared__ double sm[]; //size = blockDim.x + 1

  int x0 = blockIdx.x * blockDim.x + threadIdx.x;
  int x0_sm = threadIdx.x;
  double dist;
  for (int x = x0; x * stride < n - 1; x += blockDim.x * gridDim.x) {
    // Load coordinates
    sm[x0_sm] = dcoord[x * stride];
    // printf("block %d thread %d load[%d] %f\n", blockIdx.x, threadIdx.x, x, dcoord[x * stride]);
    if (x0_sm == 0){
      sm[blockDim.x] = dcoord[(x + blockDim.x) * stride];
    }
    __syncthreads();

    // Compute distance
    dist = _dist(sm, x0_sm, x0_sm+1);
    __syncthreads();
    ddist[x] = dist;
    __syncthreads();
  }
}


mgard_cuda_ret
calc_cpt_dist(int nrow, int row_stride, 
              double * dcoord, double * ddist) {
    int B = 16;
    int total_thread_x = ceil((float)nrow/row_stride) - 1;
    int total_thread_y = 1;
    int tbx = min(B, total_thread_x);
    int tby = 1;
    int gridx = ceil((float)total_thread_x/tbx);
    int gridy = ceil((float)total_thread_y/tby);
    dim3 threadsPerBlock(tbx, tby);
    dim3 blockPerGrid(gridx, gridy);
    size_t sm_size = (tbx + 1) * sizeof(double);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    _calc_cpt_dist<<<blockPerGrid, threadsPerBlock, sm_size>>>(nrow, row_stride, 
                                                               dcoord,  ddist);
    
    gpuErrchk(hipGetLastError ());
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return mgard_cuda_ret(0, milliseconds/1000.0);
}