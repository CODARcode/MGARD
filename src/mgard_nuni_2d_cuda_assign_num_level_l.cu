#include "hip/hip_runtime.h"
#include "mgard_nuni.h"
#include "mgard.h"
#include "mgard_nuni_2d_cuda.h"
#include "mgard_cuda_helper.h"
#include "mgard_cuda_compact_helper.h"
#include "mgard_cuda_helper_internal.h"
#include <fstream>
#include <cmath>

namespace mgard_2d {
namespace mgard_gen {

template <typename T>
__global__ void 
_assign_num_level_l_cuda(int nrow,           int ncol,
                         int nr,             int nc,
                         int row_stride,     int col_stride,
                         int * dirow,        int * dicol,
                         T * dv,        int lddv,
                         T num) {
  
  int y0 = (blockIdx.y * blockDim.y + threadIdx.y) * row_stride;
  int x0 = (blockIdx.x * blockDim.x + threadIdx.x) * col_stride;

  for (int y = y0; y < nr; y += blockDim.y * gridDim.y * row_stride) {
    for (int x = x0; x < nc; x += blockDim.x * gridDim.x * col_stride) {
      dv[get_idx(lddv, dirow[y], dicol[x])] = num;
    }
  }
}

template <typename T>
mgard_cuda_ret 
assign_num_level_l_cuda(int nrow,           int ncol,
                        int nr,             int nc,
                        int row_stride,     int col_stride,
                        int * dirow,        int * dicol,
                        T * dv,        int lddv,
                        T num,
                        int B, 
                        mgard_cuda_handle & handle, 
                        int queue_idx, bool profile) {

  hipEvent_t start, stop;
  float milliseconds = 0;
  hipStream_t stream = *(hipStream_t *)handle.get(queue_idx);

  int total_thread_y = ceil((float)nr/(row_stride));
  int total_thread_x = ceil((float)nc/(col_stride));
  int tby = min(B, total_thread_y);
  int tbx = min(B, total_thread_x);
  int gridy = ceil((float)total_thread_y/tby);
  int gridx = ceil((float)total_thread_x/tbx);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);

  if (profile) {
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));
    gpuErrchk(hipEventRecord(start, stream));
  }

  _assign_num_level_l_cuda<<<blockPerGrid, threadsPerBlock,
                             0, stream>>>(nrow,       ncol,
                                          nr,         nc,
                                          row_stride, col_stride,
                                          dirow,      dicol,
                                          dv,         lddv,
                                          num);
  gpuErrchk(hipGetLastError ());

  if (profile) {
    gpuErrchk(hipEventRecord(stop, stream));
    gpuErrchk(hipEventSynchronize(stop));
    gpuErrchk(hipEventElapsedTime(&milliseconds, start, stop));
    gpuErrchk(hipEventDestroy(start));
    gpuErrchk(hipEventDestroy(stop));
  }

  return mgard_cuda_ret(0, milliseconds/1000.0);

}

template <typename T>
__global__ void 
_assign_num_level_l_cuda_l2_sm(int nr,             int nc,
                               int row_stride,     int col_stride,
                               T * dv,        int lddv,
                               T num) {
  
  int y0 = (blockIdx.y * blockDim.y + threadIdx.y) * row_stride;
  int x0 = (blockIdx.x * blockDim.x + threadIdx.x) * col_stride;

  for (int y = y0; y < nr; y += blockDim.y * gridDim.y * row_stride) {
    for (int x = x0; x < nc; x += blockDim.x * gridDim.x * col_stride) {
      dv[get_idx(lddv, y, x)] = num;
    }
  }
}

template <typename T>
mgard_cuda_ret 
assign_num_level_l_cuda_l2_sm(int nr,             int nc,
                              int row_stride,     int col_stride,
                              T * dv,        int lddv,
                              T num,
                              int B, mgard_cuda_handle & handle, 
                              int queue_idx, bool profile) {
  hipEvent_t start, stop;
  float milliseconds = 0;
  hipStream_t stream = *(hipStream_t *)handle.get(queue_idx);

  int total_thread_y = ceil((float)nr/(row_stride));
  int total_thread_x = ceil((float)nc/(col_stride));
  int tby = min(B, total_thread_y);
  int tbx = min(B, total_thread_x);
  int gridy = ceil((float)total_thread_y/tby);
  int gridx = ceil((float)total_thread_x/tbx);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);

  if (profile) {
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));
    gpuErrchk(hipEventRecord(start, stream));
  }

  _assign_num_level_l_cuda_l2_sm<<<blockPerGrid, threadsPerBlock,
                                   0, stream>>>(nr,         nc,
                                                row_stride, col_stride,
                                                dv,         lddv,
                                                num);
  gpuErrchk(hipGetLastError ());

  if (profile) {
    gpuErrchk(hipEventRecord(stop, stream));
    gpuErrchk(hipEventSynchronize(stop));
    gpuErrchk(hipEventElapsedTime(&milliseconds, start, stop));
    gpuErrchk(hipEventDestroy(start));
    gpuErrchk(hipEventDestroy(stop));
  }

  return mgard_cuda_ret(0, milliseconds/1000.0);

}

template mgard_cuda_ret 
assign_num_level_l_cuda<double>(int nrow,           int ncol,
                        int nr,             int nc,
                        int row_stride,     int col_stride,
                        int * dirow,        int * dicol,
                        double * dv,        int lddv,
                        double num,
                        int B, mgard_cuda_handle & handle, 
                        int queue_idx, bool profile);

template mgard_cuda_ret 
assign_num_level_l_cuda<float>(int nrow,           int ncol,
                        int nr,             int nc,
                        int row_stride,     int col_stride,
                        int * dirow,        int * dicol,
                        float * dv,        int lddv,
                        float num,
                        int B, mgard_cuda_handle & handle, 
                        int queue_idx, bool profile);

template mgard_cuda_ret 
assign_num_level_l_cuda_l2_sm<double>(int nr,             int nc,
                                      int row_stride,     int col_stride,
                                      double * dv,        int lddv,
                                      double num,
                                      int B, mgard_cuda_handle & handle, 
                                      int queue_idx, bool profile);
template mgard_cuda_ret 
assign_num_level_l_cuda_l2_sm<float>(int nr,             int nc,
                                     int row_stride,     int col_stride,
                                     float * dv,        int lddv,
                                     float num,
                                     int B, mgard_cuda_handle & handle, 
                                     int queue_idx, bool profile);




}
}