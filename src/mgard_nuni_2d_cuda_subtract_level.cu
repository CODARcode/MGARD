#include "hip/hip_runtime.h"
#include "mgard_nuni.h"
#include "mgard.h"
#include "mgard_nuni_2d_cuda.h"
#include "mgard_cuda_helper.h"
#include "mgard_cuda_helper_internal.h"
#include <fstream>

namespace mgard_2d {
namespace mgard_cannon {

template <typename T>
__global__ void 
_subtract_level_cuda(int nrow,       int ncol, 
                      int row_stride, int col_stride,
                      T * dv,    int lddv, 
                      T * dwork, int lddwork) {
    //int stride = pow (2, l); // current stride
    //int Cstride = stride * 2; // coarser stride
    int idx_x = (blockIdx.x * blockDim.x + threadIdx.x) * col_stride;
    int idx_y = (blockIdx.y * blockDim.y + threadIdx.y) * row_stride;
    //printf("x = %d, y = %d, stride = %d \n", x,y,stride);
    for (int y = idx_y; y < nrow; y += blockDim.y * gridDim.y * row_stride) {
      for (int x = idx_x; x < ncol; x += blockDim.x * gridDim.x * col_stride) {
        dv[get_idx(lddv, y, x)] -= dwork[get_idx(lddwork, y, x)];
        //printf("x = %d, y = %d, stride = %d, v = %f \n", x,y,stride, work[get_idx(ncol, x, y)]);
        //y += blockDim.y * gridDim.y * stride;
      }
        //x += blockDim.x * gridDim.x * stride;
    }
}

template <typename T>
mgard_cuda_ret 
subtract_level_cuda(int nrow,       int ncol, 
                    int row_stride, int col_stride,
                    T * dv,    int lddv, 
                    T * dwork, int lddwork,
                    int B, mgard_cuda_handle & handle, 
                    int queue_idx, bool profile) {
  hipEvent_t start, stop;
  float milliseconds = 0;
  hipStream_t stream = *(hipStream_t *)handle.get(queue_idx);

  int total_thread_x = ncol/col_stride;
  int total_thread_y = nrow/row_stride;
  int tbx = min(B, total_thread_x);
  int tby = min(B, total_thread_y);
  int gridx = ceil((float)total_thread_x/tbx);
  int gridy = ceil((float)total_thread_y/tby);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);

  if (profile) {
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));
    gpuErrchk(hipEventRecord(start, stream));
  }

  _subtract_level_cuda<<<blockPerGrid, threadsPerBlock,
                         0, stream>>>(nrow,       ncol,
                                      row_stride, col_stride, 
                                      dv,         lddv,
                                      dwork,      lddwork);
  gpuErrchk(hipGetLastError ()); 

  if (profile) {
    gpuErrchk(hipEventRecord(stop, stream));
    gpuErrchk(hipEventSynchronize(stop));
    gpuErrchk(hipEventElapsedTime(&milliseconds, start, stop));
    gpuErrchk(hipEventDestroy(start));
    gpuErrchk(hipEventDestroy(stop));
  }

  return mgard_cuda_ret(0, milliseconds/1000.0);
}

template mgard_cuda_ret 
subtract_level_cuda<double>(int nrow,       int ncol, 
                            int row_stride, int col_stride,
                            double * dv,    int lddv, 
                            double * dwork, int lddwork,
                            int B, mgard_cuda_handle & handle, 
                            int queue_idx, bool profile);
template mgard_cuda_ret 
subtract_level_cuda<float>(int nrow,       int ncol, 
                            int row_stride, int col_stride,
                            float * dv,    int lddv, 
                            float * dwork, int lddwork,
                            int B, mgard_cuda_handle & handle, 
                            int queue_idx, bool profile);

}
}
