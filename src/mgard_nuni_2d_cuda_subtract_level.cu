#include "hip/hip_runtime.h"
#include "mgard_nuni.h"
#include "mgard.h"
#include "mgard_nuni_2d_cuda.h"
#include "mgard_cuda_helper.h"
#include "mgard_cuda_helper_internal.h"
#include <fstream>

namespace mgard_2d {
namespace mgard_cannon {

template <typename T>
__global__ void 
_subtract_level_cuda(int nrow,       int ncol, 
                      int row_stride, int col_stride,
                      T * dv,    int lddv, 
                      T * dwork, int lddwork) {
    //int stride = pow (2, l); // current stride
    //int Cstride = stride * 2; // coarser stride
    int idx_x = (blockIdx.x * blockDim.x + threadIdx.x) * col_stride;
    int idx_y = (blockIdx.y * blockDim.y + threadIdx.y) * row_stride;
    //printf("x = %d, y = %d, stride = %d \n", x,y,stride);
    for (int y = idx_y; y < nrow; y += blockDim.y * gridDim.y * row_stride) {
      for (int x = idx_x; x < ncol; x += blockDim.x * gridDim.x * col_stride) {
        dv[get_idx(lddv, y, x)] -= dwork[get_idx(lddwork, y, x)];
        //printf("x = %d, y = %d, stride = %d, v = %f \n", x,y,stride, work[get_idx(ncol, x, y)]);
        //y += blockDim.y * gridDim.y * stride;
      }
        //x += blockDim.x * gridDim.x * stride;
    }
}

template <typename T>
mgard_cuda_ret 
subtract_level_cuda(int nrow,       int ncol, 
                    int row_stride, int col_stride,
                    T * dv,    int lddv, 
                    T * dwork, int lddwork) {
  int B = 16;
  int total_thread_x = ncol/col_stride;
  int total_thread_y = nrow/row_stride;
  int tbx = min(B, total_thread_x);
  int tby = min(B, total_thread_y);
  int gridx = ceil((float)total_thread_x/tbx);
  int gridy = ceil((float)total_thread_y/tby);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);

  //std::cout << "thread block: " << tbx << ", " << tby <<std::endl;
  //std::cout << "grid: " << gridx << ", " << gridy <<std::endl;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  _subtract_level_cuda<<<blockPerGrid, threadsPerBlock>>>(nrow,       ncol,
                                                           row_stride, col_stride, 
                                                           dv,         lddv,
                                                           dwork,      lddwork);


  gpuErrchk(hipGetLastError ()); 

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return mgard_cuda_ret(0, milliseconds/1000.0);
}

template mgard_cuda_ret 
subtract_level_cuda<double>(int nrow,       int ncol, 
                            int row_stride, int col_stride,
                            double * dv,    int lddv, 
                            double * dwork, int lddwork);
template mgard_cuda_ret 
subtract_level_cuda<float>(int nrow,       int ncol, 
                            int row_stride, int col_stride,
                            float * dv,    int lddv, 
                            float * dwork, int lddwork);

}
}
