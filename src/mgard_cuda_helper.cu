#include "hip/hip_runtime.h"
#include <iomanip> 
#include <iostream>
#include "mgard_cuda_helper.h"
#include "mgard_cuda_helper_internal.h"


void print_matrix(int nrow, int ncol, double * v, int ldv) {
    //std::cout << std::setw(10);
    //std::cout << std::setprecision(2) << std::fixed;
    for (int i = 0; i < nrow; i++) {
        for (int j = 0; j < ncol; j++) {
            std::cout <<std::setw(8) << std::setprecision(4) << std::fixed <<  v[ldv*i + j]<<", ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

void compare_matrix(int nrow, int ncol, 
                    double * v1, int ldv1, 
                    double * v2, int ldv2) {
    //std::cout << std::setw(10);
    //std::cout << std::setprecision(2) << std::fixed;
    for (int i = 0; i < nrow; i++) {
        for (int j = 0; j < ncol; j++) {
            if (abs(v1[ldv1*i + j] - v2[ldv2*i + j]) > 0.001){
                std::cout << "Diff at (" << i << ", " << j << ")" << std::endl; 
            }
        }
    }   
}






void cudaMallocHelper(void **  devPtr, size_t  size) {
    gpuErrchk(hipMalloc(devPtr, size));
}

void cudaMemcpyHelper(void * dst, const void * src, size_t count, enum copy_type kind){
    enum hipMemcpyKind cuda_copy_type;
    switch (kind)
    {
        case H2D :  cuda_copy_type = hipMemcpyHostToDevice; break;
        case D2H :  cuda_copy_type = hipMemcpyDeviceToHost; break;
    }
    gpuErrchk(hipMemcpy(dst, src, count, cuda_copy_type));
}

void cudaMallocPitchHelper(void ** devPtr, size_t * pitch, size_t width, size_t height) {
    gpuErrchk(hipMallocPitch(devPtr, pitch, width, height));
}

void cudaMemcpy2DHelper(void * dst, size_t dpitch, 
                        const void * src, size_t spitch, 
                        size_t width, size_t height,
                        enum copy_type kind) {
    enum hipMemcpyKind cuda_copy_type;
    switch (kind)
    {
        case H2D :  cuda_copy_type = hipMemcpyHostToDevice; break;
        case D2H :  cuda_copy_type = hipMemcpyDeviceToHost; break;
    }
    gpuErrchk(hipMemcpy2D(dst, dpitch, 
                 src, spitch,
                 width, height, 
                 cuda_copy_type));
}

void cudaFreeHelper(void * devPtr) {
    gpuErrchk(hipFree(devPtr));
}

void cudaMemsetHelper(void * devPtr, int value, size_t count) {
    gpuErrchk(hipMemset(devPtr, value, count));
}

void cudaMemset2DHelper(void * devPtr,  size_t  pitch, int value, size_t width, size_t height) {
    gpuErrchk(hipMemset2D(devPtr, pitch, value, width, height));
}