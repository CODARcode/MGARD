#include "hip/hip_runtime.h"
#include <iomanip> 
#include <iostream>
#include "mgard_cuda_helper.h"
#include "mgard_cuda_helper_internal.h"


mgard_cuda_handle::mgard_cuda_handle (int num_of_queues){
  this->num_of_queues = num_of_queues;
  hipStream_t * ptr = new hipStream_t[num_of_queues];

  for (int i = 0; i < this->num_of_queues; i++) {
    gpuErrchk(hipStreamCreate(ptr+i));
    // std::cout << "created a stream\n";
  }

  this->queues = (void *)ptr;

}

void * mgard_cuda_handle::get(int i) {
  hipStream_t * ptr = (hipStream_t *)(this->queues);
  return (void *)(ptr+i);
}

void mgard_cuda_handle::sync(int i) {
  hipStream_t * ptr = (hipStream_t *)(this->queues);
  gpuErrchk(hipStreamSynchronize(ptr[i]));
}

void mgard_cuda_handle::sync_all() {
  hipStream_t * ptr = (hipStream_t *)(this->queues);
  for (int i = 0; i < this->num_of_queues; i++) {
    gpuErrchk(hipStreamSynchronize(ptr[i]));
  }
}

void mgard_cuda_handle::destory_all() {
  hipStream_t * ptr = (hipStream_t *)(this->queues);
  for (int i = 0; i < this->num_of_queues; i++) {
    gpuErrchk(hipStreamDestroy(ptr[i]));
  }
}




// print 2D CPU
template <typename T>
void print_matrix(int nrow, int ncol, T * v, int ldv) {
  //std::cout << std::setw(10);
  //std::cout << std::setprecision(2) << std::fixed;
  for (int i = 0; i < nrow; i++) {
    for (int j = 0; j < ncol; j++) {
        std::cout <<std::setw(9) << std::setprecision(6) << std::fixed <<  v[ldv*i + j]<<", ";
    }
    std::cout << std::endl;
  }
  std::cout << std::endl;
}

template void print_matrix<double>(int nrow, int ncol, double * v, int ldv);
template void print_matrix<float>(int nrow, int ncol, float * v, int ldv);
template void print_matrix<int>(int nrow, int ncol, int * v, int ldv);


// print 2D GPU
template <typename T>
void print_matrix_cuda(int nrow, int ncol, T * dv, int lddv) {
  //std::cout << std::setw(10);
  //std::cout << std::setprecision(2) << std::fixed;
  mgard_cuda_handle tmp_handle(1);
  int queue_idx = 0;
  bool profile = false;
  T * v = new T[nrow * ncol];
  cudaMemcpy2DAsyncHelper(v, ncol  * sizeof(T), 
                    dv, lddv * sizeof(T),  
                    ncol * sizeof(T), nrow, 
                    D2H, tmp_handle, queue_idx, profile);
  tmp_handle.sync(queue_idx);
  print_matrix(nrow, ncol, v, ncol); 
  delete [] v;
  tmp_handle.destory_all();
}
template void print_matrix_cuda<double>(int nrow, int ncol, double * dv, int lddv);
template void print_matrix_cuda<float>(int nrow, int ncol, float * dv, int lddv);
template void print_matrix_cuda<int>(int nrow, int ncol, int * dv, int lddv);

// print 3D GPU
template <typename T>
void print_matrix_cuda(int nrow, int ncol, int nfib, T * dv, int lddv1, int lddv2, int sizex) {
  //std::cout << std::setw(10);
  //std::cout << std::setprecision(2) << std::fixed;
  mgard_cuda_handle tmp_handle(1);
  int queue_idx = 0;
  bool profile = false;

  T * v = new T[nrow * ncol * nfib];
  cudaMemcpy3DAsyncHelper(v, nfib  * sizeof(T), nfib * sizeof(T), ncol,
                     dv, lddv1 * sizeof(T), sizex * sizeof(T), lddv2,
                     nfib * sizeof(T), ncol, nrow, 
                     D2H, tmp_handle, queue_idx, profile);
  tmp_handle.sync(queue_idx);
  print_matrix(nrow, ncol, nfib, v, nfib, ncol); 
  delete [] v;
  tmp_handle.destory_all();
}

template void print_matrix_cuda<double>(int nrow, int ncol, int nfib, double * dv, int lddv1, int lddv2, int sizex);
template void print_matrix_cuda<float>(int nrow, int ncol, int nfib, float * dv, int lddv1, int lddv2, int sizex);

// print 3D CPU
template <typename T>
void print_matrix(int nrow, int ncol, int nfib, T * v, int ldv1, int ldv2) {
  //std::cout << std::setw(10);
  //std::cout << std::setprecision(2) << std::fixed;
  for (int i = 0; i < nrow; i++) {
    std::cout << "[ nrow = " << i << " ]\n";
    print_matrix(ncol, nfib, v + i * ldv1 * ldv2, ldv1);
    std::cout << std::endl;
  }
}

template void print_matrix<double>(int nrow, int ncol, int nfib, double * v, int ldv1, int ldv2);
template void print_matrix<float>(int nrow, int ncol, int nfib, float * v, int ldv1, int ldv2);


// compare 2D CPU
template <typename T>
bool compare_matrix(int nrow, int ncol, 
                  T * v1, int ldv1, 
                  T * v2, int ldv2) {
  //std::cout << std::setw(10);
  //std::cout << std::setprecision(2) << std::fixed;
  bool correct = true;
  bool nan = false;
  double E = 1e-6;
  for (int i = 0; i < nrow; i++) {
      for (int j = 0; j < ncol; j++) {
          T a = v1[ldv1*i + j];
          T b = v2[ldv2*i + j];
          if (abs(a - b) > E){
              correct = false;
              // std::cout << "Diff at (" << i << ", " << j << ") ";
              // std::cout << a << " - " << b << " = " << abs(a-b) << std::endl; 
          }
          if (isnan(a) || isnan(b)) {
            correct = true;
            // std::cout << "NAN at (" << i << ", " << j << ") ";
            // std::cout << a << " - " << b << " = " << abs(a-b) << std::endl;
          }
      }
  }
  if (correct) printf("Compare: correct.\n");
  else printf("Compare: wrong.\n");
  if (nan) printf("Nan: include.\n");
  //else printf("Nan: not include.\n");
  return correct;
}

template bool compare_matrix<double>(int nrow, int ncol, 
                                      double * v1, int ldv1, 
                                      double * v2, int ldv2);
template bool compare_matrix<float>(int nrow, int ncol, 
                                      float * v1, int ldv1, 
                                      float * v2, int ldv2);

// compare 2D GPU
template <typename T>
bool compare_matrix_cuda(int nrow, int ncol, 
                      T * dv1, int lddv1, 
                      T * dv2, int lddv2) {
  mgard_cuda_handle tmp_handle(1);
  int queue_idx = 0;
  bool profile = false;

  T * v1 = new T[nrow * ncol];
  int ldv1 = ncol;
  cudaMemcpy2DAsyncHelper(v1, ldv1  * sizeof(T), 
                    dv1, lddv1 * sizeof(T),  
                    ncol * sizeof(T), nrow, 
                    D2H, tmp_handle, queue_idx, profile);
  T * v2 = new T[nrow * ncol];
  int ldv2 = ncol;
  cudaMemcpy2DAsyncHelper(v2, ldv2  * sizeof(T), 
                    dv2, lddv2 * sizeof(T),  
                    ncol * sizeof(T), nrow, 
                    D2H, tmp_handle, queue_idx, profile);
  tmp_handle.sync(queue_idx);
  bool ret = compare_matrix(nrow, ncol, 
                        v1,   ldv1, 
                        v2,   ldv2);
  delete [] v1;
  delete [] v2;
  tmp_handle.destory_all();
  return ret;
}

template bool compare_matrix_cuda<double>(int nrow, int ncol, 
                                          double * dv1, int lddv1, 
                                          double * dv2, int lddv2);
template bool compare_matrix_cuda<float>(int nrow, int ncol, 
                                          float * dv1, int lddv1, 
                                          float * dv2, int lddv2);


// compare 3D CPU
template <typename T>
bool compare_matrix(int nrow, int ncol, int nfib, 
                    T * v1, int ldv11, int ldv12, 
                    T * v2, int ldv21, int ldv22) {
  //std::cout << std::setw(10);
  //std::cout << std::setprecision(2) << std::fixed;
  bool correct = true;
  bool nan = false;
  double E = 1e-6;
  for (int i = 0; i < nrow; i++) {
    for (int j = 0; j < ncol; j++) {
      for (int k = 0; k < nfib; k++) {
        T a = v1[ldv11*ldv12*i + ldv11*j + k];
        T b = v2[ldv21*ldv22*i + ldv21*j + k];
        if (abs(a - b) > E){
          correct = false;
          // std::cout << "Diff at (" << i << ", " << j << ", " << k <<") ";
          // std::cout << a << " - " << b << " = " << abs(a-b) << std::endl; 
        }
        if (isnan(a) || isnan(b)) {
          correct = true;
          // std::cout << "NAN at (" << i << ", " << j << ") ";
          // std::cout << a << " - " << b << " = " << abs(a-b) << std::endl;
        }
      }
    }
  }
  if (correct) printf("Compare: correct.\n");
  else printf("Compare: wrong.\n");
  if (nan) printf("Nan: include.\n");
  //else printf("Nan: not include.\n");
  return correct;
}

template bool compare_matrix<double>(int nrow, int ncol, int nfib, 
                                    double * v1, int ldv11, int ldv12, 
                                    double * v2, int ldv21, int ldv22);
template bool compare_matrix<float>(int nrow, int ncol, int nfib, 
                                    float * v1, int ldv11, int ldv12, 
                                    float * v2, int ldv21, int ldv22);

// compare 3D GPU
template <typename T>
bool compare_matrix_cuda(int nrow, int ncol, int nfib, 
                      T * dv1, int lddv11, int lddv12, int sizex1,
                      T * dv2, int lddv21, int lddv22, int sizex2) {
  mgard_cuda_handle tmp_handle(1);
  int queue_idx = 0;
  bool profile = false;

  T * v1 = new T[nrow * ncol * nfib];
  int ldv11 = nfib;
  int ldv12 = ncol;
  cudaMemcpy3DAsyncHelper(v1, ldv11  * sizeof(T), nfib * sizeof(T), ldv12, 
                    dv1, lddv11 * sizeof(T), sizex1 * sizeof(T), lddv12,
                    nfib * sizeof(T), ncol, nrow,
                    D2H, tmp_handle, queue_idx, profile);

  T * v2 = new T[nrow * ncol * nfib];
  int ldv21 = nfib;
  int ldv22 = ncol;
  cudaMemcpy3DAsyncHelper(v2, ldv21  * sizeof(T), nfib * sizeof(T), ldv22, 
                    dv2, lddv21 * sizeof(T), sizex2 * sizeof(T), lddv22, 
                    nfib * sizeof(T), ncol, nrow,
                    D2H, tmp_handle, queue_idx, profile);
  tmp_handle.sync(queue_idx);
  bool ret = compare_matrix(nrow, ncol, nfib, 
                            v1,   ldv11, ldv12,
                            v2,   ldv21, ldv22);
  delete [] v1;
  delete [] v2;
  tmp_handle.destory_all();
  return ret;
}

template bool compare_matrix_cuda<double>(int nrow, int ncol, int nfib, 
                      double * dv1, int lddv11, int lddv12, int sizex1,
                      double * dv2, int lddv21, int lddv22, int sizex2);
template bool compare_matrix_cuda<float>(int nrow, int ncol, int nfib, 
                      float * dv1, int lddv11, int lddv12, int sizex1,
                      float * dv2, int lddv21, int lddv22, int sizex2);

// Allocate 1D
void cudaMallocHelper(void **  devPtr, size_t  size) {
  gpuErrchk(hipMalloc(devPtr, size));
}

// Allocate 2D
void cudaMallocPitchHelper(void ** devPtr, size_t * pitch, size_t width, size_t height) {
  gpuErrchk(hipMallocPitch(devPtr, pitch, width, height));
}

//Allocate 3D
void cudaMalloc3DHelper(void ** devPtr, size_t * pitch, size_t width, size_t height, size_t depth) {
  hipPitchedPtr devPitchedPtr;
  hipExtent extent = make_hipExtent(width, height, depth);
  gpuErrchk(hipMalloc3D(&devPitchedPtr, extent));
  *devPtr = devPitchedPtr.ptr;
  *pitch = devPitchedPtr.pitch;
}

// Copy 1D
mgard_cuda_ret
cudaMemcpyAsyncHelper(void * dst, const void * src, size_t count, enum copy_type kind,
                      mgard_cuda_handle & handle, int queue_idx, bool profile){

  hipEvent_t start, stop;
  float milliseconds = 0;
  hipStream_t stream = *(hipStream_t *)handle.get(queue_idx);
  enum hipMemcpyKind cuda_copy_type;
  switch (kind)
  {
    case H2D :  cuda_copy_type = hipMemcpyHostToDevice; break;
    case D2H :  cuda_copy_type = hipMemcpyDeviceToHost; break;
    case D2D :  cuda_copy_type = hipMemcpyDeviceToDevice; break;
  }
  if (profile) {
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));
    gpuErrchk(hipEventRecord(start, stream));
  }
  gpuErrchk(hipMemcpyAsync(dst, src, count, cuda_copy_type, stream));
  if (profile) {
    gpuErrchk(hipEventRecord(stop, stream));
    gpuErrchk(hipEventSynchronize(stop));
    gpuErrchk(hipEventElapsedTime(&milliseconds, start, stop));
    gpuErrchk(hipEventDestroy(start));
    gpuErrchk(hipEventDestroy(stop));
  }
  return mgard_cuda_ret(0, milliseconds/1000.00);

}

// Copy 2D
mgard_cuda_ret 
cudaMemcpy2DAsyncHelper(void * dst, size_t dpitch, 
                        const void * src, size_t spitch, 
                        size_t width, size_t height,
                        enum copy_type kind,
                        mgard_cuda_handle & handle, int queue_idx, bool profile) {

  hipEvent_t start, stop;
  float milliseconds = 0;
  //hipStream_t stream = *(hipStream_t *)handle.get(queue_idx);
  mgard_cuda_handle *handle2 = new mgard_cuda_handle(1);
  hipStream_t stream = *(hipStream_t *)(*handle2).get(queue_idx);
  // hipStream_t stream;
  // hipStreamCreate(&stream);
  enum hipMemcpyKind cuda_copy_type;
  switch (kind)
  {
    case H2D :  cuda_copy_type = hipMemcpyHostToDevice; break;
    case D2H :  cuda_copy_type = hipMemcpyDeviceToHost; break;
    case D2D :  cuda_copy_type = hipMemcpyDeviceToDevice; break;
  }
  if (profile) {
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));
    gpuErrchk(hipEventRecord(start, stream));
  }
  gpuErrchk(hipMemcpy2DAsync(dst, dpitch, 
                         src, spitch,
                         width, height, 
                         cuda_copy_type,
                         stream));
  if (profile) {
    gpuErrchk(hipEventRecord(stop, stream));
    gpuErrchk(hipEventSynchronize(stop));
    gpuErrchk(hipEventElapsedTime(&milliseconds, start, stop));
    gpuErrchk(hipEventDestroy(start));
    gpuErrchk(hipEventDestroy(stop));
  }
  return mgard_cuda_ret(0, milliseconds/1000.00);
}


// Copy 3D
mgard_cuda_ret 
cudaMemcpy3DAsyncHelper(void * dst, size_t dpitch, size_t dwidth, size_t dheight,
                        void * src, size_t spitch, size_t swidth, size_t sheight,
                        size_t width, size_t height, size_t depth,
                        enum copy_type kind,
                        mgard_cuda_handle & handle, int queue_idx, bool profile) {
  hipEvent_t start, stop;
  float milliseconds = 0;
  hipStream_t stream = *(hipStream_t *)handle.get(queue_idx);

  hipExtent extent = make_hipExtent(width, height, depth);
  hipMemcpy3DParms p = {0};
  p.dstPtr.ptr = dst;
  p.dstPtr.pitch = dpitch;
  p.dstPtr.xsize = dwidth;
  p.dstPtr.ysize = dheight;

  p.srcPtr.ptr = src;
  p.srcPtr.pitch = spitch;
  p.srcPtr.xsize = swidth;
  p.srcPtr.ysize = sheight;
  
  p.extent = extent;
  enum hipMemcpyKind cuda_copy_type;
  switch (kind)
  {
    case H2D :  cuda_copy_type = hipMemcpyHostToDevice; break;
    case D2H :  cuda_copy_type = hipMemcpyDeviceToHost; break;
    case D2D :  cuda_copy_type = hipMemcpyDeviceToDevice; break;
  }
  p.kind = cuda_copy_type;
  if (profile) {
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));
    gpuErrchk(hipEventRecord(start, stream));
  }
  gpuErrchk(hipMemcpy3DAsync(&p, stream));
  if (profile) {
    gpuErrchk(hipEventRecord(stop, stream));
    gpuErrchk(hipEventSynchronize(stop));
    gpuErrchk(hipEventElapsedTime(&milliseconds, start, stop));
    gpuErrchk(hipEventDestroy(start));
    gpuErrchk(hipEventDestroy(stop));
  }
  return mgard_cuda_ret(0, milliseconds/1000.00);
}

void cudaFreeHelper(void * devPtr) {
  gpuErrchk(hipFree(devPtr));
}

void cudaMemsetHelper(void * devPtr, int value, size_t count) {
  gpuErrchk(hipMemset(devPtr, value, count));
}

void cudaMemset2DHelper(void * devPtr,  size_t  pitch, int value, size_t width, size_t height) {
  gpuErrchk(hipMemset2D(devPtr, pitch, value, width, height));
}

void cudaMemset3DHelper(void * devPtr,  size_t  pitch, size_t dwidth, size_t dheight,
                        int value, size_t width, size_t height, size_t depth) {
  hipExtent extent = make_hipExtent(width, height, depth);
  hipPitchedPtr devPitchedPtr;
  devPitchedPtr.ptr = devPtr;
  devPitchedPtr.pitch = pitch;
  devPitchedPtr.xsize = dwidth;
  devPitchedPtr.ysize = dheight;
  gpuErrchk(hipMemset3D(devPitchedPtr, value, extent));
}