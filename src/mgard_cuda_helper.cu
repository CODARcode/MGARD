#include "hip/hip_runtime.h"
#include <iomanip> 
#include <iostream>
#include "mgard_cuda_helper.h"
#include "mgard_cuda_helper_internal.h"


mgard_cuda_handle::mgard_cuda_handle (int num_of_queues){
  this->num_of_queues = num_of_queues;
  hipStream_t * ptr = (hipStream_t *)this->queues;
  ptr = new hipStream_t[num_of_queues];
  for (int i = 0; i < this->num_of_queues; i++) {
    gpuErrchk(hipStreamCreate(&(ptr[i])));
  }
}

void * mgard_cuda_handle::get(int i) {
  hipStream_t * ptr = (hipStream_t *)this->queues;
  return (void *)ptr[i];
}

void mgard_cuda_handle::sync(int i) {
  hipStream_t * ptr = (hipStream_t *)this->queues;
  gpuErrchk(hipStreamSynchronize(ptr[i]));
}

void mgard_cuda_handle::sync_all() {
  hipStream_t * ptr = (hipStream_t *)this->queues;
  for (int i = 0; i < this->num_of_queues; i++) {
    gpuErrchk(hipStreamSynchronize(ptr[i]));
  }
}

void mgard_cuda_handle::destory_all() {
  hipStream_t * ptr = (hipStream_t *)this->queues;
  for (int i = 0; i < this->num_of_queues; i++) {
    gpuErrchk(hipStreamDestroy(ptr[i]));
  }
}




// print 2D CPU
void print_matrix(int nrow, int ncol, double * v, int ldv) {
  //std::cout << std::setw(10);
  //std::cout << std::setprecision(2) << std::fixed;
  for (int i = 0; i < nrow; i++) {
    for (int j = 0; j < ncol; j++) {
        std::cout <<std::setw(9) << std::setprecision(6) << std::fixed <<  v[ldv*i + j]<<", ";
    }
    std::cout << std::endl;
  }
  std::cout << std::endl;
}

// print 2D GPU
void print_matrix_cuda(int nrow, int ncol, double * dv, int lddv) {
  //std::cout << std::setw(10);
  //std::cout << std::setprecision(2) << std::fixed;
  double * v = new double[nrow * ncol];
  cudaMemcpy2DHelper(v, ncol  * sizeof(double), 
                    dv, lddv * sizeof(double),  
                    ncol * sizeof(double), nrow, 
                    D2H);
  print_matrix(nrow, ncol, v, ncol); 
}

// print 3D GPU
void print_matrix_cuda(int nrow, int ncol, int nfib, double * dv, int lddv1, int lddv2, int sizex) {
  //std::cout << std::setw(10);
  //std::cout << std::setprecision(2) << std::fixed;
  double * v = new double[nrow * ncol * nfib];
  cudaMemcpy3DHelper(v, nfib  * sizeof(double), nfib * sizeof(double), ncol,
                     dv, lddv1 * sizeof(double), sizex * sizeof(double), lddv2,
                     nfib * sizeof(double), ncol, nrow, 
                     D2H);
  print_matrix(nrow, ncol, nfib, v, nfib, ncol); 
}

// print 3D CPU
void print_matrix(int nrow, int ncol, int nfib, double * v, int ldv1, int ldv2) {
  //std::cout << std::setw(10);
  //std::cout << std::setprecision(2) << std::fixed;
  for (int i = 0; i < nrow; i++) {
    std::cout << "[ nrow = " << i << " ]\n";
    print_matrix(ncol, nfib, v + i * ldv1 * ldv2, ldv1);
    std::cout << std::endl;
  }
}


// print 2D GPU-int
void print_matrix_cuda(int nrow, int ncol, int * dv, int lddv) {
  //std::cout << std::setw(10);
  //std::cout << std::setprecision(2) << std::fixed;
  int * v = new int[nrow * ncol];
  cudaMemcpy2DHelper(v, ncol  * sizeof(int), 
                    dv, lddv * sizeof(int),  
                    ncol * sizeof(int), nrow, 
                    D2H);
  print_matrix(nrow, ncol, v, ncol);
    
}

// print 2D CPU-int
void print_matrix(int nrow, int ncol, int * v, int ldv) {
  //std::cout << std::setw(10);
  //std::cout << std::setprecision(2) << std::fixed;
  for (int i = 0; i < nrow; i++) {
    for (int j = 0; j < ncol; j++) {
        std::cout <<std::setw(5) << std::setprecision(0) << std::fixed <<  v[ldv*i + j]<<", ";
    }
    std::cout << std::endl;
  }
  std::cout << std::endl;
}

// compare 2D CPU
bool compare_matrix(int nrow, int ncol, 
                  double * v1, int ldv1, 
                  double * v2, int ldv2) {
  //std::cout << std::setw(10);
  //std::cout << std::setprecision(2) << std::fixed;
  bool correct = true;
  bool nan = false;
  double E = 1e-6;
  for (int i = 0; i < nrow; i++) {
      for (int j = 0; j < ncol; j++) {
          double a = v1[ldv1*i + j];
          double b = v2[ldv2*i + j];
          if (abs(a - b) > E){
              correct = false;
              // std::cout << "Diff at (" << i << ", " << j << ") ";
              // std::cout << a << " - " << b << " = " << abs(a-b) << std::endl; 
          }
          if (isnan(a) || isnan(b)) {
            correct = true;
            // std::cout << "NAN at (" << i << ", " << j << ") ";
            // std::cout << a << " - " << b << " = " << abs(a-b) << std::endl;
          }
      }
  }
  if (correct) printf("Compare: correct.\n");
  else printf("Compare: wrong.\n");
  if (nan) printf("Nan: include.\n");
  //else printf("Nan: not include.\n");
  return correct;
}

// compare 2D GPU
bool compare_matrix_cuda(int nrow, int ncol, 
                      double * dv1, int lddv1, 
                      double * dv2, int lddv2) {
  double * v1 = new double[nrow * ncol];
  int ldv1 = ncol;
  cudaMemcpy2DHelper(v1, ldv1  * sizeof(double), 
                    dv1, lddv1 * sizeof(double),  
                    ncol * sizeof(double), nrow, 
                    D2H);
  double * v2 = new double[nrow * ncol];
  int ldv2 = ncol;
  cudaMemcpy2DHelper(v2, ldv2  * sizeof(double), 
                    dv2, lddv2 * sizeof(double),  
                    ncol * sizeof(double), nrow, 
                    D2H);
  bool ret = compare_matrix(nrow, ncol, 
                        v1,   ldv1, 
                        v2,   ldv2);
  delete v1;
  delete v2;
  return ret;
}


// compare 3D CPU
bool compare_matrix(int nrow, int ncol, int nfib, 
                    double * v1, int ldv11, int ldv12, 
                    double * v2, int ldv21, int ldv22) {
  //std::cout << std::setw(10);
  //std::cout << std::setprecision(2) << std::fixed;
  bool correct = true;
  bool nan = false;
  double E = 1e-6;
  for (int i = 0; i < nrow; i++) {
    for (int j = 0; j < ncol; j++) {
      for (int k = 0; k < nfib; k++) {
        double a = v1[ldv11*ldv12*i + ldv11*j + k];
        double b = v2[ldv21*ldv22*i + ldv21*j + k];
        if (abs(a - b) > E){
          correct = false;
          // std::cout << "Diff at (" << i << ", " << j << ", " << k <<") ";
          // std::cout << a << " - " << b << " = " << abs(a-b) << std::endl; 
        }
        if (isnan(a) || isnan(b)) {
          correct = true;
          // std::cout << "NAN at (" << i << ", " << j << ") ";
          // std::cout << a << " - " << b << " = " << abs(a-b) << std::endl;
        }
      }
    }
  }
  if (correct) printf("Compare: correct.\n");
  else printf("Compare: wrong.\n");
  if (nan) printf("Nan: include.\n");
  //else printf("Nan: not include.\n");
  return correct;
}

// compare 3D GPU
bool compare_matrix_cuda(int nrow, int ncol, int nfib, 
                      double * dv1, int lddv11, int lddv12, int sizex1,
                      double * dv2, int lddv21, int lddv22, int sizex2) {
  double * v1 = new double[nrow * ncol * nfib];
  int ldv11 = nfib;
  int ldv12 = ncol;
  cudaMemcpy3DHelper(v1, ldv11  * sizeof(double), nfib * sizeof(double), ldv12, 
                    dv1, lddv11 * sizeof(double), sizex1 * sizeof(double), lddv12,
                    nfib * sizeof(double), ncol, nrow,
                    D2H);

  double * v2 = new double[nrow * ncol * nfib];
  int ldv21 = nfib;
  int ldv22 = ncol;
  cudaMemcpy3DHelper(v2, ldv21  * sizeof(double), nfib * sizeof(double), ldv22, 
                    dv2, lddv21 * sizeof(double), sizex2 * sizeof(double), lddv22, 
                    nfib * sizeof(double), ncol, nrow,
                    D2H);
  bool ret = compare_matrix(nrow, ncol, nfib, 
                            v1,   ldv11, ldv12,
                            v2,   ldv21, ldv22);
  delete [] v1;
  delete [] v2;
  return ret;
}


// Allocate 1D
void cudaMallocHelper(void **  devPtr, size_t  size) {
  gpuErrchk(hipMalloc(devPtr, size));
}

// Allocate 2D
void cudaMallocPitchHelper(void ** devPtr, size_t * pitch, size_t width, size_t height) {
  gpuErrchk(hipMallocPitch(devPtr, pitch, width, height));
}

//Allocate 3D
void cudaMalloc3DHelper(void ** devPtr, size_t * pitch, size_t width, size_t height, size_t depth) {
  hipPitchedPtr devPitchedPtr;
  hipExtent extent = make_hipExtent(width, height, depth);
  gpuErrchk(hipMalloc3D(&devPitchedPtr, extent));
  *devPtr = devPitchedPtr.ptr;
  *pitch = devPitchedPtr.pitch;
}

// Copy 1D
void cudaMemcpyHelper(void * dst, const void * src, size_t count, enum copy_type kind){
  enum hipMemcpyKind cuda_copy_type;
  switch (kind)
  {
    case H2D :  cuda_copy_type = hipMemcpyHostToDevice; break;
    case D2H :  cuda_copy_type = hipMemcpyDeviceToHost; break;
    case D2D :  cuda_copy_type = hipMemcpyDeviceToDevice; break;
  }
  gpuErrchk(hipMemcpy(dst, src, count, cuda_copy_type));
}

// Copy 2D
void cudaMemcpy2DHelper(void * dst, size_t dpitch, 
                        const void * src, size_t spitch, 
                        size_t width, size_t height,
                        enum copy_type kind) {
  enum hipMemcpyKind cuda_copy_type;
  switch (kind)
  {
    case H2D :  cuda_copy_type = hipMemcpyHostToDevice; break;
    case D2H :  cuda_copy_type = hipMemcpyDeviceToHost; break;
    case D2D :  cuda_copy_type = hipMemcpyDeviceToDevice; break;
  }
  gpuErrchk(hipMemcpy2D(dst, dpitch, 
                         src, spitch,
                         width, height, 
                         cuda_copy_type));
}


// Copy 3D
void cudaMemcpy3DHelper(void * dst, size_t dpitch, size_t dwidth, size_t dheight,
                        void * src, size_t spitch, size_t swidth, size_t sheight,
                        size_t width, size_t height, size_t depth,
                        enum copy_type kind) {
  hipExtent extent = make_hipExtent(width, height, depth);
  hipMemcpy3DParms p = {0};
  p.dstPtr.ptr = dst;
  p.dstPtr.pitch = dpitch;
  p.dstPtr.xsize = dwidth;
  p.dstPtr.ysize = dheight;

  p.srcPtr.ptr = src;
  p.srcPtr.pitch = spitch;
  p.srcPtr.xsize = swidth;
  p.srcPtr.ysize = sheight;
  
  p.extent = extent;
  enum hipMemcpyKind cuda_copy_type;
  switch (kind)
  {
    case H2D :  cuda_copy_type = hipMemcpyHostToDevice; break;
    case D2H :  cuda_copy_type = hipMemcpyDeviceToHost; break;
    case D2D :  cuda_copy_type = hipMemcpyDeviceToDevice; break;
  }
  p.kind = cuda_copy_type;
  gpuErrchk(hipMemcpy3D(&p));
}

void cudaFreeHelper(void * devPtr) {
  gpuErrchk(hipFree(devPtr));
}

void cudaMemsetHelper(void * devPtr, int value, size_t count) {
  gpuErrchk(hipMemset(devPtr, value, count));
}

void cudaMemset2DHelper(void * devPtr,  size_t  pitch, int value, size_t width, size_t height) {
  gpuErrchk(hipMemset2D(devPtr, pitch, value, width, height));
}

void cudaMemset3DHelper(void * devPtr,  size_t  pitch, size_t dwidth, size_t dheight,
                        int value, size_t width, size_t height, size_t depth) {
  hipExtent extent = make_hipExtent(width, height, depth);
  hipPitchedPtr devPitchedPtr;
  devPitchedPtr.ptr = devPtr;
  devPitchedPtr.pitch = pitch;
  devPitchedPtr.xsize = dwidth;
  devPitchedPtr.ysize = dheight;
  gpuErrchk(hipMemset3D(devPitchedPtr, value, extent));
}