#include "hip/hip_runtime.h"
#include "mgard_nuni.h"
#include "mgard.h"
#include "mgard_nuni_2d_cuda.h"
#include "mgard_cuda_helper.h"
#include "mgard_cuda_compact_helper.h"
#include "mgard_cuda_helper_internal.h"
#include <fstream>
#include <cmath>

namespace mgard_2d {
namespace mgard_gen {  

// todo: to make a function maps icol/irow index either from memory or calculation
// todo: double ghost col to avoid thread divergence

__device__ double
_dist_restriction_l(double * dcoord, int x, int y) {
  return dcoord[y] - dcoord[x];
}

// assume number of main col and ghost col are even numbers
__global__ void
_restriction_l_row_cuda_sm(int nr,         int nc,
                         int row_stride, int col_stride,
                         //int * __restrict__ dirow,    int * __restrict__ dicol,
                         double * __restrict__ dv,    int lddv,
                         //double * __restrict__ dcoords_x,
                         double * ddist_x,
                         int ghost_col) {

  //int ghost_col = 2;
  register int total_row = ceil((double)nr/(row_stride));
  register int total_col = ceil((double)nc/(col_stride));


  // index on dirow and dicol
  register int r0 = (blockIdx.y * blockDim.y + threadIdx.y) * row_stride;
  register int c0 = threadIdx.x;
  register int c0_stride = threadIdx.x * col_stride;

  // index on sm
  register int r0_sm = threadIdx.y;
  register int c0_sm = threadIdx.x;

  extern __shared__ double sm[]; // row = blockDim.y; col = blockDim.x + ghost_col;
  register int ldsm = blockDim.x + ghost_col;
  
  double * vec_sm = sm + r0_sm * ldsm;
  double * dist_x_sm = sm + (blockDim.y) * ldsm;
  
  register double result = 1;
  register double h1 = 1;
  register double h2 = 1;
  register double h3;
  register double h4;
  
  register double result1;
  register double result2;
  register int rest_col;
  register int real_ghost_col;
  register int real_main_col;

  register double prev_vec_sm;
  register double prev_h1;
  register double prev_h2;
  
  for (int r = r0; r < nr; r += gridDim.y * blockDim.y * row_stride) {
    
    double * vec = dv + r * lddv;

    rest_col = total_col;    
    real_ghost_col = min(ghost_col, rest_col);

    // load first ghost
    if (c0_sm < real_ghost_col) {
      vec_sm[c0_sm] = vec[c0_stride];
      if (r0_sm == 0) {
        dist_x_sm[c0_sm] = ddist_x[c0];
      }
    }
    rest_col -= real_ghost_col;
    __syncthreads();

    prev_vec_sm = 0.0;
    prev_h1 = dist_x_sm[0];
    prev_h2 = dist_x_sm[1];

    while (rest_col > blockDim.x - real_ghost_col) {
      //load main column
      real_main_col = min(blockDim.x, rest_col);
      if (c0_sm < real_main_col) {
        vec_sm[c0_sm + real_ghost_col] = vec[c0_stride + real_ghost_col * col_stride];
        if (r0_sm == 0) {
          dist_x_sm[c0_sm + real_ghost_col] = ddist_x[c0 + real_ghost_col];
        }
      }
      __syncthreads();
      
      if (c0_sm % 2 == 0) {
        //computation
        result = vec_sm[c0_sm];
        if (c0_sm == 0) {
          //result = vec_sm[c0_sm];
          h1 = prev_h1;
          h2 = prev_h2;
          h3 = dist_x_sm[c0_sm];
          h4 = dist_x_sm[c0_sm+1];
          result1 = h1 * prev_vec_sm / (h1+h2);
          // h1 = dist_x_sm[c0_sm];
          // h2 = dist_x_sm[c0_sm+1];
          result2 = h4 * vec_sm[c0_sm+1] / (h3+h4);
          //result = result + result1 + result2;
        } else {
          //result = vec_sm[c0_sm];
          h1 = dist_x_sm[c0_sm-2];
          h2 = dist_x_sm[c0_sm-1];
          h3 = dist_x_sm[c0_sm];
          h4 = dist_x_sm[c0_sm+1];
          result1 = h1 * vec_sm[c0_sm-1] / (h1+h2);
          // h1 = dist_x_sm[c0_sm];
          // h2 = dist_x_sm[c0_sm+1];
          result2 = h4 * vec_sm[c0_sm+1] / (h3+h4);
        }
        result = result + result1 + result2;
        vec[c0_stride] = result;
      }
       __syncthreads();
      
      // store last column
      if (c0_sm == 0) {
        prev_vec_sm = vec_sm[blockDim.x - 1];
        prev_h1 = dist_x_sm[blockDim.x - 2]; //_dist_restriction_l(dcoords_x_sm, blockDim.x - 2, blockDim.x - 1);
        prev_h2 = dist_x_sm[blockDim.x - 1]; //_dist_restriction_l(dcoords_x_sm, blockDim.x - 1, blockDim.x);
      }

      // advance c0
      c0 += blockDim.x;
      c0_stride += blockDim.x * col_stride;

      // copy ghost to main
      real_ghost_col = min(ghost_col, real_main_col - (blockDim.x - ghost_col));
      if (c0_sm < real_ghost_col) {
        vec_sm[c0_sm] = vec_sm[c0_sm + blockDim.x];
        if (r0_sm == 0) {
          // dcoords_x_sm[c0_sm] = dcoords_x_sm[c0_sm + blockDim.x];
          dist_x_sm[c0_sm] = dist_x_sm[c0_sm + blockDim.x];
        }
      }
      __syncthreads();
      rest_col -= real_main_col;
    } //end while

    if (c0_sm < rest_col) {
       vec_sm[c0_sm + real_ghost_col] = vec[c0_stride + real_ghost_col * col_stride];
       // dcoords_x_sm[c0_sm + real_ghost_col] = dcoords_x[c0_stride + real_ghost_col * col_stride];
       dist_x_sm[c0_sm + real_ghost_col] = ddist_x[c0 + real_ghost_col];
    }
    __syncthreads();

    if (real_ghost_col + rest_col == 1) {
      if (c0_sm == 0) {
        result = vec_sm[c0_sm];
        h1 = prev_h1;
        h2 = prev_h2;
        result += h1 * prev_vec_sm / (h1+h2);
        vec[c0_stride] = result;
      }
    } else {
      if (c0_sm < real_ghost_col + rest_col) {
        if (c0_sm % 2 == 0) {
          if (c0_sm == 0) {
            result = vec_sm[c0_sm];
            h1 = prev_h1;
            h2 = prev_h2;
            result += h1 * prev_vec_sm / (h1+h2);
            h1 = dist_x_sm[c0_sm]; //_dist_restriction_l(dcoords_x_sm, c0_sm, c0_sm+1);
            h2 = dist_x_sm[c0_sm+1]; //_dist_restriction_l(dcoords_x_sm, c0_sm+1, c0_sm+2);
            result += h2 * vec_sm[c0_sm+1] / (h1+h2);
          } else if (c0_sm == real_ghost_col + rest_col - 1) {
            result = vec_sm[c0_sm];
            h1 = dist_x_sm[c0_sm-2]; //_dist_restriction_l(dcoords_x_sm, c0_sm-2, c0_sm-1);;
            h2 = dist_x_sm[c0_sm-1]; //_dist_restriction_l(dcoords_x_sm, c0_sm-1, c0_sm);;
            result += h1 * vec_sm[c0_sm-1] / (h1+h2);
          } else {
            result = vec_sm[c0_sm];
            h1 = dist_x_sm[c0_sm-2]; //_dist_restriction_l(dcoords_x_sm, c0_sm-2, c0_sm-1);
            h2 = dist_x_sm[c0_sm-1]; //_dist_restriction_l(dcoords_x_sm, c0_sm-1, c0_sm);
            result += h1 * vec_sm[c0_sm-1] / (h1+h2);
            h1 = dist_x_sm[c0_sm]; //_dist_restriction_l(dcoords_x_sm, c0_sm, c0_sm+1);
            h2 = dist_x_sm[c0_sm+1]; //_dist_restriction_l(dcoords_x_sm, c0_sm+1, c0_sm+2);
            result += h2 * vec_sm[c0_sm+1] / (h1+h2);
          }
          vec[c0_stride] = result;
        }
        __syncthreads();
      }
    }
  }

}


// __global__ void
// _restriction_l_row_cuda_sm(int nrow,       int ncol,
//                          int nr,         int nc,
//                          int row_stride, int col_stride,
//                          int * __restrict__ dirow,    int * __restrict__ dicol,
//                          double * __restrict__ dv,    int lddv,
//                          double * __restrict__ dcoords_x,
//                          int ghost_col) {

//   //int ghost_col = 2;
//   register int total_row = ceil((double)nr/(row_stride));
//   register int total_col = ceil((double)nc/(col_stride));


//   // index on dirow and dicol
//   register int r0 = (blockIdx.y * blockDim.y + threadIdx.y) * row_stride;
//   register int c0 = threadIdx.x * col_stride;

//   // index on sm
//   register int r0_sm = threadIdx.y;
//   register int c0_sm = threadIdx.x;

//   extern __shared__ double sm[]; // row = blockDim.y; col = blockDim.x + ghost_col;
//   register int ldsm = blockDim.x + ghost_col;
//   // printf("ldsm = %d\n", ldsm);
  
//   double * vec_sm = sm + r0_sm * ldsm;
//   double * dcoords_x_sm = sm + blockDim.y * ldsm;
  
//   register double result = 1;
//   register double h1 = 1;
//   register double h2 = 1;
  
//   register int rest_col;
//   register int real_ghost_col;
//   register int real_main_col;

//   register double prev_vec_sm;
//   register double prev_h1;
//   register double prev_h2;
  
//   for (int r = r0; r < nr; r += gridDim.y * blockDim.y * row_stride) {
    
//     double * vec = dv + dirow[r] * lddv;

//     rest_col = total_col;    
//     real_ghost_col = min(ghost_col, rest_col);

//     // load first ghost
//     if (c0_sm < real_ghost_col) {
//       vec_sm[c0_sm] = vec[dicol[c0]];
//       if (r0_sm == 0) {
//         dcoords_x_sm[c0_sm] = dcoords_x[dicol[c0]];
//       }
//     }
//     rest_col -= real_ghost_col;
//     __syncthreads();

//     prev_vec_sm = 0.0;
//     prev_h1 = _dist_restriction_l(dcoords_x_sm, 0, 1);
//     prev_h2 = _dist_restriction_l(dcoords_x_sm, 1, 2);

//     while (rest_col > blockDim.x - real_ghost_col) {
//       //load main column
//       real_main_col = min(blockDim.x, rest_col);
//       if (c0_sm < real_main_col) {
//         vec_sm[c0_sm + real_ghost_col] = vec[dicol[c0 + real_ghost_col * col_stride]];
//         if (r0_sm == 0) {
//           dcoords_x_sm[c0_sm + real_ghost_col] = dcoords_x[dicol[c0 + real_ghost_col * col_stride]];
//         }
//       }
//       __syncthreads();

//       if (c0_sm % 2 == 0) {
//         //computation
//         if (c0_sm == 0) {
//           result = vec_sm[c0_sm];
//           h1 = prev_h1;
//           h2 = prev_h2;
//           result += h1 * prev_vec_sm / (h1+h2);
//           h1 = _dist_restriction_l(dcoords_x_sm, c0_sm, c0_sm+1);
//           h2 = _dist_restriction_l(dcoords_x_sm, c0_sm+1, c0_sm+2);
//           result += h2 * vec_sm[c0_sm+1] / (h1+h2);
//         } else {
//           result = vec_sm[c0_sm];
//           h1 = _dist_restriction_l(dcoords_x_sm, c0_sm-2, c0_sm-1);
//           h2 = _dist_restriction_l(dcoords_x_sm, c0_sm-1, c0_sm);
//           result += h1 * vec_sm[c0_sm-1] / (h1+h2);
//           h1 = _dist_restriction_l(dcoords_x_sm, c0_sm, c0_sm+1);
//           h2 = _dist_restriction_l(dcoords_x_sm, c0_sm+1, c0_sm+2);
//           result += h2 * vec_sm[c0_sm+1] / (h1+h2);
//         }
//         vec[dicol[c0]] = result;
//       }
//        __syncthreads();
    
//       // store last column
//       if (c0_sm == 0) {
//         prev_vec_sm = vec_sm[blockDim.x - 1];
//         prev_h1 = _dist_restriction_l(dcoords_x_sm, blockDim.x - 2, blockDim.x - 1);
//         prev_h2 = _dist_restriction_l(dcoords_x_sm, blockDim.x - 1, blockDim.x);
//       }

//       // advance c0
//       c0 += blockDim.x * col_stride;

//       // copy ghost to main
//       real_ghost_col = min(ghost_col, real_main_col - (blockDim.x - ghost_col));
//       if (c0_sm < real_ghost_col) {
//         vec_sm[c0_sm] = vec_sm[c0_sm + blockDim.x];
//         if (r0_sm == 0) {
//           dcoords_x_sm[c0_sm] = dcoords_x_sm[c0_sm + blockDim.x];
//         }
//       }
//       __syncthreads();
//       rest_col -= real_main_col;
//     } //end while

//     if (c0_sm < rest_col) {
//        vec_sm[c0_sm + real_ghost_col] = vec[dicol[c0 + real_ghost_col * col_stride]];
//        dcoords_x_sm[c0_sm + real_ghost_col] = dcoords_x[dicol[c0 + real_ghost_col * col_stride]];
//     }
//     __syncthreads();

//     if (real_ghost_col + rest_col == 1) {
//       if (c0_sm == 0) {
//         result = vec_sm[c0_sm];
//         h1 = prev_h1;
//         h2 = prev_h2;
//         result += h1 * prev_vec_sm / (h1+h2);
//         vec[dicol[c0]] = result;
//       }
//     } else {
//       if (c0_sm < real_ghost_col + rest_col) {
//         if (c0_sm % 2 == 0) {
//           if (c0_sm == 0) {
//             result = vec_sm[c0_sm];
//             h1 = prev_h1;
//             h2 = prev_h2;
//             result += h1 * prev_vec_sm / (h1+h2);
//             h1 = _dist_restriction_l(dcoords_x_sm, c0_sm, c0_sm+1);
//             h2 = _dist_restriction_l(dcoords_x_sm, c0_sm+1, c0_sm+2);
//             result += h2 * vec_sm[c0_sm+1] / (h1+h2);
//           } else if (c0_sm == real_ghost_col + rest_col - 1) {
//             result = vec_sm[c0_sm];
//             h1 = _dist_restriction_l(dcoords_x_sm, c0_sm-2, c0_sm-1);;
//             h2 = _dist_restriction_l(dcoords_x_sm, c0_sm-1, c0_sm);;
//             result += h1 * vec_sm[c0_sm-1] / (h1+h2);
//           } else {
//             result = vec_sm[c0_sm];
//             h1 = _dist_restriction_l(dcoords_x_sm, c0_sm-2, c0_sm-1);
//             h2 = _dist_restriction_l(dcoords_x_sm, c0_sm-1, c0_sm);
//             result += h1 * vec_sm[c0_sm-1] / (h1+h2);
//             h1 = _dist_restriction_l(dcoords_x_sm, c0_sm, c0_sm+1);
//             h2 = _dist_restriction_l(dcoords_x_sm, c0_sm+1, c0_sm+2);
//             result += h2 * vec_sm[c0_sm+1] / (h1+h2);
//           }
//           vec[dicol[c0]] = result;
//         }
//         __syncthreads();
//       }
//     }
//   }
// }

mgard_cuda_ret 
restriction_l_row_cuda_sm(int nr,         int nc,
                          int row_stride, int col_stride,
                          double * dv,    int lddv,
                          double * ddist_x,
                          int B, int ghost_col) {
 

  // //hipMemcpyToSymbol(HIP_SYMBOL(dcoords_x_const), dcoords_x, sizeof(double)*nc );
  // double * ddist_x;
  // //int len_ddist_x = ceil((float)nc/col_stride)-1;
  // int len_ddist_x = ceil((float)nc/col_stride); // add one for better consistance for backward
  // cudaMallocHelper((void**)&ddist_x, len_ddist_x*sizeof(double));
  // calc_cpt_dist(nc, col_stride, dcoords_x, ddist_x);
  // // printf("dcoords_x %d:\n", nc);
  // // print_matrix_cuda(1, nc, dcoords_x, nc);
  // // printf("ddist_x:\n");
  // // print_matrix_cuda(1, len_ddist_x, ddist_x, len_ddist_x);

  // int B = 4;
  // int ghost_col = 2;
  int total_row = ceil((double)nr/(row_stride));
  int total_col = ceil((double)nc/(col_stride));
  int total_thread_y = ceil((double)nr/(row_stride));
  int total_thread_x = min(B, total_col);

  int tby = min(B, total_thread_y);
  int tbx = min(B, total_thread_x);


  size_t sm_size = ((tbx + ghost_col) * (tby + 1)) * sizeof(double);

  int gridy = ceil((float)total_thread_y/tby);
  int gridx = 1; //ceil((float)total_thread_x/tbx);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);

  // std::cout << "thread block: " << tby << ", " << tbx << std::endl;
  // std::cout << "grid: " << gridy << ", " << gridx<< std::endl;



  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  _restriction_l_row_cuda_sm<<<blockPerGrid, threadsPerBlock, sm_size>>>(nr,         nc,
                                                                         row_stride, col_stride,
                                                                         dv,         lddv,
                                                                         ddist_x,
                                                                         ghost_col);
  gpuErrchk(hipGetLastError ());

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return mgard_cuda_ret(0, milliseconds/1000.0);
}

// assume number of main col and ghost col are even numbers
__global__ void
_restriction_l_col_cuda_sm(int nr,         int nc,
                           int row_stride, int col_stride,
                           double * __restrict__ dv,    int lddv,
                           double * ddist_y,
                           int ghost_row) {

  //int ghost_col = 2;
  register int total_row = ceil((double)nr/(row_stride));
  register int total_col = ceil((double)nc/(col_stride));


  // index on dirow and dicol
  register int c0 = (blockIdx.x * blockDim.x + threadIdx.x) * col_stride;
  register int r0 = threadIdx.y;
  register int r0_stride = threadIdx.y * row_stride;

  // index on sm
  register int r0_sm = threadIdx.y;
  register int c0_sm = threadIdx.x;

  extern __shared__ double sm[]; // row = blockDim.y; col = blockDim.x + ghost_col;
  register int ldsm = blockDim.x;
  
  double * vec_sm = sm + c0_sm;
  double * dist_y_sm = sm + (blockDim.y + ghost_row) * ldsm;
  
  register double result = 1;
  register double h1 = 1;
  register double h2 = 1;
  register double h3;
  register double h4;
  
  register double result1;
  register double result2;
  register int rest_row;
  register int real_ghost_row;
  register int real_main_row;

  register double prev_vec_sm;
  register double prev_h1;
  register double prev_h2;
  
  for (int c = c0; c < nc; c += gridDim.x * blockDim.x * col_stride) {
    
    double * vec = dv + c;

    rest_row = total_row;    
    real_ghost_row = min(ghost_row, rest_row);

    // load first ghost
    if (r0_sm < real_ghost_row) {
      vec_sm[r0_sm * ldsm] = vec[r0_stride * lddv];
    }
    if (c0_sm == 0 && r0_sm < real_ghost_row) {
        dist_y_sm[r0_sm] = ddist_y[r0];
        // printf("load dist[%d] = %f\n", c0_sm, dist_y_sm[c0_sm]);
    }
    rest_row -= real_ghost_row;
    __syncthreads();

    prev_vec_sm = 0.0;
    prev_h1 = dist_y_sm[0];
    prev_h2 = dist_y_sm[1];

    while (rest_row > blockDim.y - real_ghost_row) {
  //     //load main column
      real_main_row = min(blockDim.y, rest_row);
      if (r0_sm < real_main_row) {
        vec_sm[(r0_sm + real_ghost_row) * ldsm] = vec[(r0_stride + real_ghost_row * row_stride) * lddv];
      }
      if (c0_sm == 0 && r0_sm < real_main_row) {
          dist_y_sm[r0_sm + real_ghost_row] = ddist_y[r0 + real_ghost_row];
      }
      __syncthreads();
      
      if (r0_sm % 2 == 0) {
        //computation
        result = vec_sm[r0_sm * ldsm];
        if (r0_sm == 0) {
          //result = vec_sm[c0_sm];
          h1 = prev_h1;
          h2 = prev_h2;
          h3 = dist_y_sm[r0_sm];
          h4 = dist_y_sm[r0_sm+1];
          result1 = h1 * prev_vec_sm / (h1+h2);
          // h1 = dist_x_sm[c0_sm];
          // h2 = dist_x_sm[c0_sm+1];
          result2 = h4 * vec_sm[(r0_sm+1) * ldsm] / (h3+h4);
          //result = result + result1 + result2;
        } else {
          //result = vec_sm[c0_sm];
          h1 = dist_y_sm[r0_sm-2];
          h2 = dist_y_sm[r0_sm-1];
          h3 = dist_y_sm[r0_sm];
          h4 = dist_y_sm[r0_sm+1];
          result1 = h1 * vec_sm[(r0_sm-1) * ldsm] / (h1+h2);
          // h1 = dist_x_sm[c0_sm];
          // h2 = dist_x_sm[c0_sm+1];
          result2 = h4 * vec_sm[(r0_sm+1) * ldsm] / (h3+h4);
        }
        result = result + result1 + result2;
        vec[r0_stride * lddv] = result;
      }
       __syncthreads();
      
      // store last column
      if (r0_sm == 0) {
        prev_vec_sm = vec_sm[(blockDim.y - 1) * ldsm];
        prev_h1 = dist_y_sm[blockDim.y - 2]; //_dist_restriction_l(dcoords_x_sm, blockDim.x - 2, blockDim.x - 1);
        prev_h2 = dist_y_sm[blockDim.y - 1]; //_dist_restriction_l(dcoords_x_sm, blockDim.x - 1, blockDim.x);
      }

      // advance c0
      r0 += blockDim.y;
      r0_stride += blockDim.y * row_stride;

  //     // copy ghost to main
      real_ghost_row = min(ghost_row, real_main_row - (blockDim.y - ghost_row));
      if (r0_sm < real_ghost_row) {
        vec_sm[r0_sm * ldsm] = vec_sm[(r0_sm + blockDim.y) * ldsm];
      }
      if (c0_sm == 0 && r0_sm < real_ghost_row) {
        dist_y_sm[r0_sm] = dist_y_sm[r0_sm + blockDim.y];
      }
      __syncthreads();
      rest_row -= real_main_row;
     } //end while

    if (r0_sm < rest_row) {
      vec_sm[(r0_sm + real_ghost_row) * ldsm] = vec[(r0_stride + real_ghost_row * row_stride) * lddv]; 
    }
    if (c0_sm == 0 && r0_sm < rest_row) {
      dist_y_sm[r0_sm + real_ghost_row] = ddist_y[r0 + real_ghost_row];
    }
    __syncthreads();

    if (real_ghost_row + rest_row == 1) {
      if (r0_sm == 0) {
        result = vec_sm[r0_sm * ldsm];
        h1 = prev_h1;
        h2 = prev_h2;
        result += h1 * prev_vec_sm / (h1+h2);
        vec[r0_stride * lddv] = result;
      }
    } else {
      if (r0_sm < real_ghost_row + rest_row) {
        if (r0_sm % 2 == 0) {
          if (r0_sm == 0) {
            result = vec_sm[r0_sm * ldsm];
            h1 = prev_h1;
            h2 = prev_h2;
            result += h1 * prev_vec_sm / (h1+h2);
            h1 = dist_y_sm[r0_sm]; //_dist_restriction_l(dcoords_x_sm, c0_sm, c0_sm+1);
            h2 = dist_y_sm[r0_sm+1]; //_dist_restriction_l(dcoords_x_sm, c0_sm+1, c0_sm+2);
            result += h2 * vec_sm[(r0_sm+1) * ldsm] / (h1+h2);
          } else if (r0_sm == real_ghost_row + rest_row - 1) {
            result = vec_sm[r0_sm * ldsm];
            h1 = dist_y_sm[r0_sm-2]; //_dist_restriction_l(dcoords_x_sm, c0_sm-2, c0_sm-1);;
            h2 = dist_y_sm[r0_sm-1]; //_dist_restriction_l(dcoords_x_sm, c0_sm-1, c0_sm);;
            result += h1 * vec_sm[(r0_sm-1) * ldsm] / (h1+h2);
          } else {
            result = vec_sm[r0_sm * ldsm];
            h1 = dist_y_sm[r0_sm-2]; //_dist_restriction_l(dcoords_x_sm, c0_sm-2, c0_sm-1);
            h2 = dist_y_sm[r0_sm-1]; //_dist_restriction_l(dcoords_x_sm, c0_sm-1, c0_sm);
            result += h1 * vec_sm[(r0_sm-1) * ldsm] / (h1+h2);
            h1 = dist_y_sm[r0_sm]; //_dist_restriction_l(dcoords_x_sm, c0_sm, c0_sm+1);
            h2 = dist_y_sm[r0_sm+1]; //_dist_restriction_l(dcoords_x_sm, c0_sm+1, c0_sm+2);
            result += h2 * vec_sm[(r0_sm+1) * ldsm] / (h1+h2);
          }
          vec[r0_stride * lddv] = result;
        }
        __syncthreads();
      }
    }
  }

}


mgard_cuda_ret 
restriction_l_col_cuda_sm(int nr,         int nc,
                          int row_stride, int col_stride,
                          double * dv,    int lddv,
                          double * ddist_y,
                          int B, int ghost_row) {
 

  // //hipMemcpyToSymbol(HIP_SYMBOL(dcoords_x_const), dcoords_x, sizeof(double)*nc );
  // double * ddist_y;
  // //int len_ddist_x = ceil((float)nc/col_stride)-1;
  // int len_ddist_y = ceil((float)nr/row_stride); // add one for better consistance for backward
  // cudaMallocHelper((void**)&ddist_y, len_ddist_y*sizeof(double));
  // calc_cpt_dist(nr, row_stride, dcoords_y, ddist_y);
  // // printf("dcoords_y %d:\n", nc);
  // // print_matrix_cuda(1, nr, dcoords_y, nr);
  // // printf("ddist_y:\n");
  // // print_matrix_cuda(1, len_ddist_y, ddist_y, len_ddist_y);

  // int B = 4;
  // int ghost_col = 2;
  int total_row = ceil((double)nr/(row_stride));
  int total_col = ceil((double)nc/(col_stride));
  int total_thread_y = min(B, total_row);
  int total_thread_x = ceil((double)nc/(col_stride));

  int tby = min(B, total_thread_y);
  int tbx = min(B, total_thread_x);


  size_t sm_size = ((tby + ghost_row) * (tbx + 1)) * sizeof(double);

  int gridy = 1;
  int gridx = ceil((float)total_thread_x/tbx); //ceil((float)total_thread_x/tbx);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);

  // std::cout << "thread block: " << tby << ", " << tbx << std::endl;
  // std::cout << "grid: " << gridy << ", " << gridx<< std::endl;



  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  _restriction_l_col_cuda_sm<<<blockPerGrid, threadsPerBlock, sm_size>>>(nr,         nc,
                                                                       row_stride, col_stride,
                                                                       dv,         lddv,
                                                                       ddist_y,
                                                                       ghost_row);
  gpuErrchk(hipGetLastError ());

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return mgard_cuda_ret(0, milliseconds/1000.0);
}



__global__ void
_restriction_l_row_cuda_sm_pf(int nrow,       int ncol,
                         int nr,         int nc,
                         int row_stride, int col_stride,
                         int * __restrict__ dirow,    int * __restrict__ dicol,
                         double * __restrict__ dv,    int lddv,
                         double * __restrict__ dcoords_x,
                         int ghost_col) {

  //int ghost_col = 2;
  register int total_row = ceil((double)nr/(row_stride));
  register int total_col = ceil((double)nc/(col_stride));


  // index on dirow and dicol
  register int r0 = (blockIdx.y * blockDim.y + threadIdx.y) * row_stride;
  register int c0 = threadIdx.x * col_stride;

  // index on sm
  register int r0_sm = threadIdx.y;
  register int c0_sm = threadIdx.x;

  extern __shared__ double sm[]; // row = blockDim.y; col = blockDim.x + ghost_col;
  register int ldsm = blockDim.x + ghost_col;
  // printf("ldsm = %d\n", ldsm);
  
  double * vec_sm = sm + r0_sm * ldsm;
  double * dcoords_x_sm = sm + blockDim.y * ldsm;
  
  register double result = 1;
  register double h1 = 1;
  register double h2 = 1;
  
  register int main_col = blockDim.x;
  register int rest_load_col;
  register int rest_comp_col;
  register int curr_ghost_col;
  register int curr_main_col;
  register int next_ghost_col;
  register int next_main_col;

  register double prev_vec_sm;
  register double prev_h1;
  register double prev_h2;

  register double next_dv;
  register double next_dcoords_x;
  
  for (int r = r0; r < nr; r += gridDim.y * blockDim.y * row_stride) {
    
    double * vec = dv + dirow[r] * lddv;
    
    rest_load_col = total_col;
    rest_comp_col = total_col;
    curr_ghost_col = min(ghost_col, rest_load_col);

    // load first ghost
    if (c0_sm < curr_ghost_col) {
      vec_sm[c0_sm] = vec[dicol[c0]];
      if (r0_sm == 0) {
        dcoords_x_sm[c0_sm] = dcoords_x[dicol[c0]];
      }
    }
    rest_load_col -= curr_ghost_col;
    //load main column
    curr_main_col = min(blockDim.x, rest_load_col);
    if (c0_sm < curr_main_col) {
      vec_sm[c0_sm + curr_ghost_col] = vec[dicol[c0 + curr_ghost_col * col_stride]];
      if (r0_sm == 0) {
        dcoords_x_sm[c0_sm + curr_ghost_col] = dcoords_x[dicol[c0 + curr_ghost_col * col_stride]];
      }
    }
    rest_load_col -= curr_main_col;
    __syncthreads();

    prev_vec_sm = 0.0;
    prev_h1 = _dist_restriction_l(dcoords_x_sm, 0, 1);
    prev_h2 = _dist_restriction_l(dcoords_x_sm, 1, 2);

    while (rest_comp_col > main_col) {
      //load next main column
      next_main_col = min(blockDim.x, rest_load_col);
      int next_c0 = c0 + (curr_main_col + curr_ghost_col) * col_stride;
      if (c0_sm < next_main_col) {
        next_dv = vec[dicol[next_c0]];
        if (r0_sm == 0) {
          next_dcoords_x = dcoords_x[dicol[next_c0]];
        }
      }
      __syncthreads();

      if (c0_sm % 2 == 0) {
        //computation
        if (c0_sm == 0) {
          result = vec_sm[c0_sm];
          h1 = prev_h1;
          h2 = prev_h2;
          result += h1 * prev_vec_sm / (h1+h2);
          h1 = _dist_restriction_l(dcoords_x_sm, c0_sm, c0_sm+1);
          h2 = _dist_restriction_l(dcoords_x_sm, c0_sm+1, c0_sm+2);
          result += h2 * vec_sm[c0_sm+1] / (h1+h2);
        } else {
          result = vec_sm[c0_sm];
          h1 = _dist_restriction_l(dcoords_x_sm, c0_sm-2, c0_sm-1);
          h2 = _dist_restriction_l(dcoords_x_sm, c0_sm-1, c0_sm);
          result += h1 * vec_sm[c0_sm-1] / (h1+h2);
          h1 = _dist_restriction_l(dcoords_x_sm, c0_sm, c0_sm+1);
          h2 = _dist_restriction_l(dcoords_x_sm, c0_sm+1, c0_sm+2);
          result += h2 * vec_sm[c0_sm+1] / (h1+h2);
        }
        vec[dicol[c0]] = result;
      }
      
      rest_comp_col -= main_col;

      // store last column
      if (c0_sm == 0) {
        prev_vec_sm = vec_sm[blockDim.x - 1];
        prev_h1 = _dist_restriction_l(dcoords_x_sm, blockDim.x - 2, blockDim.x - 1);
        prev_h2 = _dist_restriction_l(dcoords_x_sm, blockDim.x - 1, blockDim.x);
      }

      __syncthreads();
      
      // advance c0
      c0 += blockDim.x * col_stride;

      // copy ghost to main
      next_ghost_col = curr_main_col + curr_ghost_col - main_col;
      if (c0_sm < next_ghost_col) {
        vec_sm[c0_sm] = vec_sm[c0_sm + main_col];
        if (r0_sm == 0) {
          dcoords_x_sm[c0_sm] = dcoords_x_sm[c0_sm + main_col];
        }
      }
      __syncthreads();
      // copy next main to main
      if (c0_sm < next_main_col) {
        vec_sm[c0_sm + next_ghost_col] = next_dv;
        if (r0_sm == 0) {
          dcoords_x_sm[c0_sm + next_ghost_col] = next_dcoords_x;
        }
      }
      rest_load_col -= next_main_col;

      curr_ghost_col = next_ghost_col;
      curr_main_col = next_main_col;
      //rest_col -= real_main_col;
    } //end while

    // if (c0_sm < col) {
    //    vec_sm[c0_sm + real_ghost_col] = vec[dicol[c0 + real_ghost_col * col_stride]];
    //    dcoords_x_sm[c0_sm + real_ghost_col] = dcoords_x[dicol[c0 + real_ghost_col * col_stride]];
    // }
    // __syncthreads();

    if (rest_comp_col == 1) {
      if (c0_sm == 0) {
        result = vec_sm[c0_sm];
        h1 = prev_h1;
        h2 = prev_h2;
        result += h1 * prev_vec_sm / (h1+h2);
        vec[dicol[c0]] = result;
      }
    } else {
      if (c0_sm < rest_comp_col) {
        if (c0_sm % 2 == 0) {
          if (c0_sm == 0) {
            result = vec_sm[c0_sm];
            h1 = prev_h1;
            h2 = prev_h2;
            result += h1 * prev_vec_sm / (h1+h2);
            h1 = _dist_restriction_l(dcoords_x_sm, c0_sm, c0_sm+1);
            h2 = _dist_restriction_l(dcoords_x_sm, c0_sm+1, c0_sm+2);
            result += h2 * vec_sm[c0_sm+1] / (h1+h2);
          } else if (c0_sm == rest_comp_col - 1) {
            result = vec_sm[c0_sm];
            h1 = _dist_restriction_l(dcoords_x_sm, c0_sm-2, c0_sm-1);;
            h2 = _dist_restriction_l(dcoords_x_sm, c0_sm-1, c0_sm);;
            result += h1 * vec_sm[c0_sm-1] / (h1+h2);
          } else {
            result = vec_sm[c0_sm];
            h1 = _dist_restriction_l(dcoords_x_sm, c0_sm-2, c0_sm-1);
            h2 = _dist_restriction_l(dcoords_x_sm, c0_sm-1, c0_sm);
            result += h1 * vec_sm[c0_sm-1] / (h1+h2);
            h1 = _dist_restriction_l(dcoords_x_sm, c0_sm, c0_sm+1);
            h2 = _dist_restriction_l(dcoords_x_sm, c0_sm+1, c0_sm+2);
            result += h2 * vec_sm[c0_sm+1] / (h1+h2);
          }
          vec[dicol[c0]] = result;
        }
        __syncthreads();      
      }
    }
  }
}


mgard_cuda_ret 
restriction_l_row_cuda_sm_pf(int nrow,       int ncol,
                     int nr,         int nc,
                     int row_stride, int col_stride,
                     int * dirow,    int * dicol,
                     double * dv,    int lddv,
                     double * dcoords_x,
                     int B, int ghost_col) {
 

  //hipMemcpyToSymbol(HIP_SYMBOL(dcoords_x_const), dcoords_x, sizeof(double)*nc );

  // int B = 4;
  // int ghost_col = 2;
  int total_row = ceil((double)nr/(row_stride));
  int total_col = ceil((double)nc/(col_stride));
  int total_thread_y = ceil((double)nr/(row_stride));
  int total_thread_x = min(B, total_col);

  int tby = min(B, total_thread_y);
  int tbx = min(B, total_thread_x);


  size_t sm_size = ((tbx + ghost_col) * (tby + 1)) * sizeof(double);

  int gridy = ceil((float)total_thread_y/tby);
  int gridx = 1; //ceil((float)total_thread_x/tbx);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);

  // std::cout << "thread block: " << tby << ", " << tbx << std::endl;
  // std::cout << "grid: " << gridy << ", " << gridx<< std::endl;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  _restriction_l_row_cuda_sm_pf<<<blockPerGrid, threadsPerBlock, sm_size>>>(nrow,       ncol,
                                                                       nr,         nc,
                                                                       row_stride, col_stride,
                                                                       dirow,      dicol,
                                                                       dv,         lddv,
                                                                       dcoords_x,
                                                                       ghost_col);
  gpuErrchk(hipGetLastError ());

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return mgard_cuda_ret(0, milliseconds/1000.0);
}
}
}
