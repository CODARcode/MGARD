#include "hip/hip_runtime.h"
#include "mgard_nuni.h"
#include "mgard.h"
#include "mgard_nuni_2d_cuda.h"
#include "mgard_cuda_helper.h"
#include "mgard_cuda_compact_helper.h"
#include "mgard_cuda_helper_internal.h"
#include <fstream>
#include <cmath>

namespace mgard_2d {
namespace mgard_gen {  

__global__ void 
_pi_Ql_cuda_sm(int nr,           int nc,
             int row_stride,   int col_stride,
             double * dv,      int lddv, 
             double * ddist_x, double * ddist_y) {

  register int c0 = blockIdx.x * blockDim.x;
  register int c0_stride = c0 * col_stride;
  register int r0 = blockIdx.y * blockDim.y;
  register int r0_stride = r0 * row_stride;

  register int total_row = ceil((double)nr/(row_stride));
  register int total_col = ceil((double)nc/(col_stride));

  register int c_sm = threadIdx.x;
  register int r_sm = threadIdx.y;

  extern __shared__ double sm[]; // size: (blockDim.x + 1) * (blockDim.y + 1)
  int ldsm = blockDim.x + 1;
  double * v_sm = sm;
  double * dist_x_sm = sm + (blockDim.x + 1) * (blockDim.y + 1);
  double * dist_y_sm = dist_x_sm + blockDim.x;

  for (int r = r0; r < total_row - 1; r += blockDim.y * gridDim.y) {
    for (int c = c0; c < total_col - 1; c += blockDim.x * gridDim.x) {
      /* Load v */
      if (c + c_sm < total_col && r + r_sm < total_row) {
        v_sm[r_sm * ldsm + c_sm] = dv[(r + r_sm) * row_stride * lddv + (c + c_sm) * col_stride];

        if (r_sm == 0 && r + blockDim.y < total_row) {
          v_sm[blockDim.y * ldsm + c_sm] = dv[(r + blockDim.y) * row_stride * lddv + (c + c_sm) * col_stride];
        }
        if (c_sm == 0 && c + blockDim.x < total_col) {
          v_sm[r_sm * ldsm + blockDim.x] = dv[(r + r_sm) * row_stride * lddv + (c + blockDim.x) * col_stride];
        }
        if (r_sm == 0 && c_sm == 0 && r + blockDim.y < total_row && c + blockDim.x < total_col) {
          v_sm[blockDim.y * ldsm + blockDim.x] = dv[(r + blockDim.y) * row_stride * lddv + (c + blockDim.x) * col_stride];
        }
      }

      /* Load dist_x */
      //if (c + c_sm < total_col) {
      if (r_sm == 0 && c + c_sm < total_col) {
        dist_x_sm[c_sm] = ddist_x[c + c_sm];
      }
      /* Load dist_y */
      //if (r + r_sm < total_row) {
      if (c_sm == 0 && r + r_sm < total_row) {  
        dist_y_sm[r_sm] = ddist_y[r + r_sm];
        // printf("load ddist_y[%d] %f\n", r_sm, dist_y_sm[r_sm]);
      }

      __syncthreads();

      /* Compute */
      if (r_sm % 2 == 0 && c_sm % 2 != 0) {
        double h1 = dist_x_sm[c_sm - 1];
        double h2 = dist_x_sm[c_sm];
        v_sm[r_sm * ldsm + c_sm] -= (h2 * v_sm[r_sm * ldsm + (c_sm - 1)] + 
                                     h1 * v_sm[r_sm * ldsm + (c_sm + 1)])/
                                    (h1 + h2);
        dv[(r + r_sm) * row_stride * lddv + (c + c_sm) * col_stride] = v_sm[r_sm * ldsm + c_sm];
      } 
      if (r_sm % 2 != 0 && c_sm % 2 == 0) {
        double h1 = dist_y_sm[r_sm - 1];
        double h2 = dist_y_sm[r_sm];
        v_sm[r_sm * ldsm + c_sm] -= (h2 * v_sm[(r_sm - 1) * ldsm + c_sm] +
                                     h1 * v_sm[(r_sm + 1) * ldsm + c_sm])/
                                    (h1 + h2);
        dv[(r + r_sm) * row_stride * lddv + (c + c_sm) * col_stride] = v_sm[r_sm * ldsm + c_sm];

        // if (r_sm == 5) {
        //   printf("dv %f h1 %f h2 %f dv-1 %f dv+1 %f\n", 
        //          v_sm[r_sm * ldsm + c_sm],
        //          dist_y_sm[r_sm - 1], dist_y_sm[r_sm], v_sm[(r_sm - 1) * ldsm + c_sm], v_sm[(r_sm + 1) * ldsm + c_sm]);
        // }
      } 
      if (r_sm % 2 != 0 && c_sm % 2 != 0) {
        double h1_col = dist_x_sm[c_sm - 1];
        double h2_col = dist_x_sm[c_sm];
        double h1_row = dist_y_sm[r_sm - 1];
        double h2_row = dist_y_sm[r_sm];
        v_sm[r_sm * ldsm + c_sm] -= (v_sm[(r_sm - 1) * ldsm + (c_sm - 1)] * h2_col * h2_row +
                                     v_sm[(r_sm - 1) * ldsm + (c_sm + 1)] * h1_col * h2_row + 
                                     v_sm[(r_sm + 1) * ldsm + (c_sm - 1)] * h2_col * h1_row + 
                                     v_sm[(r_sm + 1) * ldsm + (c_sm + 1)] * h1_col * h1_row)/
                                    ((h1_col + h2_col) * (h1_row + h2_row));
        dv[(r + r_sm) * row_stride * lddv + (c + c_sm) * col_stride] = v_sm[r_sm * ldsm + c_sm];
      }
      /* extra computaion for global boarder */
      if (c + blockDim.x == total_col - 1) {
        if (r_sm % 2 != 0 && c_sm == 0) {
          double h1 = dist_y_sm[r_sm - 1];
          double h2 = dist_y_sm[r_sm];
          v_sm[r_sm * ldsm + blockDim.x] -= (h2 * v_sm[(r_sm - 1) * ldsm + blockDim.x] +
                                             h1 * v_sm[(r_sm + 1) * ldsm + blockDim.x])/
                                            (h1 + h2);
          dv[(r + r_sm) * row_stride * lddv + (c + blockDim.x) * col_stride] = v_sm[r_sm * ldsm + blockDim.x];
        } 
      }
      if (r + blockDim.y == total_row - 1) {
        if (r_sm == 0 && c_sm % 2 != 0) {
          double h1 = dist_x_sm[c_sm - 1];
          double h2 = dist_x_sm[c_sm];
          v_sm[blockDim.y * ldsm + c_sm] -= (h2 * v_sm[blockDim.y * ldsm + (c_sm - 1)] + 
                                             h1 * v_sm[blockDim.y * ldsm + (c_sm + 1)])/
                                            (h1 + h2);
          dv[(r + blockDim.y) * row_stride * lddv + (c + c_sm) * col_stride] = v_sm[blockDim.y * ldsm + c_sm];
        }
      }
      __syncthreads();
    }
  }
}


mgard_cuda_ret 
pi_Ql_cuda_sm(int nr,         int nc,
              int row_stride, int col_stride,
              double * dv,    int lddv,
              double * ddist_x, double * ddist_y,
              int B) {
 

  //hipMemcpyToSymbol(HIP_SYMBOL(dcoords_x_const), dcoords_x, sizeof(double)*nc );
  // double * ddist_x;
  // //int len_ddist_x = ceil((float)nc/col_stride)-1;
  // int len_ddist_x = ceil((float)nc/col_stride); // add one for better consistance for backward
  // cudaMallocHelper((void**)&ddist_x, len_ddist_x*sizeof(double));
  // calc_cpt_dist(nc, col_stride, dcoords_x, ddist_x);
  // // printf("dcoords_x %d:\n", nc);
  // // print_matrix_cuda(1, nc, dcoords_x, nc);
  // printf("ddist_x:\n");
  // print_matrix_cuda(1, len_ddist_x, ddist_x, len_ddist_x);

  // double * ddist_y;
  // //int len_ddist_x = ceil((float)nc/col_stride)-1;
  // int len_ddist_y = ceil((float)nr/row_stride); // add one for better consistance for backward
  // cudaMallocHelper((void**)&ddist_y, len_ddist_y*sizeof(double));
  // calc_cpt_dist(nr, row_stride, dcoords_y, ddist_y);
  // // printf("dcoords_y %d:\n", nc);
  // // print_matrix_cuda(1, nr, dcoords_y, nr);
  // printf("ddist_y:\n");
  // print_matrix_cuda(1, len_ddist_y, ddist_y, len_ddist_y);



  // int B = 4;
  // int ghost_col = 2;
  int total_row = ceil((double)nr/(row_stride));
  int total_col = ceil((double)nc/(col_stride));
  int total_thread_y = total_row - 1;
  int total_thread_x = total_col - 1;

  int tby = min(B, total_thread_y);
  int tbx = min(B, total_thread_x);


  size_t sm_size = ((B+1) * (B+1) + 2 * B) * sizeof(double);

  int gridy = ceil((float)total_thread_y/tby);
  int gridx = ceil((float)total_thread_x/tbx);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);

  // std::cout << "thread block: " << tby << ", " << tbx << std::endl;
  // std::cout << "grid: " << gridy << ", " << gridx<< std::endl;



  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  _pi_Ql_cuda_sm<<<blockPerGrid, threadsPerBlock, sm_size>>>(nr,         nc,
                                                             row_stride, col_stride,
                                                             dv,         lddv,
                                                             ddist_x,    ddist_y);


  gpuErrchk(hipGetLastError ());

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return mgard_cuda_ret(0, milliseconds/1000.0);
}





}
}