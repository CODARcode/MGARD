#include "hip/hip_runtime.h"
/*
 * Copyright 2021, Oak Ridge National Laboratory.
 * MGARD-GPU: MultiGrid Adaptive Reduction of Data Accelerated by GPUs
 * Author: Jieyang Chen (chenj3@ornl.gov)
 * Date: April 2, 2021
 */

#include <chrono>
#include <fstream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "mgard_api_cuda.h"

#define ANSI_RED "\x1b[31m"
#define ANSI_GREEN "\x1b[32m"
#define ANSI_RESET "\x1b[0m"

using namespace std::chrono;

void print_usage_message(char *argv[], FILE *fp) {
  fprintf(fp,
          "Usage: %s infile nrow ncol nfib tolerance opt (-1: CPU; 0: CUDA, 1: "
          "CUDA-optimized)\n",
          argv[0]);
}

void print_for_more_details_message(char *argv[], FILE *fp) {
  fprintf(fp, "\nFor more details, run: %s --help\n", argv[0]);
}

void print_help_message(char *argv[], FILE *fp) {
  fprintf(fp, "\nThe input file `infile` should contain a "
              "float[`nrow`][`ncol`][`nfib`] array.\n"
              "The array will be compressed so that the error as measured in "
              "the H^`s` norm is\n"
              "no more than `tolerance`. \n");
}

int main(int argc, char *argv[]) {
  size_t result;

  if (argc == 2 && (!strcmp(argv[1], "--help") || !strcmp(argv[1], "-h"))) {
    print_usage_message(argv, stdout);
    print_help_message(argv, stdout);
    return 0;
  }

  int data_srouce; // 0: generate random data; 1: input file
  char *infile, *outfile;
  int nrow, ncol, nfib, opt, B = 16, num_of_queues = 32;
  std::vector<size_t> shape;
  float tol, s = 0;

  int i = 1;
  data_srouce = atoi(argv[i++]);
  if (data_srouce) {
    infile = argv[i++];
    printf("Input data: %s ", infile);
  } else {
    printf("Input data: random generated ");
  }
  int D = atoi(argv[i++]);
  printf(" shape: %d ( ", D);
  for (int d = 0; d < D; d++) {
    shape.push_back(atoi(argv[i++]));
    printf("%d ", shape[shape.size() - 1]);
  }
  printf(")\n");
  tol = atof(argv[i++]);
  printf("Rel. error bound: %.2e ", tol);
  s = atof(argv[i++]);
  printf("S: %.2f\n", s);
  opt = atoi(argv[i++]);
  printf("Optimization: %d\n", opt);

  long lSize;
  long num_floats;

  num_floats = 1;
  for (int d = 0; d < shape.size(); d++) {
    num_floats *= shape[d];
  }
  long num_bytes = sizeof(float) * num_floats;
  lSize = num_bytes;

  float *in_buff;
  mgard_cuda::cudaMallocHostHelper((void **)&in_buff,
                                   sizeof(float) * num_floats);
  if (in_buff == NULL) {
    fputs("Memory error", stderr);
    exit(2);
  }

  if (data_srouce == 0) {
    fprintf(stdout,
            "No input file provided. Generating random data for testing\n");
    for (int i = 0; i < num_floats; i++) {
      in_buff[i] = rand() % 10 + 1;
    }
    // printf("num_floats %d\n", num_floats);

    fprintf(stdout, "Done Generating data.\n");
  } else {
    fprintf(stdout, "Loading file: %s\n", infile);
    FILE *pFile;
    pFile = fopen(infile, "rb");
    if (pFile == NULL) {
      fputs("File error", stderr);
      exit(1);
    }
    fseek(pFile, 0, SEEK_END);
    long lSize = ftell(pFile);

    rewind(pFile);

    lSize = num_bytes;

    if (lSize != num_bytes) {
      fprintf(stderr,
              "%s contains %lu bytes when %lu were expected. Exiting.\n",
              infile, lSize, num_bytes);
      return 1;
    }

    result = fread(in_buff, 1, lSize, pFile);
    if (result != lSize) {
      fputs("Reading error", stderr);
      exit(3);
    }
    fclose(pFile);
  }

  float data_L_inf_norm = 0;
  for (int i = 0; i < num_floats; ++i) {
    float temp = fabs(in_buff[i]);
    if (temp > data_L_inf_norm)
      data_L_inf_norm = temp;
  }

  size_t out_size;
  unsigned char *mgard_comp_buff;
  float *mgard_out_buff;

  printf("Start compressing and decompressing with GPU\n");
  if (D == 1) {
    mgard::mgard_cuda_handle<float, 1> handle(shape);
    mgard_comp_buff = mgard::compress_cuda(handle, in_buff, out_size, tol, s);
    mgard_out_buff = mgard::decompress_cuda(handle, mgard_comp_buff, out_size);
  } else if (D == 2) {
    mgard_cuda_handle<float, 2> handle(shape);
    mgard_comp_buff = mgard::compress_cuda(handle, in_buff, out_size, tol, s);
    mgard_out_buff = mgard::decompress_cuda(handle, mgard_comp_buff, out_size);
  } else if (D == 3) {
    mgard_cuda_handle<float, 3> handle(shape);
    mgard_comp_buff = mgard::compress_cuda(handle, in_buff, out_size, tol, s);
    mgard_out_buff = mgard::decompress_cuda(handle, mgard_comp_buff, out_size);
  } else if (D == 4) {
    mgard_cuda_handle<float, 4> handle(shape);
    mgard_comp_buff = mgard::compress_cuda(handle, in_buff, out_size, tol, s);
    mgard_out_buff = mgard::decompress_cuda(handle, mgard_comp_buff, out_size);
  }

  printf("In size:  %10ld  Out size: %10d  Compression ratio: %10ld \n", lSize,
         out_size, lSize / out_size);

  // FILE *qfile;
  // qfile = fopen ( outfile , "wb" );
  // result = fwrite (mgard_out_buff, 1, lSize, qfile);
  // fclose(qfile);
  // if (result != lSize) {fputs ("Writing error",stderr); exit (4);}
  int error_count = 100;
  float error_L_inf_norm = 0;
  float sum = 0;
  for (int i = 0; i < num_floats; ++i) {
    float temp = fabs(in_buff[i] - mgard_out_buff[i]);
    if (temp > error_L_inf_norm)
      error_L_inf_norm = temp;
    if (temp / data_L_inf_norm >= tol && error_count) {
      printf("not bounded: buffer[%d]: %f vs. mgard_out_buff[%d]: %f \n", i,
             in_buff[i], i, mgard_out_buff[i]);
      error_count--;
    }
    sum += temp * temp;
  }

  mgard_cuda::cudaFreeHostHelper(in_buff);

  // printf("sum: %e\n", sum/num_floats);
  float relative_L_inf_error = error_L_inf_norm / data_L_inf_norm;

  // std::ofstream fout("mgard_out.dat", std::ios::binary);
  // fout.write(reinterpret_cast<const char *>(mgard_comp_buff), out_size);
  // fout.close();

  printf("Rel. L^infty error bound: %10.5E \n", tol);
  printf("Rel. L^infty error: %10.5E \n", relative_L_inf_error);

  if (relative_L_inf_error < tol) {
    printf(ANSI_GREEN "SUCCESS: Error tolerance met!" ANSI_RESET "\n");
    return 0;
  } else {
    printf(ANSI_RED "FAILURE: Error tolerance NOT met!" ANSI_RESET "\n");
    return -1;
  }
}
