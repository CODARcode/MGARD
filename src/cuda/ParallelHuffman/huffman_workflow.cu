#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <algorithm>
#include <bitset>
#include <cassert>
#include <cmath>
#include <functional>
#include <iostream>
#include <numeric>
#include <stdexcept>
#include <string>
#include <sys/stat.h>
#include <tuple>
#include <type_traits>
#include <unistd.h>
#include <vector>

#include <chrono>

using namespace std::chrono;

#include "cuda/Common.h"
#include "cuda/CommonInternal.h"

#include "cuda/ParallelHuffman/canonical.cuh"
#include "cuda/ParallelHuffman/cuda_error_handling.cuh"
#include "cuda/ParallelHuffman/cuda_mem.cuh"
#include "cuda/ParallelHuffman/dbg_gpu_printing.cuh"
#include "cuda/ParallelHuffman/format.hh"
#include "cuda/ParallelHuffman/histogram.cuh"
#include "cuda/ParallelHuffman/huffman.cuh"
#include "cuda/ParallelHuffman/huffman_codec.cuh"
#include "cuda/ParallelHuffman/huffman_workflow.cuh"
#include "cuda/ParallelHuffman/par_huffman.cuh"
#include "cuda/ParallelHuffman/types.hh"

#include "cuda/ParallelHuffman/Histogram.hpp"
#include "cuda/ParallelHuffman/GetCodebook.hpp"
#include "cuda/ParallelHuffman/EncodeFixedLen.hpp"
#include "cuda/ParallelHuffman/Deflate.hpp"
#include "cuda/ParallelHuffman/Decode.hpp"

int ht_state_num;
int ht_all_nodes;
using uint8__t = uint8_t;

template <typename Q>
void wrapper::GetFrequency(Q *d_bcode, size_t len, unsigned int *d_freq,
                           int dict_size) {
  // Parameters for thread and block count optimization

  // Initialize to device-specific values
  int deviceId;
  int maxbytes;
  int maxbytesOptIn;
  int numSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&maxbytes, hipDeviceAttributeMaxSharedMemoryPerBlock,
                         deviceId);
  hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

  // Account for opt-in extra shared memory on certain architectures
  hipDeviceGetAttribute(&maxbytesOptIn,
                         hipDeviceAttributeSharedMemPerBlockOptin, deviceId);
  maxbytes = std::max(maxbytes, maxbytesOptIn);

  // Optimize launch
  int numBuckets = dict_size;
  int numValues = len;
  int itemsPerThread = 1;
  int RPerBlock = (maxbytes / (int)sizeof(int)) / (numBuckets + 1);
  int numBlocks = numSMs;
  hipFuncSetAttribute(reinterpret_cast<const void*>(p2013Histogram<Q), unsigned int>,
                       hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);

  // printf("dict_size: %u, RPerBlock: %d\n", dict_size, RPerBlock);

  // fits to size
  int threadsPerBlock =
      ((((numValues / (numBlocks * itemsPerThread)) + 1) / 64) + 1) * 64;

  while (threadsPerBlock > 1024) {
    if (RPerBlock <= 1) {
      threadsPerBlock = 1024;
    } else {
      RPerBlock /= 2;
      numBlocks *= 2;
      threadsPerBlock =
          ((((numValues / (numBlocks * itemsPerThread)) + 1) / 64) + 1) * 64;
    }
  }

  // mgard_cuda::print_matrix_cuda(1, 10, (int *)d_bcode, 10);

  // printf("maxbytes: %d, p2013Histogram: %d\n", maxbytes,(numBuckets + 1) *
  // sizeof(int));

  // printf("numBlocks: %d, threadsPerBlock: %d, sm: %d\n", numBlocks, threadsPerBlock, ((numBuckets + 1) * RPerBlock) * sizeof(int));
  p2013Histogram //
      <<<numBlocks, threadsPerBlock,
         ((numBuckets + 1) * RPerBlock) * sizeof(int)>>> //
      (d_bcode, d_freq, numValues, numBuckets, RPerBlock);
  hipDeviceSynchronize();

  // TODO make entropy optional
  // {
  //     auto   freq    = mem::CreateHostSpaceAndMemcpyFromDevice(d_freq,
  //     dict_size); double entropy = 0.0; for (auto i = 0; i < dict_size; i++)
  //         if (freq[i]) {
  //             auto possibility = freq[i] / (1.0 * len);
  //             entropy -= possibility * log(possibility);
  //             cout << i << ": " << freq[i] << "\n";
  //         }
  //     cout << log_info << "entropy:\t\t" << entropy << endl;
  //     delete[] freq;
  // }

  // #ifdef DEBUG_PRINT
  //     print_histogram<unsigned int><<<1, 32>>>(d_freq, dict_size, dict_size /
  //     2); hipDeviceSynchronize();
  // #endif
}

template <typename H>
void PrintChunkHuffmanCoding(size_t *dH_bit_meta, //
                             size_t *dH_uInt_meta, size_t len, int chunk_size,
                             size_t total_bits, size_t total_uInts) {
  cout << "\n" << log_dbg << "Huffman coding detail start ------" << endl;
  printf("| %s\t%s\t%s\t%s\t%9s\n", "chunk", "bits", "bytes", "uInt",
         "chunkCR");
  for (size_t i = 0; i < 8; i++) {
    size_t n_byte = (dH_bit_meta[i] - 1) / 8 + 1;
    auto chunk_CR = ((double)chunk_size * sizeof(float) /
                     (1.0 * (double)dH_uInt_meta[i] * sizeof(H)));
    printf("| %lu\t%lu\t%lu\t%lu\t%9.6lf\n", i, dH_bit_meta[i], n_byte,
           dH_uInt_meta[i], chunk_CR);
  }
  cout << "| ..." << endl
       << "| Huff.total.bits:\t" << total_bits << endl
       << "| Huff.total.bytes:\t" << total_uInts * sizeof(H) << endl
       << "| Huff.CR (uInt):\t"
       << (double)len * sizeof(float) / (total_uInts * 1.0 * sizeof(H)) << endl;
  cout << log_dbg << "coding detail end ----------------" << endl;
  cout << endl;
}

// template <mgard_cuda::DIM D, typename T, typename S, typename Q, typename H, typename DeviceType>
// void HuffmanEncode(mgard_cuda::Handle<D, T> &handle, S *dqv, size_t n,
//                    std::vector<size_t> &outlier_idx, H *&dmeta,
//                    size_t &dmeta_size, H *&ddata, size_t &ddata_size,
//                    int chunk_size, int dict_size) {

template <mgard_cuda::DIM D, typename T, typename S, typename Q, typename H, typename DeviceType>
void HuffmanEncode(mgard_cuda::Handle<D, T> &handle, S *dqv, size_t n,
                   std::vector<size_t> &outlier_idx, H *&dmeta,
                   size_t &dmeta_size, H *&ddata, size_t &ddata_size,
                   int chunk_size, int dict_size) {

  high_resolution_clock::time_point t1, t2, start, end;
  duration<double> time_span;

  int queue_idx = 0;

  Q *dprimary = (Q *)dqv;
  size_t primary_count = n;

  t1 = high_resolution_clock::now();

  ht_state_num = 2 * dict_size;
  ht_all_nodes = 2 * ht_state_num;

  mgard_cuda::Array<1, unsigned int, DeviceType> freq_array({(mgard_cuda::SIZE)ht_all_nodes});
  freq_array.memset(0);

  mgard_cuda::SubArray<1, Q, DeviceType> dprimary_subarray({(mgard_cuda::SIZE)n}, dprimary);
  mgard_cuda::SubArray<1, unsigned int, DeviceType> freq_subarray(freq_array);
  mgard_cuda::Histogram<Q, unsigned int, DeviceType>().Execute(dprimary_subarray, freq_subarray, primary_count, dict_size, 0);
  gpuErrchk(hipDeviceSynchronize());

  auto type_bw = sizeof(H) * 8;
  size_t decodebook_size = sizeof(H) * (2 * type_bw) + sizeof(Q) * dict_size;
  mgard_cuda::Array<1, H, DeviceType> codebook_array({(mgard_cuda::SIZE)dict_size});
  codebook_array.memset(0);
  mgard_cuda::Array<1, uint8_t, DeviceType> decodebook_array({(mgard_cuda::SIZE)decodebook_size});
  codebook_array.memset(0xff);

  H * codebook = codebook_array.get_dv();
  uint8_t *decodebook = decodebook_array.get_dv();


  mgard_cuda::SubArray<1, H, DeviceType> codebook_subarray(codebook_array);
  mgard_cuda::SubArray<1, uint8_t, DeviceType> decodebook_subarray(decodebook_array);

  mgard_cuda::GetCodebook<Q, H, DeviceType>(dict_size, freq_subarray, codebook_subarray, decodebook_subarray);
  hipDeviceSynchronize();

  mgard_cuda::Array<1, H, DeviceType> huff_array({(mgard_cuda::SIZE)primary_count});
  huff_array.memset(0);
  H * huff = huff_array.get_dv();

  gpuErrchk(hipDeviceSynchronize());

  mgard_cuda::SubArray<1, H, DeviceType> huff_subarray(huff_array);
  mgard_cuda::EncodeFixedLen<unsigned int, H, DeviceType>().Execute(dprimary_subarray,
                                                              huff_subarray,
                                                              primary_count,
                                                              codebook_subarray, 0);

  // deflate
  auto nchunk = (primary_count - 1) / chunk_size + 1; 
  mgard_cuda::Array<1, size_t, DeviceType> huff_bitwidths_array({(mgard_cuda::SIZE)nchunk});
  huff_bitwidths_array.memset(0);
  size_t * huff_bitwidths = huff_bitwidths_array.get_dv();

  mgard_cuda::SubArray<1, size_t, DeviceType> huff_bitwidths_subarray({(mgard_cuda::SIZE)nchunk}, huff_bitwidths);
  mgard_cuda::Deflate<H, DeviceType>().Execute(huff_subarray, primary_count, huff_bitwidths_subarray, chunk_size, 0);

  mgard_cuda::DeviceRuntime<DeviceType>::SyncQueue(0);


  // dump TODO change to int
  size_t* h_meta = new size_t[nchunk * 3]();
  size_t* dH_uInt_meta = h_meta;
  size_t* dH_bit_meta = h_meta + nchunk;
  size_t* dH_uInt_entry = h_meta + nchunk * 2;

  mgard_cuda::MemoryManager<DeviceType>().Copy1D(dH_bit_meta, huff_bitwidths, nchunk, 0);
  gpuErrchk(hipDeviceSynchronize());
  // transform in uInt
  memcpy(dH_uInt_meta, dH_bit_meta, nchunk * sizeof(size_t));
  for_each(dH_uInt_meta, dH_uInt_meta + nchunk,
           [&](size_t &i) { i = (i - 1) / (sizeof(H) * 8) + 1; });
  // make it entries
  memcpy(dH_uInt_entry + 1, dH_uInt_meta, (nchunk - 1) * sizeof(size_t));
  for (auto i = 1; i < nchunk; i++)
    dH_uInt_entry[i] += dH_uInt_entry[i - 1];

  // sum bits from each chunk
  auto total_bits =
      std::accumulate(dH_bit_meta, dH_bit_meta + nchunk, (size_t)0);
  auto total_uInts =
      std::accumulate(dH_uInt_meta, dH_uInt_meta + nchunk, (size_t)0);

  gpuErrchk(hipDeviceSynchronize());
  t2 = high_resolution_clock::now();
  time_span = duration_cast<duration<double>>(t2 - t1);
  // printf("huffman encode time: %.6f s\n", time_span.count());

  // out_meta: |outlier count|outlier idx|outlier data|primary count|dict
  // size|chunk size|huffmeta size|huffmeta|decodebook size|decodebook|
  // out_data: |huffman data|

  t1 = high_resolution_clock::now();
  dmeta_size = // sizeof(size_t) + outlier_count * sizeof(size_t) +
               // outlier_count * sizeof(S) + //outlier
      sizeof(size_t) + sizeof(int) + sizeof(int) + // primary
      sizeof(size_t) + 2 * nchunk * sizeof(size_t) + sizeof(size_t) +
      (sizeof(H) * (2 * type_bw) + sizeof(Q) * dict_size) * sizeof(uint8_t);


  mgard_cuda::cudaMallocHelper(handle, (void **)&dmeta, dmeta_size);
  ddata_size = total_uInts * sizeof(H);
  mgard_cuda::cudaMallocHelper(handle, (void **)&ddata, ddata_size);

  mgard_cuda::Array<1, mgard_cuda::Byte, DeviceType> meta_array({(mgard_cuda::SIZE)dmeta_size});
  mgard_cuda::Array<1, mgard_cuda::Byte, DeviceType> data_array({(mgard_cuda::SIZE)ddata_size});

  void *dmeta_p = (void *)dmeta;
  mgard_cuda::cudaMemcpyAsyncHelper(handle, dmeta_p, &primary_count,
                                    sizeof(size_t), mgard_cuda::H2D,
                                    (queue_idx++) % handle.num_of_queues);
  dmeta_p = dmeta_p + sizeof(size_t);
  mgard_cuda::cudaMemcpyAsyncHelper(handle, dmeta_p, &dict_size, sizeof(int),
                                    mgard_cuda::H2D,
                                    (queue_idx++) % handle.num_of_queues);
  dmeta_p = dmeta_p + sizeof(int);
  mgard_cuda::cudaMemcpyAsyncHelper(handle, dmeta_p, &chunk_size, sizeof(int),
                                    mgard_cuda::H2D,
                                    (queue_idx++) % handle.num_of_queues);
  dmeta_p = dmeta_p + sizeof(int);
  size_t huffmeta_size = 2 * nchunk * sizeof(size_t);
  // printf("compress huffmeta_size: %llu\n", huffmeta_size);
  mgard_cuda::cudaMemcpyAsyncHelper(handle, dmeta_p, &huffmeta_size,
                                    sizeof(size_t), mgard_cuda::H2D,
                                    (queue_idx++) % handle.num_of_queues);
  dmeta_p = dmeta_p + sizeof(size_t);
  mgard_cuda::cudaMemcpyAsyncHelper(handle, dmeta_p, h_meta + nchunk,
                                    huffmeta_size, mgard_cuda::H2D,
                                    (queue_idx++) % handle.num_of_queues);
  dmeta_p = dmeta_p + huffmeta_size;
  mgard_cuda::cudaMemcpyAsyncHelper(handle, dmeta_p, &decodebook_size,
                                    sizeof(size_t), mgard_cuda::H2D,
                                    (queue_idx++) % handle.num_of_queues);
  dmeta_p = dmeta_p + sizeof(size_t);
  // printf("compress decodebook_size: %llu\n", decodebook_size);
  mgard_cuda::cudaMemcpyAsyncHelper(handle, dmeta_p, decodebook,
                                    decodebook_size, mgard_cuda::H2D,
                                    (queue_idx++) % handle.num_of_queues);
  dmeta_p = dmeta_p + decodebook_size;

  gpuErrchk(hipDeviceSynchronize());
  t2 = high_resolution_clock::now();
  time_span = duration_cast<duration<double>>(t2 - t1);
  // printf("serilization time1: %.6f s\n", time_span.count());

  t1 = high_resolution_clock::now();

  for (auto i = 0; i < nchunk; i++) {
    mgard_cuda::cudaMemcpyAsyncHelper(
        handle, ddata + dH_uInt_entry[i], (void *)(huff + i * chunk_size),
        dH_uInt_meta[i] * sizeof(H), mgard_cuda::D2D,
        (queue_idx++) % handle.num_of_queues);
  }

  gpuErrchk(hipDeviceSynchronize());
  t2 = high_resolution_clock::now();
  time_span = duration_cast<duration<double>>(t2 - t1);
  // printf("serilization time2: %.6f s\n", time_span.count());

  //////// clean up
  // hipHostFree(flags);
  // hipFree(doutlier);
  // hipFree(dprimary);
  // hipFree(freq);
  // hipFree(codebook);
  // hipFree(decodebook);
  // hipFree(huff);
  // hipFree(huff_bitwidths);
  delete[] h_meta;
}

template <mgard_cuda::DIM D, typename T, typename S, typename Q, typename H, typename DeviceType>
void HuffmanDecode(mgard_cuda::Handle<D, T> &handle, S *&dqv, size_t &n,
                   H *dmeta, size_t dmeta_size, H *ddata, size_t ddata_size) {

  Q *dprimary;
  S *doutlier;
  size_t primary_count;
  size_t outlier_count;
  size_t *outlier_idx;
  size_t huffmeta_size;

  int dict_size;
  int chunk_size;
  size_t *huffmeta;
  uint8_t *decodebook;
  size_t decodebook_size;

  void *dmeta_p = (void *)dmeta;

  // primary
  mgard_cuda::cudaMemcpyAsyncHelper(handle, &primary_count, dmeta_p,
                                    sizeof(size_t), mgard_cuda::D2H, 0);
  dmeta_p = dmeta_p + sizeof(size_t);
  // printf("decompress primary_count: %llu\n", primary_count);
  mgard_cuda::cudaMallocHelper(handle, (void **)&dprimary, primary_count * sizeof(Q));

  mgard_cuda::cudaMemcpyAsyncHelper(handle, &dict_size, dmeta_p, sizeof(int),
                                    mgard_cuda::D2H, 0);
  dmeta_p = dmeta_p + sizeof(int);
  mgard_cuda::cudaMemcpyAsyncHelper(handle, &chunk_size, dmeta_p, sizeof(int),
                                    mgard_cuda::D2H, 0);
  dmeta_p = dmeta_p + sizeof(int);
  mgard_cuda::cudaMemcpyAsyncHelper(handle, &huffmeta_size, dmeta_p,
                                    sizeof(size_t), mgard_cuda::D2H, 0);
  dmeta_p = dmeta_p + sizeof(size_t);
  // printf("decompress huffmeta_size: %llu\n", huffmeta_size);
  mgard_cuda::cudaMallocHelper(handle, (void **)&huffmeta, huffmeta_size);
  mgard_cuda::cudaMemcpyAsyncHelper(handle, huffmeta, dmeta_p, huffmeta_size,
                                    mgard_cuda::D2D, 0);
  // // huffmeta = (size_t *)dmeta_p;
  dmeta_p = dmeta_p + huffmeta_size;
  mgard_cuda::cudaMemcpyAsyncHelper(handle, &decodebook_size, dmeta_p,
                                    sizeof(size_t), mgard_cuda::D2H, 0);
  dmeta_p = dmeta_p + sizeof(size_t);
  // printf("decompress decodebook_size: %llu\n", decodebook_size);
  mgard_cuda::cudaMallocHelper(handle, (void **)&decodebook, decodebook_size);
  mgard_cuda::cudaMemcpyAsyncHelper(handle, decodebook, dmeta_p,
                                    decodebook_size, mgard_cuda::D2D, 0);
  // // decodebook = (uint8_t *)dmeta_p;
  dmeta_p = dmeta_p + decodebook_size;

  // printf("start decoding\n");
  int nchunk = (primary_count - 1) / chunk_size + 1;
  auto blockDim = tBLK_DEFLATE; // the same as deflating
  auto gridDim = (nchunk - 1) / blockDim + 1;

  // Decode<<<gridDim, blockDim, decodebook_size>>>( //
  //     ddata, huffmeta, dprimary, primary_count, chunk_size, nchunk,
      // (uint8_t *)decodebook, (size_t)decodebook_size);
  hipDeviceSynchronize();

  mgard_cuda::SubArray<1, H, mgard_cuda::CUDA> ddata_subarray({(mgard_cuda::SIZE)ddata_size}, ddata);
  mgard_cuda::SubArray<1, size_t, mgard_cuda::CUDA> huffmeta_subarray({(mgard_cuda::SIZE)huffmeta_size/(mgard_cuda::SIZE)sizeof(size_t)}, huffmeta);
  mgard_cuda::SubArray<1, Q, mgard_cuda::CUDA> dprimary_subarray({(mgard_cuda::SIZE)primary_count}, dprimary);
  mgard_cuda::SubArray<1, uint8_t, mgard_cuda::CUDA> decodebook_subarray({(mgard_cuda::SIZE)decodebook_size}, decodebook);
  mgard_cuda::Decode<Q, H, mgard_cuda::CUDA>().Execute(ddata_subarray,
                                                       huffmeta_subarray,
                                                       dprimary_subarray,
                                                       primary_count, chunk_size, nchunk,
                                                       decodebook_subarray, decodebook_size, 0);
  hipDeviceSynchronize();
  dqv = (S *)dprimary;
  n = primary_count;
}

template void wrapper::GetFrequency<uint8__t>(uint8__t *, size_t,
                                              unsigned int *, int);
template void wrapper::GetFrequency<uint16_t>(uint16_t *, size_t,
                                              unsigned int *, int);
template void wrapper::GetFrequency<uint32_t>(uint32_t *, size_t,
                                              unsigned int *, int);

template void PrintChunkHuffmanCoding<uint32_t>(size_t *, size_t *, size_t, int,
                                                size_t, size_t);
template void PrintChunkHuffmanCoding<uint64_t>(size_t *, size_t *, size_t, int,
                                                size_t, size_t);

// template tuple3ul HuffmanEncode<uint8__t, uint32_t, float>(Handle<uint8__t>
// &, string&, uint8__t*, size_t, void * &, size_t &, int, int); template
// tuple3ul HuffmanEncode<uint16_t, uint32_t, float>(Handle<uint16_t> &,
// string&, uint16_t*, size_t, void * &, size_t &, int, int); template tuple3ul
// HuffmanEncode<uint32_t, uint32_t, float>(Handle<uint32_t> &, string&,
// uint32_t*, size_t, void * &, size_t &, int, int); template tuple3ul
// HuffmanEncode<uint8__t, uint64_t, float>(Handle<uint8__t> &, string&,
// uint8__t*, size_t, void * &, size_t &, int, int); template tuple3ul
// HuffmanEncode<uint16_t, uint64_t, float>(Handle<uint16_t> &, string&,
// uint16_t*, size_t, void * &, size_t &, int, int);

// template uint8__t* HuffmanDecode<uint8__t, uint32_t, float>(std::string&,
// void * d_in, size_t, int, int, int); template uint16_t*
// HuffmanDecode<uint16_t, uint32_t, float>(std::string&, void * d_in, size_t,
// int, int, int); template uint32_t* HuffmanDecode<uint32_t, uint32_t,
// float>(std::string&, void * d_in, size_t, int, int, int); template uint8__t*
// HuffmanDecode<uint8__t, uint64_t, float>(std::string&, void * d_in, size_t,
// int, int, int); template uint16_t* HuffmanDecode<uint16_t, uint64_t,
// float>(std::string&, void * d_in, size_t, int, int, int); template uint32_t*
// HuffmanDecode<uint32_t, uint64_t, float>(std::string&, void * d_in, size_t,
// int, int, int);

// template void HuffmanEncode<double, int, uint32_t>(Handle<double> &handle,
//     int* dqv, size_t n, bool * dflags, uint32_t * &dmeta, size_t &dmeta_size,
//     uint32_t * &ddata, size_t &ddata_size, int chunk_size, int dict_size);
// template void HuffmanEncode<float, int, uint32_t>(Handle<float> &handle,
//     int* dqv, size_t n, bool * dflags, uint32_t * &dmeta, size_t &dmeta_size,
//     uint32_t * &ddata, size_t &ddata_size, int chunk_size, int dict_size);

// template void HuffmanDecode<float, int, uint32_t>(Handle<float> &handle,
//                           int* &dqv, size_t &n, uint32_t * dmeta, size_t
//                           dmeta_size, uint32_t * ddata, size_t ddata_size);
// template void HuffmanDecode<double, int, uint32_t>(Handle<double> &handle,
//                           int* &dqv, size_t &n, uint32_t * dmeta, size_t
//                           dmeta_size, uint32_t * ddata, size_t ddata_size);

#define KERNELS(D, T, S, Q, H)                                                 \
  template void HuffmanEncode<D, T, S, Q, H, mgard_cuda::CUDA>(                                  \
      mgard_cuda::Handle<D, T> & handle, S * dqv, size_t n,                    \
      std::vector<size_t> & outlier_idx, H * &dmeta, size_t & dmeta_size,      \
      H * &ddata, size_t & ddata_size, int chunk_size, int dict_size);         \
  template void HuffmanDecode<D, T, S, Q, H, mgard_cuda::CUDA>(                                  \
      mgard_cuda::Handle<D, T> & handle, S * &dqv, size_t & n, H * dmeta,      \
      size_t dmeta_size, H * ddata, size_t ddata_size);

KERNELS(1, double, int, uint32_t, uint32_t)
KERNELS(1, float, int, uint32_t, uint32_t)
KERNELS(2, double, int, uint32_t, uint32_t)
KERNELS(2, float, int, uint32_t, uint32_t)
KERNELS(3, double, int, uint32_t, uint32_t)
KERNELS(3, float, int, uint32_t, uint32_t)
KERNELS(4, double, int, uint32_t, uint32_t)
KERNELS(4, float, int, uint32_t, uint32_t)
KERNELS(5, double, int, uint32_t, uint32_t)
KERNELS(5, float, int, uint32_t, uint32_t)
KERNELS(1, double, int, uint32_t, uint64_t)
KERNELS(1, float, int, uint32_t, uint64_t)
KERNELS(2, double, int, uint32_t, uint64_t)
KERNELS(2, float, int, uint32_t, uint64_t)
KERNELS(3, double, int, uint32_t, uint64_t)
KERNELS(3, float, int, uint32_t, uint64_t)
KERNELS(4, double, int, uint32_t, uint64_t)
KERNELS(4, float, int, uint32_t, uint64_t)
KERNELS(5, double, int, uint32_t, uint64_t)
KERNELS(5, float, int, uint32_t, uint64_t)

// clang-format off
