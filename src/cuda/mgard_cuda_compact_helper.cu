#include "hip/hip_runtime.h"
#include "cuda/mgard_cuda_common.h"
#include "cuda/mgard_cuda_common_internal.h"
#include "cuda/mgard_cuda_compact_helper.h"
#include <iomanip>
#include <iostream>

namespace mgard_cuda {
/* 3D Original to (2^k)+1 */
template <typename T>
__global__ void _org_to_pow2p1(int nrow, int ncol, int nfib, int nr, int nc,
                               int nf, int *dirow, int *dicol, int *difib,
                               T *dv, int lddv1, int lddv2, T *dcv, int lddcv1,
                               int lddcv2) {

  int r0 = blockIdx.z * blockDim.z + threadIdx.z;
  int c0 = blockIdx.y * blockDim.y + threadIdx.y;
  int f0 = blockIdx.x * blockDim.x + threadIdx.x;

  for (int r = r0; r < nr; r += blockDim.z * gridDim.z) {
    for (int c = c0; c < nc; c += blockDim.y * gridDim.y) {
      for (int f = f0; f < nf; f += blockDim.x * gridDim.x) {
        dcv[get_idx(lddcv1, lddcv2, r, c, f)] =
            dv[get_idx(lddv1, lddv2, dirow[r], dicol[c], difib[f])];
      }
    }
  }
}

template <typename T>
void org_to_pow2p1(mgard_cuda_handle<T> &handle, int nrow, int ncol, int nfib,
                   int nr, int nc, int nf, int *dirow, int *dicol, int *difib,
                   T *dv, int lddv1, int lddv2, T *dcv, int lddcv1, int lddcv2,
                   int queue_idx) {

  int B_adjusted = min(8, handle.B);
  int total_thread_z = nr;
  int total_thread_y = nc;
  int total_thread_x = nf;
  int tbz = min(B_adjusted, total_thread_z);
  int tby = min(B_adjusted, total_thread_y);
  int tbx = min(B_adjusted, total_thread_x);
  int gridz = ceil((float)total_thread_z / tbz);
  int gridy = ceil((float)total_thread_y / tby);
  int gridx = ceil((float)total_thread_x / tbx);
  dim3 threadsPerBlock(tbx, tby, tbz);
  dim3 blockPerGrid(gridx, gridy, gridz);
  _org_to_pow2p1<<<blockPerGrid, threadsPerBlock, 0,
                   *(hipStream_t *)handle.get(queue_idx)>>>(
      nrow, ncol, nfib, nr, nc, nf, dirow, dicol, difib, dv, lddv1, lddv2, dcv,
      lddcv1, lddcv2);
  gpuErrchk(hipGetLastError());
#ifdef MGARD_CUDA_DEBUG
  gpuErrchk(hipDeviceSynchronize());
#endif
}

template void org_to_pow2p1<double>(mgard_cuda_handle<double> &handle, int nrow,
                                    int ncol, int nfib, int nr, int nc, int nf,
                                    int *dirow, int *dicol, int *difib,
                                    double *dv, int lddv1, int lddv2,
                                    double *dcv, int lddcv1, int lddcv2,
                                    int queue_idx);
template void org_to_pow2p1<float>(mgard_cuda_handle<float> &handle, int nrow,
                                   int ncol, int nfib, int nr, int nc, int nf,
                                   int *dirow, int *dicol, int *difib,
                                   float *dv, int lddv1, int lddv2, float *dcv,
                                   int lddcv1, int lddcv2, int queue_idx);

/* 2D Original to (2^k)+1 */
template <typename T>
__global__ void _org_to_pow2p1(int nrow, int ncol, int nr, int nc, int *dirow,
                               int *dicol, T *dv, int lddv, T *dcv, int lddcv) {

  int y0 = blockIdx.y * blockDim.y + threadIdx.y;
  int x0 = blockIdx.x * blockDim.x + threadIdx.x;
  for (int y = y0; y < nr; y += blockDim.y * gridDim.y) {
    for (int x = x0; x < nc; x += blockDim.x * gridDim.x) {
      dcv[get_idx(lddcv, y, x)] = dv[get_idx(lddv, dirow[y], dicol[x])];
    }
  }
}

template <typename T>
void org_to_pow2p1(mgard_cuda_handle<T> &handle, int nrow, int ncol, int nr,
                   int nc, int *dirow, int *dicol, T *dv, int lddv, T *dcv,
                   int lddcv, int queue_idx) {

  int total_thread_y = nr;
  int total_thread_x = nc;
  int tby = min(handle.B, total_thread_y);
  int tbx = min(handle.B, total_thread_x);
  int gridy = ceil((float)total_thread_y / tby);
  int gridx = ceil((float)total_thread_x / tbx);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);
  _org_to_pow2p1<<<blockPerGrid, threadsPerBlock, 0,
                   *(hipStream_t *)handle.get(queue_idx)>>>(
      nrow, ncol, nr, nc, dirow, dicol, dv, lddv, dcv, lddcv);

  gpuErrchk(hipGetLastError());
#ifdef MGARD_CUDA_DEBUG
  gpuErrchk(hipDeviceSynchronize());
#endif
}

template void org_to_pow2p1<double>(mgard_cuda_handle<double> &handle, int nrow,
                                    int ncol, int nr, int nc, int *dirow,
                                    int *dicol, double *dv, int lddv,
                                    double *dcv, int lddcv, int queue_idx);
template void org_to_pow2p1<float>(mgard_cuda_handle<float> &handle, int nrow,
                                   int ncol, int nr, int nc, int *dirow,
                                   int *dicol, float *dv, int lddv, float *dcv,
                                   int lddcv, int queue_idx);

/* 1D Original to (2^k)+1 */
template <typename T>
__global__ void _org_to_pow2p1(int nrow, int nr, int *dirow, T *dv, T *dcv) {

  int x0 = blockIdx.x * blockDim.x + threadIdx.x;
  for (int x = x0; x < nr; x += blockDim.x * gridDim.x) {
    dcv[x] = dv[dirow[x]];
  }
}

template <typename T>
void org_to_pow2p1(mgard_cuda_handle<T> &handle, int nrow, int nr, int *dirow,
                   T *dv, T *dcv, int queue_idx) {

  int total_thread_y = 1;
  int total_thread_x = nr;
  int tby = 1;
  int tbx = min(handle.B, total_thread_x);
  int gridy = ceil((float)total_thread_y / tby);
  int gridx = ceil((float)total_thread_x / tbx);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);
  _org_to_pow2p1<<<blockPerGrid, threadsPerBlock, 0,
                   *(hipStream_t *)handle.get(queue_idx)>>>(nrow, nr, dirow,
                                                             dv, dcv);

  gpuErrchk(hipGetLastError());
#ifdef MGARD_CUDA_DEBUG
  gpuErrchk(hipDeviceSynchronize());
#endif
}

template void org_to_pow2p1<double>(mgard_cuda_handle<double> &handle, int nrow,
                                    int nr, int *dirow, double *dv, double *dcv,
                                    int queue_idx);
template void org_to_pow2p1<float>(mgard_cuda_handle<float> &handle, int nrow,
                                   int nr, int *dirow, float *dv, float *dcv,
                                   int queue_idx);

/* 3D (2^k)+1 to original*/
template <typename T>
__global__ void _pow2p1_to_org(int nrow, int ncol, int nfib, int nr, int nc,
                               int nf, int *dirow, int *dicol, int *difib,
                               T *dcv, int lddcv1, int lddcv2, T *dv, int lddv1,
                               int lddv2) {

  int r0 = blockIdx.z * blockDim.z + threadIdx.z;
  int c0 = blockIdx.y * blockDim.y + threadIdx.y;
  int f0 = blockIdx.x * blockDim.x + threadIdx.x;
  for (int r = r0; r < nr; r += blockDim.z * gridDim.z) {
    for (int c = c0; c < nc; c += blockDim.y * gridDim.y) {
      for (int f = f0; f < nf; f += blockDim.x * gridDim.x) {
        dv[get_idx(lddv1, lddv2, dirow[r], dicol[c], difib[f])] =
            dcv[get_idx(lddcv1, lddcv2, r, c, f)];
      }
    }
  }
}

template <typename T>
void pow2p1_to_org(mgard_cuda_handle<T> &handle, int nrow, int ncol, int nfib,
                   int nr, int nc, int nf, int *dirow, int *dicol, int *difib,
                   T *dcv, int lddcv1, int lddcv2, T *dv, int lddv1, int lddv2,
                   int queue_idx) {

  int B_adjusted = min(8, handle.B);
  int total_thread_z = nr;
  int total_thread_y = nc;
  int total_thread_x = nf;
  int tbz = min(B_adjusted, total_thread_z);
  int tby = min(B_adjusted, total_thread_y);
  int tbx = min(B_adjusted, total_thread_x);
  int gridz = ceil((float)total_thread_z / tbz);
  int gridy = ceil((float)total_thread_y / tby);
  int gridx = ceil((float)total_thread_x / tbx);
  dim3 threadsPerBlock(tbx, tby, tbz);
  dim3 blockPerGrid(gridx, gridy, gridz);
  _pow2p1_to_org<<<blockPerGrid, threadsPerBlock, 0,
                   *(hipStream_t *)handle.get(queue_idx)>>>(
      nrow, ncol, nfib, nr, nc, nf, dirow, dicol, difib, dcv, lddcv1, lddcv2,
      dv, lddv1, lddv2);
  gpuErrchk(hipGetLastError());
#ifdef MGARD_CUDA_DEBUG
  gpuErrchk(hipDeviceSynchronize());
#endif
}

template void pow2p1_to_org<double>(mgard_cuda_handle<double> &handle, int nrow,
                                    int ncol, int nfib, int nr, int nc, int nf,
                                    int *dirow, int *dicol, int *difib,
                                    double *dcv, int lddcv1, int lddcv2,
                                    double *dv, int lddv1, int lddv2,
                                    int queue_idx);
template void pow2p1_to_org<float>(mgard_cuda_handle<float> &handle, int nrow,
                                   int ncol, int nfib, int nr, int nc, int nf,
                                   int *dirow, int *dicol, int *difib,
                                   float *dcv, int lddcv1, int lddcv2,
                                   float *dv, int lddv1, int lddv2,
                                   int queue_idx);

/* 2D (2^k)+1 to original*/
template <typename T>
__global__ void _pow2p1_to_org(int nrow, int ncol, int nr, int nc, int *dirow,
                               int *dicol, T *dcv, int lddcv, T *dv, int lddv) {

  int y0 = blockIdx.y * blockDim.y + threadIdx.y;
  int x0 = blockIdx.x * blockDim.x + threadIdx.x;
  for (int y = y0; y < nr; y += blockDim.y * gridDim.y) {
    for (int x = x0; x < nc; x += blockDim.x * gridDim.x) {
      dv[get_idx(lddv, dirow[y], dicol[x])] = dcv[get_idx(lddcv, y, x)];
    }
  }
}

template <typename T>
void pow2p1_to_org(mgard_cuda_handle<T> &handle, int nrow, int ncol, int nr,
                   int nc, int *dirow, int *dicol, T *dcv, int lddcv, T *dv,
                   int lddv, int queue_idx) {

  int total_thread_y = nr;
  int total_thread_x = nc;
  int tby = min(handle.B, total_thread_y);
  int tbx = min(handle.B, total_thread_x);
  int gridy = ceil((float)total_thread_y / tby);
  int gridx = ceil((float)total_thread_x / tbx);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);
  _pow2p1_to_org<<<blockPerGrid, threadsPerBlock, 0,
                   *(hipStream_t *)handle.get(queue_idx)>>>(
      nrow, ncol, nr, nc, dirow, dicol, dcv, lddcv, dv, lddv);

  gpuErrchk(hipGetLastError());
#ifdef MGARD_CUDA_DEBUG
  gpuErrchk(hipDeviceSynchronize());
#endif
}

template void pow2p1_to_org<double>(mgard_cuda_handle<double> &handle, int nrow,
                                    int ncol, int nr, int nc, int *dirow,
                                    int *dicol, double *dcv, int lddcv,
                                    double *dv, int lddv, int queue_idx);
template void pow2p1_to_org<float>(mgard_cuda_handle<float> &handle, int nrow,
                                   int ncol, int nr, int nc, int *dirow,
                                   int *dicol, float *dcv, int lddcv, float *dv,
                                   int lddv, int queue_idx);

/* 1D (2^k)+1 to original */
template <typename T>
__global__ void _pow2p1_to_org(int nrow, int nr, int *dirow, T *dcv, T *dv) {
  int x0 = blockIdx.x * blockDim.x + threadIdx.x;
  for (int x = x0; x < nr; x += blockDim.x * gridDim.x) {
    dv[dirow[x]] = dcv[x];
  }
}

template <typename T>
void pow2p1_to_org(mgard_cuda_handle<T> &handle, int nrow, int nr, int *dirow,
                   T *dcv, T *dv, int queue_idx) {

  int total_thread_y = 1;
  int total_thread_x = nr;
  int tby = 1;
  int tbx = min(handle.B, total_thread_x);
  int gridy = ceil((float)total_thread_y / tby);
  int gridx = ceil((float)total_thread_x / tbx);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);
  _pow2p1_to_org<<<blockPerGrid, threadsPerBlock, 0,
                   *(hipStream_t *)handle.get(queue_idx)>>>(nrow, nr, dirow,
                                                             dcv, dv);

  gpuErrchk(hipGetLastError());
#ifdef MGARD_CUDA_DEBUG
  gpuErrchk(hipDeviceSynchronize());
#endif
}

template void pow2p1_to_org<double>(mgard_cuda_handle<double> &handle, int nrow,
                                    int nr, int *dirow, double *dcv, double *dv,
                                    int queue_idx);
template void pow2p1_to_org<float>(mgard_cuda_handle<float> &handle, int nrow,
                                   int nr, int *dirow, float *dcv, float *dv,
                                   int queue_idx);

/* 3D (2^k)+1 to compact */
template <typename T>
__global__ void _pow2p1_to_cpt(int nrow, int ncol, int nfib, int row_stride,
                               int col_stride, int fib_stride, T *dv, int lddv1,
                               int lddv2, T *dcv, int lddcv1, int lddcv2) {
  int r0 = blockIdx.z * blockDim.z + threadIdx.z;
  int c0 = blockIdx.y * blockDim.y + threadIdx.y;
  int f0 = blockIdx.x * blockDim.x + threadIdx.x;
  for (int r = r0; r * row_stride < nrow; r += blockDim.z * gridDim.z) {
    for (int c = c0; c * col_stride < ncol; c += blockDim.y * gridDim.y) {
      for (int f = f0; f * fib_stride < nfib; f += blockDim.x * gridDim.x) {
        int r_strided = r * row_stride;
        int c_strided = c * col_stride;
        int f_strided = f * fib_stride;
        dcv[get_idx(lddcv1, lddcv2, r, c, f)] =
            dv[get_idx(lddv1, lddv2, r_strided, c_strided, f_strided)];
      }
    }
  }
}

template <typename T>
void pow2p1_to_cpt(mgard_cuda_handle<T> &handle, int nrow, int ncol, int nfib,
                   int row_stride, int col_stride, int fib_stride, T *dv,
                   int lddv1, int lddv2, T *dcv, int lddcv1, int lddcv2,
                   int queue_idx) {

  int B_adjusted = min(8, handle.B);
  int total_thread_z = ceil((float)nrow / row_stride);
  int total_thread_y = ceil((float)ncol / col_stride);
  int total_thread_x = ceil((float)nfib / fib_stride);
  int tbz = min(B_adjusted, total_thread_z);
  int tby = min(B_adjusted, total_thread_y);
  int tbx = min(B_adjusted, total_thread_x);
  int gridz = ceil((float)total_thread_z / tbz);
  int gridy = ceil((float)total_thread_y / tby);
  int gridx = ceil((float)total_thread_x / tbx);
  dim3 threadsPerBlock(tbx, tby, tbz);
  dim3 blockPerGrid(gridx, gridy, gridz);
  _pow2p1_to_cpt<<<blockPerGrid, threadsPerBlock, 0,
                   *(hipStream_t *)handle.get(queue_idx)>>>(
      nrow, ncol, nfib, row_stride, col_stride, fib_stride, dv, lddv1, lddv2,
      dcv, lddcv1, lddcv2);

  gpuErrchk(hipGetLastError());
#ifdef MGARD_CUDA_DEBUG
  gpuErrchk(hipDeviceSynchronize());
#endif
}

template void pow2p1_to_cpt<double>(mgard_cuda_handle<double> &handle, int nrow,
                                    int ncol, int nfib, int row_stride,
                                    int col_stride, int fib_stride, double *dv,
                                    int lddv1, int lddv2, double *dcv,
                                    int lddcv1, int lddcv2, int queue_idx);
template void pow2p1_to_cpt<float>(mgard_cuda_handle<float> &handle, int nrow,
                                   int ncol, int nfib, int row_stride,
                                   int col_stride, int fib_stride, float *dv,
                                   int lddv1, int lddv2, float *dcv, int lddcv1,
                                   int lddcv2, int queue_idx);

/* 2D (2^k)+1 to compact */
template <typename T>
__global__ void _pow2p1_to_cpt(int nrow, int ncol, int row_stride,
                               int col_stride, T *dv, int lddv, T *dcv,
                               int lddcv) {
  int x0 = blockIdx.x * blockDim.x + threadIdx.x;
  int y0 = blockIdx.y * blockDim.y + threadIdx.y;
  for (int y = y0; y * row_stride < nrow; y += blockDim.y * gridDim.y) {
    for (int x = x0; x * col_stride < ncol; x += blockDim.x * gridDim.x) {
      int x_strided = x * col_stride;
      int y_strided = y * row_stride;
      // printf("load dv[%d, %d] = %f\n", y_strided, x_strided, dv[get_idx(lddv,
      // y_strided, x_strided)]);
      dcv[get_idx(lddcv, y, x)] = dv[get_idx(lddv, y_strided, x_strided)];
    }
  }
}

template <typename T>
void pow2p1_to_cpt(mgard_cuda_handle<T> &handle, int nrow, int ncol,
                   int row_stride, int col_stride, T *dv, int lddv, T *dcv,
                   int lddcv, int queue_idx) {

  int total_thread_y = ceil((float)nrow / row_stride);
  int total_thread_x = ceil((float)ncol / col_stride);
  int tby = min(handle.B, total_thread_y);
  int tbx = min(handle.B, total_thread_x);
  int gridy = ceil((float)total_thread_y / tby);
  int gridx = ceil((float)total_thread_x / tbx);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);
  _pow2p1_to_cpt<<<blockPerGrid, threadsPerBlock, 0,
                   *(hipStream_t *)handle.get(queue_idx)>>>(
      nrow, ncol, row_stride, col_stride, dv, lddv, dcv, lddcv);
  gpuErrchk(hipGetLastError());
#ifdef MGARD_CUDA_DEBUG
  gpuErrchk(hipDeviceSynchronize());
#endif
}

template void pow2p1_to_cpt<double>(mgard_cuda_handle<double> &handle, int nrow,
                                    int ncol, int row_stride, int col_stride,
                                    double *dv, int lddv, double *dcv,
                                    int lddcv, int queue_idx);
template void pow2p1_to_cpt<float>(mgard_cuda_handle<float> &handle, int nrow,
                                   int ncol, int row_stride, int col_stride,
                                   float *dv, int lddv, float *dcv, int lddcv,
                                   int queue_idx);

/* 2D (2^k)+1 to compact with number assign */
template <typename T>
__global__ void _pow2p1_to_cpt_num_assign(int nrow, int ncol, int row_stride,
                                          int col_stride, T val, T *dv,
                                          int lddv, T *dcv, int lddcv) {
  int x0 = blockIdx.x * blockDim.x + threadIdx.x;
  int y0 = blockIdx.y * blockDim.y + threadIdx.y;
  for (int y = y0; y * row_stride < nrow; y += blockDim.y * gridDim.y) {
    for (int x = x0; x * col_stride < ncol; x += blockDim.x * gridDim.x) {
      int x_strided = x * col_stride;
      int y_strided = y * row_stride;
      if (y % 2 == 0 && x % 2 == 0)
        dcv[get_idx(lddcv, y, x)] = val;
      else
        dcv[get_idx(lddcv, y, x)] = dv[get_idx(lddv, y_strided, x_strided)];
    }
  }
}

template <typename T>
void pow2p1_to_cpt_num_assign(mgard_cuda_handle<T> &handle, int nrow, int ncol,
                              int row_stride, int col_stride, T val, T *dv,
                              int lddv, T *dcv, int lddcv, int queue_idx) {

  int total_thread_y = ceil((float)nrow / row_stride);
  int total_thread_x = ceil((float)ncol / col_stride);
  int tby = min(handle.B, total_thread_y);
  int tbx = min(handle.B, total_thread_x);
  int gridy = ceil((float)total_thread_y / tby);
  int gridx = ceil((float)total_thread_x / tbx);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);
  _pow2p1_to_cpt_num_assign<<<blockPerGrid, threadsPerBlock, 0,
                              *(hipStream_t *)handle.get(queue_idx)>>>(
      nrow, ncol, row_stride, col_stride, val, dv, lddv, dcv, lddcv);
  gpuErrchk(hipGetLastError());
#ifdef MGARD_CUDA_DEBUG
  gpuErrchk(hipDeviceSynchronize());
#endif
}

template void
pow2p1_to_cpt_num_assign<double>(mgard_cuda_handle<double> &handle, int nrow,
                                 int ncol, int row_stride, int col_stride,
                                 double val, double *dv, int lddv, double *dcv,
                                 int lddcv, int queue_idx);
template void pow2p1_to_cpt_num_assign<float>(mgard_cuda_handle<float> &handle,
                                              int nrow, int ncol,
                                              int row_stride, int col_stride,
                                              float val, float *dv, int lddv,
                                              float *dcv, int lddcv,
                                              int queue_idx);

/* 1D (2^k)+1 to compact */
template <typename T>
__global__ void _pow2p1_to_cpt(int nrow, int row_stride, T *dv, T *dcv) {
  int x0 = blockIdx.x * blockDim.x + threadIdx.x;
  for (int x = x0; x * row_stride < nrow; x += blockDim.x * gridDim.x) {
    int x_strided = x * row_stride;
    dcv[x] = dv[x_strided];
  }
}

template <typename T>
void pow2p1_to_cpt(mgard_cuda_handle<T> &handle, int nrow, int row_stride,
                   T *dv, T *dcv, int queue_idx) {

  int total_thread_y = 1;
  int total_thread_x = ceil((float)nrow / row_stride);
  int tby = 1;
  int tbx = min(handle.B, total_thread_x);
  int gridy = ceil((float)total_thread_y / tby);
  int gridx = ceil((float)total_thread_x / tbx);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);
  _pow2p1_to_cpt<<<blockPerGrid, threadsPerBlock, 0,
                   *(hipStream_t *)handle.get(queue_idx)>>>(nrow, row_stride,
                                                             dv, dcv);
  gpuErrchk(hipGetLastError());
#ifdef MGARD_CUDA_DEBUG
  gpuErrchk(hipDeviceSynchronize());
#endif
}

template void pow2p1_to_cpt<double>(mgard_cuda_handle<double> &handle, int nrow,
                                    int row_stride, double *dv, double *dcv,
                                    int queue_idx);
template void pow2p1_to_cpt<float>(mgard_cuda_handle<float> &handle, int nrow,
                                   int row_stride, float *dv, float *dcv,
                                   int queue_idx);

/* 3D compact to (2^k)+1*/
template <typename T>
__global__ void _cpt_to_pow2p1(int nrow, int ncol, int nfib, int row_stride,
                               int col_stride, int fib_stride, T *dcv,
                               int lddcv1, int lddcv2, T *dv, int lddv1,
                               int lddv2) {
  int r0 = blockIdx.z * blockDim.z + threadIdx.z;
  int c0 = blockIdx.y * blockDim.y + threadIdx.y;
  int f0 = blockIdx.x * blockDim.x + threadIdx.x;
  for (int r = r0; r * row_stride < nrow; r += blockDim.z * gridDim.z) {
    for (int c = c0; c * col_stride < ncol; c += blockDim.y * gridDim.y) {
      for (int f = f0; f * fib_stride < nfib; f += blockDim.x * gridDim.x) {
        int r_strided = r * row_stride;
        int c_strided = c * col_stride;
        int f_strided = f * fib_stride;
        dv[get_idx(lddv1, lddv2, r_strided, c_strided, f_strided)] =
            dcv[get_idx(lddcv1, lddcv2, r, c, f)];
      }
    }
  }
}

template <typename T>
void cpt_to_pow2p1(mgard_cuda_handle<T> &handle, int nrow, int ncol, int nfib,
                   int row_stride, int col_stride, int fib_stride, T *dcv,
                   int lddcv1, int lddcv2, T *dv, int lddv1, int lddv2,
                   int queue_idx) {

  int B_adjusted = min(8, handle.B);
  int total_thread_z = ceil((float)nrow / row_stride);
  int total_thread_y = ceil((float)ncol / col_stride);
  int total_thread_x = ceil((float)nfib / fib_stride);
  int tbz = min(B_adjusted, total_thread_z);
  int tby = min(B_adjusted, total_thread_y);
  int tbx = min(B_adjusted, total_thread_x);
  int gridz = ceil((float)total_thread_z / tbz);
  int gridy = ceil((float)total_thread_y / tby);
  int gridx = ceil((float)total_thread_x / tbx);
  dim3 threadsPerBlock(tbx, tby, tbz);
  dim3 blockPerGrid(gridx, gridy, gridz);
  _cpt_to_pow2p1<<<blockPerGrid, threadsPerBlock, 0,
                   *(hipStream_t *)handle.get(queue_idx)>>>(
      nrow, ncol, nfib, row_stride, col_stride, fib_stride, dcv, lddcv1, lddcv2,
      dv, lddv1, lddv2);
  gpuErrchk(hipGetLastError());
#ifdef MGARD_CUDA_DEBUG
  gpuErrchk(hipDeviceSynchronize());
#endif
}

template void cpt_to_pow2p1<double>(mgard_cuda_handle<double> &handle, int nrow,
                                    int ncol, int nfib, int row_stride,
                                    int col_stride, int fib_stride, double *dcv,
                                    int lddcv1, int lddcv2, double *dv,
                                    int lddv1, int lddv2, int queue_idx);
template void cpt_to_pow2p1<float>(mgard_cuda_handle<float> &handle, int nrow,
                                   int ncol, int nfib, int row_stride,
                                   int col_stride, int fib_stride, float *dcv,
                                   int lddcv1, int lddcv2, float *dv, int lddv1,
                                   int lddv2, int queue_idx);

/* 2D compact to (2^k)+1*/
template <typename T>
__global__ void _cpt_to_pow2p1(int nrow, int ncol, int row_stride,
                               int col_stride, T *dcv, int lddcv, T *dv,
                               int lddv) {
  int x0 = blockIdx.x * blockDim.x + threadIdx.x;
  int y0 = blockIdx.y * blockDim.y + threadIdx.y;
  for (int y = y0; y * row_stride < nrow; y += blockDim.y * gridDim.y) {
    for (int x = x0; x * col_stride < ncol; x += blockDim.x * gridDim.x) {
      int x_strided = x * col_stride;
      int y_strided = y * row_stride;
      dv[get_idx(lddv, y_strided, x_strided)] = dcv[get_idx(lddcv, y, x)];
    }
  }
}

template <typename T>
void cpt_to_pow2p1(mgard_cuda_handle<T> &handle, int nrow, int ncol,
                   int row_stride, int col_stride, T *dcv, int lddcv, T *dv,
                   int lddv, int queue_idx) {

  int total_thread_x = ceil((float)nrow / row_stride);
  int total_thread_y = ceil((float)ncol / col_stride);
  int tbx = min(handle.B, total_thread_x);
  int tby = min(handle.B, total_thread_y);
  int gridx = ceil((float)total_thread_x / tbx);
  int gridy = ceil((float)total_thread_y / tby);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);
  _cpt_to_pow2p1<<<blockPerGrid, threadsPerBlock, 0,
                   *(hipStream_t *)handle.get(queue_idx)>>>(
      nrow, ncol, row_stride, col_stride, dcv, lddcv, dv, lddv);

  gpuErrchk(hipGetLastError());
#ifdef MGARD_CUDA_DEBUG
  gpuErrchk(hipDeviceSynchronize());
#endif
}

template void cpt_to_pow2p1<double>(mgard_cuda_handle<double> &handle, int nrow,
                                    int ncol, int row_stride, int col_stride,
                                    double *dcv, int lddcv, double *dv,
                                    int lddv, int queue_idx);
template void cpt_to_pow2p1<float>(mgard_cuda_handle<float> &handle, int nrow,
                                   int ncol, int row_stride, int col_stride,
                                   float *dcv, int lddcv, float *dv, int lddv,
                                   int queue_idx);

/* 2D compact to (2^k)+1 with add */
template <typename T>
__global__ void _cpt_to_pow2p1_add(int nrow, int ncol, int row_stride1,
                                   int col_stride1, int row_stride2,
                                   int col_stride2, T *dcv, int lddcv, T *dv,
                                   int lddv) {
  int x0 = blockIdx.x * blockDim.x + threadIdx.x;
  int y0 = blockIdx.y * blockDim.y + threadIdx.y;
  for (int y = y0; y * row_stride2 < nrow; y += blockDim.y * gridDim.y) {
    for (int x = x0; x * col_stride2 < ncol; x += blockDim.x * gridDim.x) {
      int x_strided1 = x * col_stride1;
      int y_strided1 = y * row_stride1;
      int x_strided2 = x * col_stride2;
      int y_strided2 = y * row_stride2;
      dv[get_idx(lddv, y_strided2, x_strided2)] +=
          dcv[get_idx(lddcv, y_strided1, x_strided1)];
    }
  }
}

template <typename T>
void cpt_to_pow2p1_add(mgard_cuda_handle<T> &handle, int nrow, int ncol,
                       int row_stride1, int col_stride1, int row_stride2,
                       int col_stride2, T *dcv, int lddcv, T *dv, int lddv,
                       int queue_idx) {

  int total_thread_x = ceil((float)nrow / row_stride2);
  int total_thread_y = ceil((float)ncol / col_stride2);
  int tbx = min(handle.B, total_thread_x);
  int tby = min(handle.B, total_thread_y);
  int gridx = ceil((float)total_thread_x / tbx);
  int gridy = ceil((float)total_thread_y / tby);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);
  _cpt_to_pow2p1_add<<<blockPerGrid, threadsPerBlock, 0,
                       *(hipStream_t *)handle.get(queue_idx)>>>(
      nrow, ncol, row_stride1, col_stride1, row_stride2, col_stride2, dcv,
      lddcv, dv, lddv);

  gpuErrchk(hipGetLastError());
#ifdef MGARD_CUDA_DEBUG
  gpuErrchk(hipDeviceSynchronize());
#endif
}

template void cpt_to_pow2p1_add<double>(mgard_cuda_handle<double> &handle,
                                        int nrow, int ncol, int row_stride1,
                                        int col_stride1, int row_stride2,
                                        int col_stride2, double *dcv, int lddcv,
                                        double *dv, int lddv, int queue_idx);
template void cpt_to_pow2p1_add<float>(mgard_cuda_handle<float> &handle,
                                       int nrow, int ncol, int row_stride1,
                                       int col_stride1, int row_stride2,
                                       int col_stride2, float *dcv, int lddcv,
                                       float *dv, int lddv, int queue_idx);

/* 2D compact to (2^k)+1 with add */
template <typename T>
__global__ void _cpt_to_pow2p1_subtract(int nrow, int ncol, int row_stride1,
                                        int col_stride1, int row_stride2,
                                        int col_stride2, T *dcv, int lddcv,
                                        T *dv, int lddv) {
  int x0 = blockIdx.x * blockDim.x + threadIdx.x;
  int y0 = blockIdx.y * blockDim.y + threadIdx.y;
  for (int y = y0; y * row_stride2 < nrow; y += blockDim.y * gridDim.y) {
    for (int x = x0; x * col_stride2 < ncol; x += blockDim.x * gridDim.x) {
      int x_strided1 = x * col_stride1;
      int y_strided1 = y * row_stride1;
      int x_strided2 = x * col_stride2;
      int y_strided2 = y * row_stride2;
      dv[get_idx(lddv, y_strided2, x_strided2)] -=
          dcv[get_idx(lddcv, y_strided1, x_strided1)];
    }
  }
}

template <typename T>
void cpt_to_pow2p1_subtract(mgard_cuda_handle<T> &handle, int nrow, int ncol,
                            int row_stride1, int col_stride1, int row_stride2,
                            int col_stride2, T *dcv, int lddcv, T *dv, int lddv,
                            int queue_idx) {

  int total_thread_x = ceil((float)nrow / row_stride2);
  int total_thread_y = ceil((float)ncol / col_stride2);
  int tbx = min(handle.B, total_thread_x);
  int tby = min(handle.B, total_thread_y);
  int gridx = ceil((float)total_thread_x / tbx);
  int gridy = ceil((float)total_thread_y / tby);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);
  _cpt_to_pow2p1_subtract<<<blockPerGrid, threadsPerBlock, 0,
                            *(hipStream_t *)handle.get(queue_idx)>>>(
      nrow, ncol, row_stride1, col_stride1, row_stride2, col_stride2, dcv,
      lddcv, dv, lddv);
  gpuErrchk(hipGetLastError());
#ifdef MGARD_CUDA_DEBUG
  gpuErrchk(hipDeviceSynchronize());
#endif
}

template void cpt_to_pow2p1_subtract<double>(mgard_cuda_handle<double> &handle,
                                             int nrow, int ncol,
                                             int row_stride1, int col_stride1,
                                             int row_stride2, int col_stride2,
                                             double *dcv, int lddcv, double *dv,
                                             int lddv, int queue_idx);
template void cpt_to_pow2p1_subtract<float>(mgard_cuda_handle<float> &handle,
                                            int nrow, int ncol, int row_stride1,
                                            int col_stride1, int row_stride2,
                                            int col_stride2, float *dcv,
                                            int lddcv, float *dv, int lddv,
                                            int queue_idx);

/* 1D compact to (2^k)+1*/
template <typename T>
__global__ void _cpt_to_pow2p1(int nrow, int row_stride, T *dcv, T *dv) {
  int x0 = blockIdx.x * blockDim.x + threadIdx.x;
  for (int x = x0; x * row_stride < nrow; x += blockDim.x * gridDim.x) {
    int x_strided = x * row_stride;
    dv[x_strided] = dcv[x];
  }
}

template <typename T>
void cpt_to_pow2p1(mgard_cuda_handle<T> &handle, int nrow, int row_stride,
                   T *dcv, T *dv, int B, int queue_idx) {

  int total_thread_x = ceil((float)nrow / row_stride);
  int total_thread_y = 1;
  int tbx = min(B, total_thread_x);
  int tby = 1;
  int gridx = ceil((float)total_thread_x / tbx);
  int gridy = ceil((float)total_thread_y / tby);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);
  _cpt_to_pow2p1<<<blockPerGrid, threadsPerBlock, 0,
                   *(hipStream_t *)handle.get(queue_idx)>>>(nrow, row_stride,
                                                             dcv, dv);

  gpuErrchk(hipGetLastError());
#ifdef MGARD_CUDA_DEBUG
  gpuErrchk(hipDeviceSynchronize());
#endif
}

template void cpt_to_pow2p1<double>(mgard_cuda_handle<double> &handle, int nrow,
                                    int row_stride, double *dcv, double *dv,
                                    int B, int queue_idx);
template void cpt_to_pow2p1<float>(mgard_cuda_handle<float> &handle, int nrow,
                                   int row_stride, float *dcv, float *dv, int B,
                                   int queue_idx);

template <typename T> __device__ T _dist(T *dcoord, int x, int y) {
  return dcoord[y] - dcoord[x];
}

template <typename T>
__global__ void _calc_cpt_dist(int n, int stride, T *dcoord, T *ddist) {

  // extern __shared__ __align__(sizeof(T)) unsigned char smem[];
  // T * sm = reinterpret_cast<T *>(smem);
  T *sm = SharedMemory<T>();
  // extern __shared__ double sm[]; //size = blockDim.x + 1

  int x0 = blockIdx.x * blockDim.x + threadIdx.x;
  int x0_sm = threadIdx.x;
  T dist;
  for (int x = x0; x * stride < n - 1; x += blockDim.x * gridDim.x) {
    // Load coordinates
    sm[x0_sm] = dcoord[x * stride];
    // printf("sm[%d] block %d thread %d load[%d] %f\n", x0_sm, blockIdx.x,
    // threadIdx.x, x, dcoord[x * stride]);
    if (x0_sm == 0) {
      // sm[blockDim.x] = dcoord[(x + blockDim.x) * stride];
      int left = (n - 1) / stride + 1 - blockIdx.x * blockDim.x;
      sm[min(blockDim.x, left - 1)] =
          dcoord[min((x + blockDim.x) * stride, n - 1)];
      // printf("sm[%d] extra block %d thread %d load[%d] %f\n", min(blockDim.x,
      // left-1), blockIdx.x, threadIdx.x, min((x + blockDim.x) * stride, n-1),
      // dcoord[min((x + blockDim.x) * stride, n-1)]);
    }
    __syncthreads();

    // Compute distance
    dist = _get_dist(sm, x0_sm, x0_sm + 1);
    __syncthreads();
    ddist[x] = dist;
    __syncthreads();
  }
}

template <typename T>
void calc_cpt_dist(mgard_cuda_handle<T> &handle, int nrow, int row_stride,
                   T *dcoord, T *ddist, int queue_idx) {

  int total_thread_x = ceil((float)nrow / row_stride) - 1;
  int total_thread_y = 1;
  int tbx = min(handle.B, total_thread_x);
  int tby = 1;
  int gridx = ceil((float)total_thread_x / tbx);
  int gridy = ceil((float)total_thread_y / tby);
  dim3 threadsPerBlock(tbx, tby);
  dim3 blockPerGrid(gridx, gridy);
  size_t sm_size = (tbx + 1) * sizeof(T);
  _calc_cpt_dist<<<blockPerGrid, threadsPerBlock, sm_size,
                   *(hipStream_t *)handle.get(queue_idx)>>>(nrow, row_stride,
                                                             dcoord, ddist);
  gpuErrchk(hipGetLastError());
#ifdef MGARD_CUDA_DEBUG
  gpuErrchk(hipDeviceSynchronize());
#endif
}

template void calc_cpt_dist<double>(mgard_cuda_handle<double> &handle, int nrow,
                                    int row_stride, double *dcoord,
                                    double *ddist, int queue_idx);
template void calc_cpt_dist<float>(mgard_cuda_handle<float> &handle, int nrow,
                                   int row_stride, float *dcoord, float *ddist,
                                   int queue_idx);
} // namespace mgard_cuda