#include "hip/hip_runtime.h"
/*
 * Copyright 2021, Oak Ridge National Laboratory.
 * MGARD-GPU: MultiGrid Adaptive Reduction of Data Accelerated by GPUs
 * Author: Jieyang Chen (chenj3@ornl.gov)
 * Date: September 27, 2021
 */

#include <chrono>
#include <iomanip>
#include <iostream>
#include <numeric>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <vector>

#include "cuda/CommonInternal.h"

// #include "cuda/CompressionWorkflow.h"
#include "compress_cuda.hpp"

#include "cuda/MemoryManagement.h"

#include "cuda/DataRefactoring.h"
#include "cuda/LinearQuantization.h"
#include "cuda/LosslessCompression.h"

namespace mgard_cuda {

bool verify(const void *compressed_data, size_t compressed_size) {
  char magic_word[MAGIC_WORD_SIZE + 1];
  if (compressed_size < sizeof(magic_word))
    return false;
  SIZE meta_size = *(SIZE *)compressed_data;
  Metadata meta;
  meta.Deserialize((SERIALIZED_TYPE *)compressed_data, meta_size);
  std::memcpy(magic_word, meta.magic_word, MAGIC_WORD_SIZE);
  magic_word[MAGIC_WORD_SIZE] = '\0';
  if (strcmp(magic_word, MAGIC_WORD) == 0) {
    return true;
  } else {
    return false;
  }
}

enum data_type infer_type(const void *compressed_data, size_t compressed_size) {
  if (!verify(compressed_data, compressed_size)) {
    std::cout << log::log_err << "cannot verify the data!\n";
    exit(-1);
  }
  Metadata meta;
  SIZE meta_size = *(SIZE *)compressed_data + meta.metadata_size_offset();
  meta.Deserialize((SERIALIZED_TYPE *)compressed_data, meta_size);
  return meta.dtype;
}

std::vector<SIZE> infer_shape(const void *compressed_data,
                              size_t compressed_size) {
  if (!verify(compressed_data, compressed_size)) {
    std::cout << log::log_err << "cannot verify the data!\n";
    exit(-1);
  }
  Metadata meta;
  uint32_t meta_size =
      *(uint32_t *)compressed_data + meta.metadata_size_offset();
  meta.Deserialize((SERIALIZED_TYPE *)compressed_data, meta_size);
  std::vector<SIZE> shape(meta.total_dims);
  for (DIM d = 0; d < meta.total_dims; d++) {
    shape[d] = (SIZE)(*(meta.shape + d));
  }
  return shape;
}

enum data_structure_type infer_data_structure(const void *compressed_data,
                                              size_t compressed_size) {
  if (!verify(compressed_data, compressed_size)) {
    std::cout << log::log_err << "cannot verify the data!\n";
    exit(-1);
  }
  Metadata meta;
  uint32_t meta_size =
      *(uint32_t *)compressed_data + meta.metadata_size_offset();
  meta.Deserialize((SERIALIZED_TYPE *)compressed_data, meta_size);
  return meta.dstype;
}

std::string infer_nonuniform_coords_file(const void *compressed_data,
                                         size_t compressed_size) {
  if (!verify(compressed_data, compressed_size)) {
    std::cout << log::log_err << "cannot verify the data!\n";
    exit(-1);
  }
  Metadata meta;
  uint32_t meta_size =
      *(uint32_t *)compressed_data + meta.metadata_size_offset();
  meta.Deserialize((SERIALIZED_TYPE *)compressed_data, meta_size);
  return std::string(meta.nonuniform_coords_file);
}

template <DIM D, typename T>
void compress(std::vector<SIZE> shape, T tol, T s, enum error_bound_type mode,
              const void *original_data, void *&compressed_data,
              size_t &compressed_size, Config config, bool isAllocated) {
  Handle<D, T> handle(shape, config);
  mgard_cuda::Array<D, T> in_array(shape);
  in_array.loadData((const T *)original_data);
  Array<1, unsigned char> compressed_array =
      compress(handle, in_array, mode, tol, s);
  compressed_size = compressed_array.getShape()[0];
  if (!isAllocated)
    compressed_data = (void *)std::malloc(compressed_size);
  std::memcpy(compressed_data, compressed_array.getDataHost(), compressed_size);
}

template <DIM D, typename T>
void compress(std::vector<SIZE> shape, T tol, T s, enum error_bound_type mode,
              const void *original_data, void *&compressed_data,
              size_t &compressed_size, Config config, bool isAllocated,
              std::vector<T *> coords) {
  Handle<D, T> handle(shape, coords, config);
  mgard_cuda::Array<D, T> in_array(shape);
  in_array.loadData((const T *)original_data);
  Array<1, unsigned char> compressed_array =
      compress(handle, in_array, mode, tol, s);
  compressed_size = compressed_array.getShape()[0];
  if (!isAllocated)
    compressed_data = (void *)std::malloc(compressed_size);
  std::memcpy(compressed_data, compressed_array.getDataHost(), compressed_size);
}

template <DIM D, typename T>
void decompress(std::vector<SIZE> shape, const void *compressed_data,
                size_t compressed_size, void *&decompressed_data, Config config,
                bool isAllocated) {
  size_t original_size = 1;
  for (int i = 0; i < D; i++)
    original_size *= shape[i];
  Handle<D, T> handle(shape, config);
  std::vector<SIZE> compressed_shape(1);
  compressed_shape[0] = compressed_size;
  Array<1, unsigned char> compressed_array(compressed_shape);
  compressed_array.loadData((const unsigned char *)compressed_data);
  Array<D, T> out_array = decompress(handle, compressed_array);
  if (!isAllocated)
    decompressed_data = (void *)std::malloc(original_size * sizeof(T));
  std::memcpy(decompressed_data, out_array.getDataHost(),
              original_size * sizeof(T));
}

void compress(DIM D, data_type dtype, std::vector<SIZE> shape, double tol,
              double s, enum error_bound_type mode, const void *original_data,
              void *&compressed_data, size_t &compressed_size, Config config,
              bool isAllocated) {
  if (dtype == data_type::Float) {
    if (D == 1) {
      compress<1, float>(shape, tol, s, mode, original_data, compressed_data,
                         compressed_size, config, isAllocated);
    } else if (D == 2) {
      compress<2, float>(shape, tol, s, mode, original_data, compressed_data,
                         compressed_size, config, isAllocated);
    } else if (D == 3) {
      compress<3, float>(shape, tol, s, mode, original_data, compressed_data,
                         compressed_size, config, isAllocated);
    } else if (D == 4) {
      compress<4, float>(shape, tol, s, mode, original_data, compressed_data,
                         compressed_size, config, isAllocated);
    } else if (D == 5) {
      compress<5, float>(shape, tol, s, mode, original_data, compressed_data,
                         compressed_size, config, isAllocated);
    } else {
      std::cout << log::log_err
                << "do not support higher than five dimentions!\n";
      exit(-1);
    }
  } else if (dtype == data_type::Double) {
    if (D == 1) {
      compress<1, double>(shape, tol, s, mode, original_data, compressed_data,
                          compressed_size, config, isAllocated);
    } else if (D == 2) {
      compress<2, double>(shape, tol, s, mode, original_data, compressed_data,
                          compressed_size, config, isAllocated);
    } else if (D == 3) {
      compress<3, double>(shape, tol, s, mode, original_data, compressed_data,
                          compressed_size, config, isAllocated);
    } else if (D == 4) {
      compress<4, double>(shape, tol, s, mode, original_data, compressed_data,
                          compressed_size, config, isAllocated);
    } else if (D == 5) {
      compress<5, double>(shape, tol, s, mode, original_data, compressed_data,
                          compressed_size, config, isAllocated);
    } else {
      std::cout << log::log_err
                << "do not support higher than five dimentions!\n";
      exit(-1);
    }
  } else {
    std::cout << log::log_err
              << "do not support types other than double and float!\n";
    exit(-1);
  }
}

void compress(DIM D, data_type dtype, std::vector<SIZE> shape, double tol,
              double s, enum error_bound_type mode, const void *original_data,
              void *&compressed_data, size_t &compressed_size, Config config,
              bool isAllocated, std::vector<const Byte *> coords) {
  if (dtype == data_type::Float) {
    std::vector<float *> float_coords;
    for (auto &coord : coords)
      float_coords.push_back((float *)coord);
    if (D == 1) {
      compress<1, float>(shape, tol, s, mode, original_data, compressed_data,
                         compressed_size, config, isAllocated, float_coords);
    } else if (D == 2) {
      compress<2, float>(shape, tol, s, mode, original_data, compressed_data,
                         compressed_size, config, isAllocated, float_coords);
    } else if (D == 3) {
      compress<3, float>(shape, tol, s, mode, original_data, compressed_data,
                         compressed_size, config, isAllocated, float_coords);
    } else if (D == 4) {
      compress<4, float>(shape, tol, s, mode, original_data, compressed_data,
                         compressed_size, config, isAllocated, float_coords);
    } else if (D == 5) {
      compress<5, float>(shape, tol, s, mode, original_data, compressed_data,
                         compressed_size, config, isAllocated, float_coords);
    } else {
      std::cout << log::log_err
                << "do not support higher than five dimentions!\n";
      exit(-1);
    }
  } else if (dtype == data_type::Double) {
    std::vector<double *> double_coords;
    for (auto &coord : coords)
      double_coords.push_back((double *)coord);
    if (D == 1) {
      compress<1, double>(shape, tol, s, mode, original_data, compressed_data,
                          compressed_size, config, isAllocated, double_coords);
    } else if (D == 2) {
      compress<2, double>(shape, tol, s, mode, original_data, compressed_data,
                          compressed_size, config, isAllocated, double_coords);
    } else if (D == 3) {
      compress<3, double>(shape, tol, s, mode, original_data, compressed_data,
                          compressed_size, config, isAllocated, double_coords);
    } else if (D == 4) {
      compress<4, double>(shape, tol, s, mode, original_data, compressed_data,
                          compressed_size, config, isAllocated, double_coords);
    } else if (D == 5) {
      compress<5, double>(shape, tol, s, mode, original_data, compressed_data,
                          compressed_size, config, isAllocated, double_coords);
    } else {
      std::cout << log::log_err
                << "do not support higher than five dimentions!\n";
      exit(-1);
    }
  } else {
    std::cout << log::log_err
              << "do not support types other than double and float!\n";
    exit(-1);
  }
}

void decompress(const void *compressed_data, size_t compressed_size,
                void *&decompressed_data, Config config, bool isAllocated) {
  std::vector<mgard_cuda::SIZE> shape =
      mgard_cuda::infer_shape(compressed_data, compressed_size);
  mgard_cuda::data_type dtype =
      mgard_cuda::infer_type(compressed_data, compressed_size);

  if (dtype == data_type::Float) {
    if (shape.size() == 1) {
      decompress<1, float>(shape, compressed_data, compressed_size,
                           decompressed_data, config, isAllocated);
    } else if (shape.size() == 2) {
      decompress<2, float>(shape, compressed_data, compressed_size,
                           decompressed_data, config, isAllocated);
    } else if (shape.size() == 3) {
      decompress<3, float>(shape, compressed_data, compressed_size,
                           decompressed_data, config, isAllocated);
    } else if (shape.size() == 4) {
      decompress<4, float>(shape, compressed_data, compressed_size,
                           decompressed_data, config, isAllocated);
    } else if (shape.size() == 5) {
      decompress<5, float>(shape, compressed_data, compressed_size,
                           decompressed_data, config, isAllocated);
    } else {
      std::cout << log::log_err
                << "do not support higher than five dimentions!\n";
      exit(-1);
    }
  } else if (dtype == data_type::Double) {
    if (shape.size() == 1) {
      decompress<1, double>(shape, compressed_data, compressed_size,
                            decompressed_data, config, isAllocated);
    } else if (shape.size() == 2) {
      decompress<2, double>(shape, compressed_data, compressed_size,
                            decompressed_data, config, isAllocated);
    } else if (shape.size() == 3) {
      decompress<3, double>(shape, compressed_data, compressed_size,
                            decompressed_data, config, isAllocated);
    } else if (shape.size() == 4) {
      decompress<4, double>(shape, compressed_data, compressed_size,
                            decompressed_data, config, isAllocated);
    } else if (shape.size() == 5) {
      decompress<5, double>(shape, compressed_data, compressed_size,
                            decompressed_data, config, isAllocated);
    } else {
      std::cout << log::log_err
                << "do not support higher than five dimentions!\n";
      exit(-1);
    }
  } else {
    std::cout << log::log_err
              << "do not support types other than double and float!\n";
    exit(-1);
  }
}

} // namespace mgard_cuda
