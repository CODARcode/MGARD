/*
 * Copyright 2021, Oak Ridge National Laboratory.
 * MGARD-GPU: MultiGrid Adaptive Reduction of Data Accelerated by GPUs
 * Author: Jieyang Chen (chenj3@ornl.gov)
 * Date: April 2, 2021
 */

#include "cuda/DataRefactoring.hpp"

#include <iostream>

#include <chrono>
namespace mgard_cuda {

#define KERNELS(D, T)                                                          \
  template void calc_coeff_pointers<D, T>(                                     \
      Handle<D, T> & handle, DIM curr_dims[3], DIM l, SubArray<D, T> doutput,  \
      SubArray<D, T> & dcoarse, SubArray<D, T> & dcoeff_f,                     \
      SubArray<D, T> & dcoeff_c, SubArray<D, T> & dcoeff_r,                    \
      SubArray<D, T> & dcoeff_cf, SubArray<D, T> & dcoeff_rf,                  \
      SubArray<D, T> & dcoeff_rc, SubArray<D, T> & dcoeff_rcf);                \
  template void decompose<D, T>(Handle<D, T> & handle, T * dv,                 \
                                std::vector<SIZE> ldvs_h, SIZE * ldvs_d,       \
                                SIZE l_target, int queue_idx);                 \
  template void recompose<D, T>(Handle<D, T> & handle, T * dv,                 \
                                std::vector<SIZE> ldvs_h, SIZE * ldvs_d,       \
                                SIZE l_target, int queue_idx);

KERNELS(1, double)
KERNELS(1, float)
KERNELS(2, double)
KERNELS(2, float)
KERNELS(3, double)
KERNELS(3, float)
KERNELS(4, double)
KERNELS(4, float)
KERNELS(5, double)
KERNELS(5, float)
#undef KERNELS

#define KERNELS(D, T)                                                          \
  template void calc_coefficients_3d<D, T>(                                    \
      Handle<D, T> & handle, SubArray<D, T> dinput, SubArray<D, T> & doutput,  \
      SIZE l, int queue_idx);                                                  \
  template void coefficients_restore_3d<D, T>(                                 \
      Handle<D, T> & handle, SubArray<D, T> dinput, SubArray<D, T> & doutput,  \
      SIZE l, int queue_idx);                                                  \
  template void calc_correction_3d<D, T>(                                      \
      Handle<D, T> & handle, SubArray<D, T> dcoeff,                            \
      SubArray<D, T> & dcorrection, SIZE l, int queue_idx);
KERNELS(1, double)
KERNELS(1, float)
KERNELS(2, double)
KERNELS(2, float)
KERNELS(3, double)
KERNELS(3, float)
#undef KERNELS

#define KERNELS(D, T)                                                          \
  template void calc_coefficients_nd<D, T>(                                    \
      Handle<D, T> & handle, SubArray<D, T> dinput1, SubArray<D, T> dinput2,   \
      SubArray<D, T> & doutput, SIZE l, int queue_idx);                        \
  template void coefficients_restore_nd<D, T>(                                 \
      Handle<D, T> & handle, SubArray<D, T> dinput1, SubArray<D, T> dinput2,   \
      SubArray<D, T> & doutput, SIZE l, int queue_idx);                        \
  template void calc_correction_nd<D, T>(                                      \
      Handle<D, T> & handle, SubArray<D, T> dcoeff,                            \
      SubArray<D, T> & dcorrection, SIZE l, int queue_idx);

KERNELS(4, double)
KERNELS(4, float)
KERNELS(5, double)
KERNELS(5, float)
#undef KERNELS

} // namespace mgard_cuda
