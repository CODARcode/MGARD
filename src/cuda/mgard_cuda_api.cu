#include "hip/hip_runtime.h"
/* 
 * Copyright 2021, Oak Ridge National Laboratory.
 * MGARD-GPU: MultiGrid Adaptive Reduction of Data Accelerated by GPUs
 * Author: Jieyang Chen (chenj3@ornl.gov)
 * Date: April 2, 2021
 */

#include <assert.h>
#include <iostream>

#include "cuda/mgard_cuda_compression_workflow.h"
#include "cuda/mgard_cuda_common.h"

namespace mgard_cuda {

template <typename T, int D>
unsigned char *compress(mgard_cuda_handle<T, D> &handle, T *v, size_t &out_size, T tol, T s)
// Perform compression preserving the tolerance in the L-infty norm
{
  assert(tol >= 1e-7);
  unsigned char *mgard_compressed_ptr = mgard_cuda::refactor_qz_cuda<T, D>(handle, v, out_size, tol, s);
  return mgard_compressed_ptr;
}

template <typename T, int D>
T *decompress(mgard_cuda_handle<T, D> &handle, unsigned char *data,
                         size_t data_len) {
  T *mgard_decompressed_ptr = mgard_cuda::recompose_udq_cuda<T, D>(handle, data, data_len);
  return mgard_decompressed_ptr;
}


#define API(T, D) \
        template unsigned char * compress<T, D>(\
        mgard_cuda_handle<T, D> &handle,\
        T *v, size_t &out_size, T tol, T s);\
        template T * decompress<T, D>(\
        mgard_cuda_handle<T, D> &handle,\
        unsigned char *data, size_t data_len);

API(double, 1)
API(float,  1)
API(double, 2)
API(float,  2)
API(double, 3)
API(float,  3)
API(double, 4)
API(float,  4)
API(double, 5)
API(float,  5)

#undef API

} // end namespace