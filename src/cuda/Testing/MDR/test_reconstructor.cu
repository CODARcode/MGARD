#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <ctime>
#include <cstdlib>
#include <vector>
#include <iomanip>
#include <cmath>
#include <bitset>
// #include "utils.hpp"
#include "cuda/MDR/Reconstructor/Reconstructor.hpp"
// #include "evaluate.hpp"

using namespace std;

template<typename Type>
std::vector<Type> readfile(const char *file, size_t &num) {
    std::ifstream fin(file, std::ios::binary);
    if (!fin) {
        std::cout << " Error, Couldn't find the file" << "\n";
        return std::vector<Type>();
    }
    fin.seekg(0, std::ios::end);
    const size_t num_elements = fin.tellg() / sizeof(Type);
    fin.seekg(0, std::ios::beg);
    auto data = std::vector<Type>(num_elements);
    fin.read(reinterpret_cast<char *>(&data[0]), num_elements * sizeof(Type));
    fin.close();
    num = num_elements;
    return data;
}


template <class T>
void print_statistics(const T * data_ori, const T * data_dec, size_t data_size){
    double max_val = data_ori[0];
    double min_val = data_ori[0];
    double max_abs = fabs(data_ori[0]);
    for(int i=0; i<data_size; i++){
        if(data_ori[i] > max_val) max_val = data_ori[i];
        if(data_ori[i] < min_val) min_val = data_ori[i];
        if(fabs(data_ori[i]) > max_abs) max_abs = fabs(data_ori[i]);
    }
    double max_err = 0;
    int pos = 0;
    double mse = 0;
    for(int i=0; i<data_size; i++){
        double err = data_ori[i] - data_dec[i];
        mse += err * err;
        if(fabs(err) > max_err){
            pos = i;
            max_err = fabs(err);
        }
    }
    mse /= data_size;
    double psnr = 20 * log10((max_val - min_val) / sqrt(mse));
    cout << "Max value = " << max_val << ", min value = " << min_val << endl;
    cout << "Max error = " << max_err << ", pos = " << pos << endl;
    cout << "MSE = " << mse << ", PSNR = " << psnr << endl;
}


template <class T, class Reconstructor>
void evaluate(const vector<T>& data, const vector<double>& tolerance, Reconstructor reconstructor){
    struct timespec start, end;
    int err = 0;
    // auto a1 = compute_average(data.data(), dims[0], dims[1], dims[2], 3);
    // auto a12 = compute_average(data.data(), dims[0], dims[1], dims[2], 5);
    for(int i=0; i<tolerance.size(); i++){
        cout << "Start reconstruction" << endl;
        err = clock_gettime(CLOCK_REALTIME, &start);
        auto reconstructed_data = reconstructor.progressive_reconstruct(tolerance[i]);
        err = clock_gettime(CLOCK_REALTIME, &end);
        cout << "Reconstruct time: " << (double)(end.tv_sec - start.tv_sec) + (double)(end.tv_nsec - start.tv_nsec)/(double)1000000000 << "s" << endl;
        auto dims = reconstructor.get_dimensions();
        size_t size = 1;
        for (int i = 0; i < dims.size(); i++) {
            size *= dims[i];
        }
        print_statistics(data.data(), reconstructed_data, size);
        // COMP_UTILS::evaluate_gradients(data.data(), reconstructed_data, dims[0], dims[1], dims[2]);
        // COMP_UTILS::evaluate_average(data.data(), reconstructed_data, dims[0], dims[1], dims[2], 0);
    }
}

template <class T, class Decomposer, class Interleaver, class Encoder, class Compressor, class ErrorEstimator, class SizeInterpreter, class Retriever>
void test(string filename, const vector<double>& tolerance, Decomposer decomposer, Interleaver interleaver, Encoder encoder, Compressor compressor, ErrorEstimator estimator, SizeInterpreter interpreter, Retriever retriever){
    auto reconstructor = mgard_cuda::MDR::ComposedReconstructor<T, Decomposer, Interleaver, Encoder, Compressor, SizeInterpreter, ErrorEstimator, Retriever>(decomposer, interleaver, encoder, compressor, interpreter, retriever);
    cout << "loading metadata" << endl;
    reconstructor.load_metadata();

    size_t num_elements = 0;
    auto data = readfile<T>(filename.c_str(), num_elements);
    evaluate(data, tolerance, reconstructor);
}

template <typename HandleType, mgard_cuda::DIM D, class T, class T_stream, class Decomposer, class Interleaver, class Encoder, class Compressor, class ErrorEstimator, class SizeInterpreter, class Retriever>
void test2(string filename, const vector<double>& tolerance, HandleType& handle, Decomposer decomposer, Interleaver interleaver, Encoder encoder, Compressor compressor, ErrorEstimator estimator, SizeInterpreter interpreter, Retriever retriever){
    auto reconstructor = mgard_m::MDR::ComposedReconstructor<HandleType, D, T, T_stream, Decomposer, Interleaver, Encoder, Compressor, SizeInterpreter, ErrorEstimator, Retriever>(handle, decomposer, interleaver, encoder, compressor, interpreter, retriever);
    cout << "loading metadata" << endl;
    reconstructor.load_metadata();

    size_t num_elements = 0;
    auto data = readfile<T>(filename.c_str(), num_elements);
    evaluate(data, tolerance, reconstructor);
}

int main(int argc, char ** argv){

    int argv_id = 1;
    string filename = string(argv[argv_id ++]);
    int error_mode = atoi(argv[argv_id++]);
    int num_tolerance = atoi(argv[argv_id ++]);
    vector<double> tolerance(num_tolerance, 0);
    for(int i=0; i<num_tolerance; i++){
        tolerance[i] = atof(argv[argv_id ++]);    
    }
    double s = atof(argv[argv_id ++]);

    string metadata_file = "refactored_data/metadata.bin";
    int num_levels = 0;
    int num_dims = 0;
    vector<uint32_t> dims;
    {
        // metadata interpreter, otherwise information needs to be provided
        size_t num_bytes = 0;
        auto metadata = readfile<uint8_t>(metadata_file.c_str(), num_bytes);
        assert(num_bytes > num_dims * sizeof(uint32_t) + 2);
        num_dims = metadata[0];
        uint32_t * dim = (uint32_t*)&(metadata[1]);
        printf("dim: ");
        for (int i = 0; i < num_dims; i++) {
            dims.push_back(dim[i]);
            printf("%u ", dim[i]);
        }
        printf("\n");
        num_levels = metadata[num_dims * sizeof(uint32_t) + 1];
        cout << "number of dimension = " << num_dims << ", number of levels = " << num_levels << endl;

    }
    vector<string> files;
    for(int i=0; i<num_levels; i++){
        string filename = "refactored_data/level_" + to_string(i) + ".bin";
        files.push_back(filename);
    }

    using T = float;
    using T_stream = uint32_t;
    using T_error = double;

    const mgard_cuda::DIM D = 3;
    using HandleType = mgard_cuda::Handle<D, T>;
    mgard_cuda::Config config;
    config.l_target = num_levels - 1;
    HandleType handle(dims, config);

    if (false) {
        auto decomposer = mgard_cuda::MDR::MGARDOrthoganalDecomposer<D, T>(handle);
        // auto decomposer = MDR::MGARDHierarchicalDecomposer<T>();
        auto interleaver = mgard_cuda::MDR::DirectInterleaver<D, T>(handle);
        // auto interleaver = MDR::SFCInterleaver<T>();
        // auto interleaver = MDR::BlockedInterleaver<T>();



        auto encoder = mgard_cuda::MDR::GroupedBPEncoder<D, T, T_stream>(handle);
        // auto encoder = mgard_cuda::MDR::NegaBinaryBPEncoder<D, T, T_stream>(handle);
        // auto encoder = mgard_cuda::MDR::PerBitBPEncoder<D, T, T_stream>(handle);


        // auto encoder = mgard_cuda::MDR::PerBitBPEncoderGPU<D, T, T_stream>(handle);
        // auto encoder = mgard_cuda::MDR::GroupedBPEncoderGPU<D, T, T_stream>(handle);
        

        auto compressor = mgard_cuda::MDR::DefaultLevelCompressor();
        // auto compressor = mgard_cuda::MDR::AdaptiveLevelCompressor(32);
        // auto compressor = MDR::NullLevelCompressor();
        auto retriever = mgard_cuda::MDR::ConcatLevelFileRetriever(metadata_file, files);
        switch(error_mode){
            case 1:{
                auto estimator = mgard_cuda::MDR::SNormErrorEstimator<T>(num_dims, num_levels - 1, s);
                // auto interpreter = mgard_cuda::MDR::SignExcludeGreedyBasedSizeInterpreter<mgard_cuda::MDR::SNormErrorEstimator<T>>(estimator);
                // auto interpreter = mgard_cuda::MDR::NegaBinaryGreedyBasedSizeInterpreter<mgard_cuda::MDR::SNormErrorEstimator<T>>(estimator);
                auto interpreter = mgard_cuda::MDR::RoundRobinSizeInterpreter<mgard_cuda::MDR::SNormErrorEstimator<T>>(estimator);
                // auto interpreter = mgard_cuda::MDR::InorderSizeInterpreter<mgard_cuda::MDR::SNormErrorEstimator<T>>(estimator);
                // auto estimator = mgard_cuda::MDR::L2ErrorEstimator_HB<T>(num_dims, num_levels - 1);
                // auto interpreter = mgard_cuda::MDR::SignExcludeGreedyBasedSizeInterpreter<mgard_cuda::MDR::L2ErrorEstimator_HB<T>>(estimator);
                test<T>(filename, tolerance, decomposer, interleaver, encoder, compressor, estimator, interpreter, retriever);            
                break;
            }
            default:{
                auto estimator = mgard_cuda::MDR::MaxErrorEstimatorOB<T>(num_dims);
                auto interpreter = mgard_cuda::MDR::SignExcludeGreedyBasedSizeInterpreter<mgard_cuda::MDR::MaxErrorEstimatorOB<T>>(estimator);
                // auto interpreter = MDR::RoundRobinSizeInterpreter<MDR::MaxErrorEstimatorOB<T>>(estimator);
                // auto interpreter = MDR::InorderSizeInterpreter<MDR::MaxErrorEstimatorOB<T>>(estimator);
                // auto estimator = MDR::MaxErrorEstimatorHB<T>();
                // auto interpreter = MDR::SignExcludeGreedyBasedSizeInterpreter<MDR::MaxErrorEstimatorHB<T>>(estimator);
                test<T>(filename, tolerance, decomposer, interleaver, encoder, compressor, estimator, interpreter, retriever);
            }
        }    

    }

    if (true) {
        auto decomposer = mgard_m::MDR::MGARDOrthoganalDecomposer<HandleType, D, T>(handle);
        auto interleaver = mgard_m::MDR::DirectInterleaver<HandleType, D, T>(handle);
        // auto encoder = mgard_m::MDR::GroupedBPEncoder<HandleType, D, T, T_stream, T_error>(handle);
        auto encoder = mgard_m::MDR::GroupedWarpBPEncoder<HandleType, D, T, T_stream, T_error>(handle);

        auto compressor = mgard_m::MDR::DefaultLevelCompressor<HandleType, D, T_stream>(handle);
        auto retriever = mgard_cuda::MDR::ConcatLevelFileRetriever(metadata_file, files);
        switch(error_mode){
            case 1:{
                auto estimator = mgard_cuda::MDR::SNormErrorEstimator<T>(num_dims, num_levels - 1, s);
                // auto interpreter = MDR::SignExcludeGreedyBasedSizeInterpreter<MDR::SNormErrorEstimator<T>>(estimator);
                // auto interpreter = mgard_cuda::MDR::NegaBinaryGreedyBasedSizeInterpreter<mgard_cuda::MDR::SNormErrorEstimator<T>>(estimator);
                auto interpreter = mgard_cuda::MDR::RoundRobinSizeInterpreter<mgard_cuda::MDR::SNormErrorEstimator<T>>(estimator);
                // auto interpreter = mgard_cuda::MDR::InorderSizeInterpreter<MDR::SNormErrorEstimator<T>>(estimator);
                // auto estimator = mgard_cuda::MDR::L2ErrorEstimator_HB<T>(num_dims, num_levels - 1);
                // auto interpreter = mgard_cuda::MDR::SignExcludeGreedyBasedSizeInterpreter<MDR::L2ErrorEstimator_HB<T>>(estimator);
                test2<HandleType, D, T, T_stream>(filename, tolerance, handle, decomposer, interleaver, encoder, compressor, estimator, interpreter, retriever);            
                break;
            }
            default:{
                auto estimator = mgard_cuda::MDR::MaxErrorEstimatorOB<T>(num_dims);
                auto interpreter = mgard_cuda::MDR::SignExcludeGreedyBasedSizeInterpreter<mgard_cuda::MDR::MaxErrorEstimatorOB<T>>(estimator);
                // auto interpreter = mgard_cuda::MDR::RoundRobinSizeInterpreter<MDR::MaxErrorEstimatorOB<T>>(estimator);
                // auto interpreter = mgard_cuda::MDR::InorderSizeInterpreter<MDR::MaxErrorEstimatorOB<T>>(estimator);
                // auto estimator = mgard_cuda::MDR::MaxErrorEstimatorHB<T>();
                // auto interpreter = mgard_cuda::MDR::SignExcludeGreedyBasedSizeInterpreter<MDR::MaxErrorEstimatorHB<T>>(estimator);
                test2<HandleType, D, T, T_stream>(filename, tolerance, handle, decomposer, interleaver, encoder, compressor, estimator, interpreter, retriever);
            }
        }    

    }


    return 0;
}