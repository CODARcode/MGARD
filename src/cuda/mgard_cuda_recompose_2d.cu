#include "hip/hip_runtime.h"
#include "cuda/mgard_cuda_recompose_2d.h"
#include "cuda/mgard_cuda_common_internal.h"
#include "cuda/mgard_cuda_kernels.h"
#include <fstream>

namespace mgard_cuda {

template <typename T> 
void 
recompose_2D_cuda(mgard_cuda_handle<T> & handle, T * dv, int lddv){ 
 
  for (int l = handle.l_target; l > 0; --l) {

    int stride = std::pow(2, l); // current stride
    int Pstride = stride / 2;

    copy_level(handle,
               handle.nrow, handle.ncol,
               handle.nr, handle.nc,
               Pstride, Pstride,
               handle.dirow, handle.dicol,
               dv, lddv, 
               handle.dwork, handle.lddwork,
               0);

    assign_num_level(handle,
                     handle.nrow, handle.ncol,
                     handle.nr, handle.nc,
                     stride, stride,
                     handle.dirow, handle.dicol,
                     (T)0.0,
                     handle.dwork, handle.lddwork, 
                     0);

    mass_multiply_1(handle,
                    handle.nrow, handle.ncol,
                    handle.nr, handle.nc,
                    Pstride, Pstride,
                    handle.dirow, handle.dicol,
                    handle.dcoords_c, 
                    handle.dwork, handle.lddwork,
                    0);

    restriction_1(handle,
                  handle.nrow, handle.ncol,
                  handle.nr, handle.nc,
                  Pstride, Pstride,
                  handle.dirow, handle.dicol,
                  handle.dcoords_c,
                  handle.dwork, handle.lddwork,
                  0);

    solve_tridiag_1(handle,
                    handle.nrow, handle.ncol,
                    handle.nr, handle.nc,
                    Pstride, stride,
                    handle.dirow, handle.dicol,
                    handle.dcoords_c,
                    handle.dwork, handle.lddwork,
                    0);

    mass_multiply_2(handle,
                    handle.nrow, handle.ncol,
                    handle.nr, handle.nc,
                    Pstride, stride,
                    handle.dirow, handle.dicol,
                    handle.dcoords_r,
                    handle.dwork, handle.lddwork,
                    0);

    restriction_2(handle,
                  handle.nrow, handle.ncol,
                  handle.nr, handle.nc,
                  Pstride, stride,
                  handle.dirow, handle.dicol,
                  handle.dcoords_r,
                  handle.dwork, handle.lddwork,
                  0);

    solve_tridiag_2(handle,
                    handle.nrow, handle.ncol,
                    handle.nr, handle.nc,
                    stride, stride,
                    handle.dirow, handle.dicol,
                    handle.dcoords_r,
                    handle.dwork, handle.lddwork,
                    0);

    subtract_level(handle, 
                   handle.nrow, handle.ncol, 
                   handle.nr, handle.nc,
                   stride, stride,
                   handle.dirow, handle.dicol,
                   dv, lddv, 
                   handle.dwork, handle.lddwork,
                   0);

    prolongate(handle,
               handle.nrow, handle.ncol,
               handle.nr, handle.nc,
               Pstride, Pstride,
               handle.dirow, handle.dicol,
               handle.dcoords_r, handle.dcoords_c,
               dv, lddv,
               0);
  }
}

template void 
recompose_2D_cuda<double>(mgard_cuda_handle<double> & handle, double * dv, int lddv);

template void 
recompose_2D_cuda<float>(mgard_cuda_handle<float> & handle, float * dv, int lddv);


template <typename T> 
void 
recompose_2D_cuda_cpt(mgard_cuda_handle<T> & handle, T * dv, int lddv) {
 
  T * dcv;
  size_t dcv_pitch;
  cudaMallocPitchHelper((void**)&dcv, &dcv_pitch, handle.nc * sizeof(T), handle.nr);
  int lddcv = dcv_pitch / sizeof(T);

  org_to_pow2p1(handle,
                handle.nrow, handle.ncol,
                handle.nr, handle.nc,
                handle.dirow, handle.dicol,
                dv, lddv,
                dcv, lddcv,
                0);

  for (int l = handle.l_target; l > 0; --l) {
    int stride = std::pow(2, l); // current stride
    int Pstride = stride / 2;

    pow2p1_to_cpt_num_assign(handle,
                             handle.nr, handle.nc,
                             Pstride, Pstride,
                             (T)0.0,
                             dcv, lddcv,  
                             handle.dwork, handle.lddwork,
                             0);

    mass_multiply_1_cpt(handle,
                        handle.nr_l[l-1], handle.nc_l[l-1],
                        1, 1,
                        handle.ddist_c_l[l-1],
                        handle.dwork, handle.lddwork,
                        0);

    restriction_1_cpt(handle,
                      handle.nr_l[l-1], handle.nc_l[l-1],
                      1, 1,
                      handle.ddist_c_l[l-1],
                      handle.dwork, handle.lddwork,
                      0);

    solve_tridiag_1_cpt(handle, 
                        handle.nr_l[l-1], handle.nc_l[l-1],
                        1, 2,
                        handle.ddist_c_l[l],
                        handle.am_col[0], handle.bm_col[0],
                        handle.dwork, handle.lddwork,
                        0);

    mass_multiply_2_cpt(handle,
                        handle.nr_l[l-1], handle.nc_l[l-1],
                        1, 2,
                        handle.ddist_r_l[l-1],
                        handle.dwork, handle.lddwork,
                        0);

    restriction_2_cpt(handle,
                      handle.nr_l[l-1], handle.nc_l[l-1],
                      1, 2,
                      handle.ddist_r_l[l-1],
                      handle.dwork, handle.lddwork,
                      0);
    
    solve_tridiag_2_cpt(handle,
                      handle.nr_l[l-1], handle.nc_l[l-1],
                      2, 2,
                      handle.ddist_r_l[l],
                      handle.am_row[0], handle.bm_row[0],
                      handle.dwork, handle.lddwork,
                      0);

    cpt_to_pow2p1_subtract(handle,
                           handle.nr, handle.nc,
                           2,2, 
                           stride, stride,
                           handle.dwork, handle.lddwork,
                           dcv, lddcv,
                           0);

    prolongate_cpt(handle,
                   handle.nr, handle.nc, 
                   Pstride, Pstride,
                   handle.ddist_r_l[l-1], handle.ddist_c_l[l-1], 
                   dcv,        lddcv,  
                   0);
  }

  pow2p1_to_org(handle,
                handle.nrow, handle.ncol,
                handle.nr, handle.nc,
                handle.dirow, handle.dicol,
                dcv, lddcv,
                dv, lddv, 
                0);

  cudaFreeHelper(dcv);
}

template void 
recompose_2D_cuda_cpt<double>(mgard_cuda_handle<double> & handle, double * dv, int lddv);

template void 
recompose_2D_cuda_cpt<float>(mgard_cuda_handle<float> & handle, float * dv, int lddv);

}